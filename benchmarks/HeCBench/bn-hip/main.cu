#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include "kernels.cu"

const int HIGHEST = 3;
const int ITER = 100;
const int WORKLOAD = 1;
int sizepernode;

// global var
float preScore = -99999999999.f;
float score = 0.f;
float maxScore[HIGHEST] = {-999999999.f};
bool orders[NODE_N][NODE_N];
bool preOrders[NODE_N][NODE_N];
bool preGraph[NODE_N][NODE_N];
bool bestGraph[HIGHEST][NODE_N][NODE_N];
bool graph[NODE_N][NODE_N];
float *localscore, *scores;
float *LG;
int *parents;

void initial();  // initial orders and data
int genOrders(); // swap
int ConCore();   // discard new order or not
// get every possible set of parents for a node
void incr(int *bit, int n);  // binary code increases 1 each time
void incrS(int *bit, int n); // STATE_N code increases 1 each time
// get every possible combination of state for a parent set
bool getState( int parN, int *state, int time); 
float logGamma(int N); // log and gamma
float findBestGraph(float* D_localscore, int* D_resP, float* D_Score, bool *D_parent);
void genScore();
void sortGraph();
void swap(int a, int b);
void Pre_logGamma();
int findindex(int *arr, int size);
int C(int n, int a);

FILE *fpout;

int main(int argc, char** argv) {

  // save output in a file
  fpout = fopen(argv[1], "w");
  if (fpout == NULL) {
    printf("Usage: ./%s <output file>\n", argv[0]);
    return -1;
  }

  int i, j, c = 0, tmp, a, b;
  float tmpd;

  clock_t start, finish, total = 0, pre1, pre2;

  printf("NODE_N=%d\nInitialization...\n", NODE_N);

  srand(2);

  initial(); // update sizepernode
  scores = (float*) malloc ((sizepernode / (256 * WORKLOAD) + 1) * sizeof(float));
  parents = (int*) malloc ((sizepernode / (256 * WORKLOAD) + 1) * 4 * sizeof(int));


  Pre_logGamma();

  int *D_data;
  float *D_LG;
  float *D_localscore;
  float *D_Score;
  bool *D_parent;
  int *D_resP;
  hipMalloc((void **)&D_data, NODE_N * DATA_N * sizeof(int));
  hipMalloc((void **)&D_localscore, NODE_N * sizepernode * sizeof(float));
  hipMalloc((void **)&D_LG, (DATA_N + 2) * sizeof(float));
  hipMalloc((void **)&D_Score, (sizepernode / (256 * WORKLOAD) + 1) * sizeof(float));
  hipMalloc((void **)&D_parent, NODE_N * sizeof(bool)); 
  hipMalloc((void **)&D_resP, (sizepernode / (256 * WORKLOAD) + 1) * 4 * sizeof(int));

  pre1 = clock();

  dim3 grid(sizepernode / 256 + 1, 1, 1);
  dim3 threads(256, 1, 1);

  hipMemset(D_localscore, 0.f, NODE_N * sizepernode * sizeof(float));
  hipMemcpy(D_data, data, NODE_N * DATA_N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(D_LG, LG, (DATA_N + 2) * sizeof(float), hipMemcpyHostToDevice);
  hipLaunchKernelGGL(genScoreKernel, dim3(grid), dim3(threads), 0, 0, sizepernode, D_localscore, D_data, D_LG);
  hipMemcpy(localscore, D_localscore, NODE_N * sizepernode * sizeof(float), hipMemcpyDeviceToHost);

  printf("Begin to generate orders.\n");


  pre2 = clock();
  i = 0;
  while (i != ITER) {

    start = clock();

    i++;
    score = 0;

    for (a = 0; a < NODE_N; a++) {
      for (j = 0; j < NODE_N; j++) {
        orders[a][j] = preOrders[a][j];
      }
    }

    tmp = rand() % 6;
    for (j = 0; j < tmp; j++)
      genOrders();

    score = findBestGraph(D_localscore, D_resP, D_Score, D_parent);

    finish = clock();
    total += finish - start;

    ConCore();

    // store the top HIGHEST highest orders
    if (c < HIGHEST) {
      tmp = 1;
      for (j = 0; j < c; j++) {
        if (maxScore[j] == preScore) {
          tmp = 0;
        }
      }
      if (tmp != 0) {
        maxScore[c] = preScore;
        for (a = 0; a < NODE_N; a++) {
          for (b = 0; b < NODE_N; b++) {
            bestGraph[c][a][b] = preGraph[a][b];
          }
        }
        c++;
      }

    } else if (c == HIGHEST) {
      sortGraph();
      c++;
    } else {

      tmp = 1;
      for (j = 0; j < HIGHEST; j++) {
        if (maxScore[j] == preScore) {
          tmp = 0;
          break;
        }
      }
      if (tmp != 0 && preScore > maxScore[HIGHEST - 1]) {
        maxScore[HIGHEST - 1] = preScore;
        for (a = 0; a < NODE_N; a++) {
          for (b = 0; b < NODE_N; b++) {
            bestGraph[HIGHEST - 1][a][b] = preGraph[a][b];
          }
        }
        b = HIGHEST - 1;
        for (a = HIGHEST - 2; a >= 0; a--) {
          if (maxScore[b] > maxScore[a]) {
            swap(a, b);
            tmpd = maxScore[a];
            maxScore[a] = maxScore[b];
            maxScore[b] = tmpd;
            b = a;
          }
        }
      }
    }

  } // endwhile

  free(localscore);
  free(scores);
  free(parents);
  free(LG);
  hipFree(D_LG);
  hipFree(D_data);
  hipFree(D_localscore);
  hipFree(D_parent);
  hipFree(D_Score);
  hipFree(D_resP);

  for(j=0;j<HIGHEST;j++){
    fprintf(fpout,"score:%f\n",maxScore[j]);
    fprintf(fpout,"Best Graph:\n");
    for(int a=0;a<NODE_N;a++){
      for(int b=0;b<NODE_N;b++)
        fprintf(fpout,"%d ",bestGraph[j][a][b]);
      fprintf(fpout,"\n");
    }
    fprintf(fpout,"--------------------------------------------------------------------\n");
  }

  fprintf(fpout, "Duration per iteration is %f seconds.\n",
      ((float)total / ITER) / CLOCKS_PER_SEC);
  fprintf(fpout, "Total duration is %f seconds.\n",
      (float)(pre2 - pre1 + total) / CLOCKS_PER_SEC);
  fprintf(fpout, "Preprocessing duration is %f seconds.\n",
      (float)(pre2 - pre1) / CLOCKS_PER_SEC);

  printf("Duration per iteration is %f seconds.\n",
      ((float)total / ITER) / CLOCKS_PER_SEC);
  printf("Total duration is %f seconds.\n",
      (float)(pre2 - pre1 + total) / CLOCKS_PER_SEC);
  printf("Preprocessing duration is %f seconds.\n",
      (float)(pre2 - pre1) / CLOCKS_PER_SEC);

  return 0;
}


float findBestGraph(float* D_localscore, int* D_resP, float* D_Score, bool *D_parent) {
  float bestls = -99999999.f;
  int bestparent[5];
  int bestpN, total;
  int node, index;
  int pre[NODE_N] = {0};
  int parent[NODE_N] = {0};
  int posN = 0, i, j, parN, tmp, k, l;
  float ls = -99999999999.f, score = 0;
  int blocknum;

  for (i = 0; i < NODE_N; i++)
    for (j = 0; j < NODE_N; j++)
      graph[i][j] = 0;

  for (node = 0; node < NODE_N; node++) {

    bestls = -99999999.f;
    posN = 0;

    for (i = 0; i < NODE_N; i++) {
      if (orders[node][i] == 1) {
        pre[posN++] = i;
      }
    }

    if (posN >= 0) {
      total = C(posN, 4) + C(posN, 3) + C(posN, 2) + posN + 1;
      blocknum = total / (256 * WORKLOAD) + 1;

      hipMemset(D_resP, 0, blocknum * 4 * sizeof(int));
      hipMemset(D_Score, -999999.f, blocknum * sizeof(float));
      hipMemcpy(D_parent, orders[node], NODE_N * sizeof(bool), hipMemcpyHostToDevice);

      hipLaunchKernelGGL(computeKernel, dim3(blocknum), dim3(256), 256 * sizeof(float), 0, 
          WORKLOAD, sizepernode, D_localscore, D_parent, node, total, D_Score,
          D_resP);
      hipMemcpy(parents, D_resP, blocknum * 4 * sizeof(int), hipMemcpyDeviceToHost);
      hipMemcpy(scores, D_Score, blocknum * sizeof(float), hipMemcpyDeviceToHost);

      for (i = 0; i < blocknum; i++) {

        if (scores[i] > bestls) {

          bestls = scores[i];

          parN = 0;
          for (tmp = 0; tmp < 4; tmp++) {
            if (parents[i * 4 + tmp] < 0)
              break;

            bestparent[tmp] = parents[i * 4 + tmp];

            parN++;
          }

          bestpN = parN;
        }
      }
    } else {
      if (posN >= 4) {
        for (i = 0; i < posN; i++) {
          for (j = i + 1; j < posN; j++) {
            for (k = j + 1; k < posN; k++) {
              for (l = k + 1; l < posN; l++) {
                parN = 4;
                if (pre[i] > node)
                  parent[1] = pre[i];
                else
                  parent[1] = pre[i] + 1;
                if (pre[j] > node)
                  parent[2] = pre[j];
                else
                  parent[2] = pre[j] + 1;
                if (pre[k] > node)
                  parent[3] = pre[k];
                else
                  parent[3] = pre[k] + 1;
                if (pre[l] > node)
                  parent[4] = pre[l];
                else
                  parent[4] = pre[l] + 1;

                index = findindex(parent, parN);
                index += sizepernode * node;
                ls = localscore[index];

                if (ls > bestls) {
                  bestls = ls;
                  bestpN = parN;
                  for (tmp = 0; tmp < parN; tmp++)
                    bestparent[tmp] = parent[tmp + 1];
                }
              }
            }
          }
        }
      }

      if (posN >= 3) {
        for (i = 0; i < posN; i++) {
          for (j = i + 1; j < posN; j++) {
            for (k = j + 1; k < posN; k++) {

              parN = 3;
              if (pre[i] > node)
                parent[1] = pre[i];
              else
                parent[1] = pre[i] + 1;
              if (pre[j] > node)
                parent[2] = pre[j];
              else
                parent[2] = pre[j] + 1;
              if (pre[k] > node)
                parent[3] = pre[k];
              else
                parent[3] = pre[k] + 1;

              index = findindex(parent, parN);
              index += sizepernode * node;
              ls = localscore[index];

              if (ls > bestls) {
                bestls = ls;
                bestpN = parN;
                for (tmp = 0; tmp < parN; tmp++)
                  bestparent[tmp] = parent[tmp + 1];
              }
            }
          }
        }
      }

      if (posN >= 2) {
        for (i = 0; i < posN; i++) {
          for (j = i + 1; j < posN; j++) {

            parN = 2;
            if (pre[i] > node)
              parent[1] = pre[i];
            else
              parent[1] = pre[i] + 1;
            if (pre[j] > node)
              parent[2] = pre[j];
            else
              parent[2] = pre[j] + 1;

            index = findindex(parent, parN);
            index += sizepernode * node;
            ls = localscore[index];

            if (ls > bestls) {
              bestls = ls;
              bestpN = parN;
              for (tmp = 0; tmp < parN; tmp++)
                bestparent[tmp] = parent[tmp + 1];
            }
          }
        }
      }

      if (posN >= 1) {
        for (i = 0; i < posN; i++) {

          parN = 1;
          if (pre[i] > node)
            parent[1] = pre[i];
          else
            parent[1] = pre[i] + 1;

          index = findindex(parent, parN);
          index += sizepernode * node;
          ls = localscore[index];

          if (ls > bestls) {
            bestls = ls;
            bestpN = parN;
            for (tmp = 0; tmp < parN; tmp++)
              bestparent[tmp] = parent[tmp + 1];
          }
        }
      }

      parN = 0;
      index = sizepernode * node;

      ls = localscore[index];

      if (ls > bestls) {
        bestls = ls;
        bestpN = 0;
      }
    }
    if (bestls > -99999999.f) {

      for (i = 0; i < bestpN; i++) {
        if (bestparent[i] < node)
          graph[node][bestparent[i] - 1] = 1;
        else
          graph[node][bestparent[i]] = 1;
      }
      score += bestls;
    }
  }

  return score;
}


void sortGraph() {
  float max = -99999999999999.f;
  int maxi, i, j;
  float tmp;

  for (j = 0; j < HIGHEST - 1; j++) {
    max = maxScore[j];
    maxi = j;
    for (i = j + 1; i < HIGHEST; i++) {
      if (maxScore[i] > max) {
        max = maxScore[i];
        maxi = i;
      }
    }

    swap(j, maxi);
    tmp = maxScore[j];
    maxScore[j] = max;
    maxScore[maxi] = tmp;
  }
}

void swap(int a, int b) {
  int i, j;
  bool tmp;

  for (i = 0; i < NODE_N; i++) {
    for (j = 0; j < NODE_N; j++) {

      tmp = bestGraph[a][i][j];
      bestGraph[a][i][j] = bestGraph[b][i][j];
      bestGraph[b][i][j] = tmp;
    }
  }
}

void initial() {
  int i, j, tmp, a, b, r;
  bool tmpd;
  tmp = 1;
  for (i = 1; i <= 4; i++) {
    tmp += C(NODE_N - 1, i);
  }
  sizepernode = tmp;
  tmp *= NODE_N;

  localscore = (float*) malloc(tmp * sizeof(float));

  for (i = 0; i < tmp; i++)
    localscore[i] = 0;

  for (i = 0; i < NODE_N; i++) {
    for (j = 0; j < NODE_N; j++)
      orders[i][j] = 0;
  }
  for (i = 0; i < NODE_N; i++) {
    for (j = 0; j < i; j++)
      orders[i][j] = 1;
  }
  r = rand() % 10000;
  for (i = 0; i < r; i++) {
    a = rand() % NODE_N;
    b = rand() % NODE_N;
    for (j = 0; j < NODE_N; j++) {
      tmpd = orders[j][a];
      orders[j][a] = orders[j][b];
      orders[j][b] = tmpd;
    }

    for (j = 0; j < NODE_N; j++) {
      tmpd = orders[a][j];
      orders[a][j] = orders[b][j];
      orders[b][j] = tmpd;
    }
  }

  for (i = 0; i < NODE_N; i++) {
    for (j = 0; j < NODE_N; j++) {
      preOrders[i][j] = orders[i][j];
    }
  }
}

// generate ramdom order
int genOrders() {

  int a, b, j;
  bool tmp;
  a = rand() % NODE_N;
  b = rand() % NODE_N;

  for (j = 0; j < NODE_N; j++) {
    tmp = orders[a][j];
    orders[a][j] = orders[b][j];
    orders[b][j] = tmp;
  }
  for (j = 0; j < NODE_N; j++) {
    tmp = orders[j][a];
    orders[j][a] = orders[j][b];
    orders[j][b] = tmp;
  }

  return 1;
}

// decide leave or discard an order
int ConCore() {
  int i, j;
  float tmp;
  tmp = log((rand() % 100000) / 100000.0);
  if (tmp < (score - preScore)) {

    for (i = 0; i < NODE_N; i++) {
      for (j = 0; j < NODE_N; j++) {
        preOrders[i][j] = orders[i][j];
        preGraph[i][j] = graph[i][j];
      }
    }
    preScore = score;

    return 1;
  }

  return 0;
}

void genScore() {
}

void Pre_logGamma() {

  LG = (float*) malloc ((DATA_N + 2) * sizeof(float));

  LG[1] = log(1.0);
  float i;
  for (i = 2; i <= DATA_N + 1; i++) {
    LG[(int)i] = LG[(int)i - 1] + log((float)i);
  }
}

void incr(int *bit, int n) {

  bit[n]++;
  if (bit[n] >= 2) {
    bit[n] = 0;
    incr(bit, n + 1);
  }

  return;
}

void incrS(int *bit, int n) {

  bit[n]++;
  if (bit[n] >= STATE_N) {
    bit[n] = 0;
    incr(bit, n + 1);
  }

  return;
}

bool getState(int parN, int *state, int time) {
  int j = 1;

  j = pow(STATE_N, (float)parN) - 1;

  if (time > j)
    return false;

  if (time >= 1)
    incrS(state, 0);

  return true;
}

int findindex(int *arr, int size) { // reminder: arr[0] has to be 0 && size ==
  // array size-1 && index start from 0
  int i, j, index = 0;

  for (i = 1; i < size; i++) {
    index += C(NODE_N - 1, i);
  }

  for (i = 1; i <= size - 1; i++) {
    for (j = arr[i - 1] + 1; j <= arr[i] - 1; j++) {
      index += C(NODE_N - 1 - j, size - i);
    }
  }

  index += arr[size] - arr[size - 1];

  return index;
}

int C(int n, int a) {
  int i, res = 1, atmp = a;

  for (i = 0; i < atmp; i++) {
    res *= n;
    n--;
  }

  for (i = 0; i < atmp; i++) {
    res /= a;
    a--;
  }

  return res;
}
