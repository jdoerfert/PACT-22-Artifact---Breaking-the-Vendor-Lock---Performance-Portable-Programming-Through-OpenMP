#include "hip/hip_runtime.h"
#include <iostream>
#include <limits>
#include <cmath>
#include <hip/hip_runtime.h>

using namespace std;

#define BLOCK_SIZE 16

/**
 * Each element of the product matrix c[i][j] is computed from a unique row and
 * column of the factor matrices, a[i][k] and b[k][j]
 */

// Matrix size constants.
constexpr int m_size = 768 * 8;  // Must be a multiple of 8.
constexpr int M = m_size / 8;
constexpr int N = m_size / 4;
constexpr int P = m_size / 2;

#include "verify.cpp"

__global__ 
void hellinger(
  const float *__restrict a, 
  const float *__restrict b, 
        float *__restrict c, 
  const int m, const int n, const int k)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if( col < k && row < m)
    {
        float sum = 0;
        for(int i = 0; i < n; i++)
        {
            sum += sqrtf(a[row * n + i] * b[i * k + col]);
        }
        const float value = 1.f - sum;
        const float gate = (!signbit(value));
        c[row * k + col] = sqrtf(gate * value);
    }
}

int main() {
  int i, j;

  // 2D arrays on host side.
  float(*a_host)[N] = new float[M][N];
  float(*b_host)[P] = new float[N][P];
  // host-side cpu result
  float(*c_host)[P] = new float[M][P];
  // host-side gpu result
  float(*c_back)[P] = new float[M][P];

  for (i = 0; i < M; i++)
    for (j = 0; j < N; j++)
      a_host[i][j] = 1.f / N;

  srand(123);
  for (i = 0; i < N; i++)
    for (j = 0; j < P; j++)
      b_host[i][j] = rand() % 256;

  for (j = 0; j < P; j++) { 
    float sum = 0;
    for (i = 0; i < N; i++)
      sum += b_host[i][j];
    for (i = 0; i < N; i++)
      b_host[i][j] /= sum;
  }

  float *a_device, *b_device, *c_device;

  hipMalloc((void **) &a_device, sizeof(float)*M*N);
  hipMalloc((void **) &b_device, sizeof(float)*N*P);
  hipMalloc((void **) &c_device, sizeof(float)*M*P);

  hipMemcpy(a_device, a_host, sizeof(float)*M*N, hipMemcpyHostToDevice);
  hipMemcpy(b_device, b_host, sizeof(float)*N*P, hipMemcpyHostToDevice);

  unsigned int grid_cols = (P + BLOCK_SIZE - 1) / BLOCK_SIZE;
  unsigned int grid_rows = (M + BLOCK_SIZE - 1) / BLOCK_SIZE;
  dim3 dimGrid(grid_cols, grid_rows);
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

  for (int i = 0; i < 100; i++)
    hipLaunchKernelGGL(hellinger, dimGrid, dimBlock, 0, 0, a_device, b_device, c_device, M, N, P);

  hipMemcpy(c_back, c_device, sizeof(int)*M*P, hipMemcpyDeviceToHost);

  cout << "Problem size: c(" << M << "," << P << ") = a(" << M << "," << N
       << ") * b(" << N << "," << P << ")\n";

#ifdef VERIFY
  VerifyResult(a_host, b_host, c_host, c_back);
#endif

  delete[] a_host;
  delete[] b_host;
  delete[] c_host;
  delete[] c_back;
  hipFree(a_device);
  hipFree(b_device);
  hipFree(c_device);
  return 0;
}

