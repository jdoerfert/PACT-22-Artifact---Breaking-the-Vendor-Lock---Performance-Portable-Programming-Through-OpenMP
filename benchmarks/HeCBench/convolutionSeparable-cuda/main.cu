#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "conv.h"

int main(int argc, char **argv)
{
  const unsigned int imageW = 3072;
  const unsigned int imageH = 3072;

  float* h_Kernel    = (float*)malloc(KERNEL_LENGTH * sizeof(float));
  float* h_Input     = (float*)malloc(imageW * imageH * sizeof(float));
  float* h_Buffer    = (float*)malloc(imageW * imageH * sizeof(float));
  float* h_OutputCPU = (float*)malloc(imageW * imageH * sizeof(float));
  float* h_OutputGPU = (float*)malloc(imageW * imageH * sizeof(float));

  srand(2009);
  for(unsigned int i = 0; i < KERNEL_LENGTH; i++)
    h_Kernel[i] = (float)(rand() % 16);

  for(unsigned int i = 0; i < imageW * imageH; i++)
    h_Input[i] = (float)(rand() % 16);


  float* d_Kernel;
  hipMalloc((void**)&d_Kernel, sizeof(float)*KERNEL_LENGTH);
  hipMemcpy(d_Kernel, h_Kernel, sizeof(float)*KERNEL_LENGTH, hipMemcpyHostToDevice);

  float* d_Input;
  hipMalloc((void**)&d_Input, sizeof(float)*imageW*imageH);
  hipMemcpy(d_Input, h_Input, sizeof(float)*imageW*imageH, hipMemcpyHostToDevice);

  float* d_Buffer;
  hipMalloc((void**)&d_Buffer, sizeof(float)*imageW*imageH);

  float* d_Output;
  hipMalloc((void**)&d_Output, sizeof(float)*imageW*imageH);


  //Just a single run or a warmup iteration
  convolutionRows(
      d_Buffer,
      d_Input,
      d_Kernel,
      imageW,
      imageH,
      imageW);

  hipMemcpy(h_Buffer, d_Buffer, sizeof(float)*imageW * imageH, hipMemcpyDeviceToHost);

  convolutionColumns(
      d_Output,
      d_Buffer,
      d_Kernel,
      imageW,
      imageH,
      imageW);

  const int numIterations = 100;

  for(int iter = 0; iter < numIterations; iter++){
    convolutionRows(
        d_Buffer,
        d_Input,
        d_Kernel,
        imageW,
        imageH,
        imageW);

    convolutionColumns(
        d_Output,
        d_Buffer,
        d_Kernel,
        imageW,
        imageH,
        imageW);
  }

  hipMemcpy(h_OutputGPU, d_Output, sizeof(float)*imageW * imageH, hipMemcpyDeviceToHost);

  printf("Comparing against Host/C++ computation...\n"); 
  convolutionRowHost(h_Buffer, h_Input, h_Kernel, imageW, imageH, KERNEL_RADIUS);
  convolutionColumnHost(h_OutputCPU, h_Buffer, h_Kernel, imageW, imageH, KERNEL_RADIUS);
  double sum = 0, delta = 0;
  double L2norm;
  for(unsigned int i = 0; i < imageW * imageH; i++){
    delta += (h_OutputCPU[i] - h_OutputGPU[i]) * (h_OutputCPU[i] - h_OutputGPU[i]);
    sum += h_OutputCPU[i] * h_OutputCPU[i];
  }
  L2norm = sqrt(delta / sum);
  printf("Relative L2 norm: %.3e\n\n", L2norm);

  free(h_OutputGPU);
  free(h_OutputCPU);
  free(h_Buffer);
  free(h_Input);
  free(h_Kernel);
  hipFree(d_Kernel);
  hipFree(d_Input);
  hipFree(d_Buffer);
  hipFree(d_Output);

  printf("%s\n", L2norm < 1e-6 ? "PASS" : "FAIL");
  return 0;
}
