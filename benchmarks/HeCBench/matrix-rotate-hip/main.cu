#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>

__global__ void rotate_matrix_parallel (float *matrix, const int n) {
  int layer = blockIdx.x * blockDim.x + threadIdx.x;
  if (layer < n/2) {
    int first = layer;
    int last = n - 1 - layer;
    for(int i = first; i < last; ++i) {
      int offset = i - first;

      float top = matrix[first*n+i]; // save top
      // left -> top
      matrix[first*n+i] = matrix[(last-offset)*n+first];

      // bottom -> left
      matrix[(last-offset)*n+first] = matrix[last*n+(last-offset)];

      // right -> bottom
      matrix[last*n+(last-offset)] = matrix[i*n+last];

      // top -> right
      matrix[i*n+last] = top; // right <- saved top
    }
  }
}

void rotate_matrix_serial(float *matrix, int n) {

  for (int layer = 0; layer < n / 2; ++layer) {
    int first = layer;
    int last = n - 1 - layer;
    for(int i = first; i < last; ++i) {
      int offset = i - first;
        float top = matrix[first*n+i]; // save top
        // left -> top
        matrix[first*n+i] = matrix[(last-offset)*n+first];

        // bottom -> left
        matrix[(last-offset)*n+first] = matrix[last*n+(last-offset)];

        // right -> bottom
        matrix[last*n+(last-offset)] = matrix[i*n+last];

        // top -> right
        matrix[i*n+last] = top; // right <- saved top
    }
  }
}

int main(int argc, char** argv) {

  const int n = atoi(argv[1]);
  float *serial_res = (float*) aligned_alloc(1024, n*n*sizeof(float));
  float *parallel_res = (float*) aligned_alloc(1024, n*n*sizeof(float));

  for (int i = 0; i < n; i++)
    for (int j = 0; j < n; j++)
      serial_res[i*n+j] = parallel_res[i*n+j] = i*n+j;

  float *d_parallel_res;
  hipMalloc((void**)&d_parallel_res, n*n*sizeof(float));
  hipMemcpy(d_parallel_res, parallel_res, n*n*sizeof(float), hipMemcpyHostToDevice);

  for (int i = 0; i < 100; i++) {
    rotate_matrix_serial(serial_res, n);
    hipLaunchKernelGGL(rotate_matrix_parallel, dim3((n/2+255)/256), dim3(256), 0, 0, d_parallel_res, n);
  }
  hipMemcpy(parallel_res, d_parallel_res, n*n*sizeof(float), hipMemcpyDeviceToHost);

  int errors = 0;
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      if (serial_res[i*n+j] != parallel_res[i*n+j]) {
        errors++; 
        break;
      }
    }
  }
  if (errors) 
    printf("fail\n");
  else 
    printf("success\n");

  free(serial_res);
  free(parallel_res);
  hipFree(d_parallel_res);
  return 0;
}

