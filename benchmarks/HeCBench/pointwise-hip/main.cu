#include "hip/hip_runtime.h"
/* Copyright (c) 1993-2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

// Define some error checking macros.
#define hipErrCheck(stat) { hipErrCheck_((stat), __FILE__, __LINE__); }
void hipErrCheck_(hipError_t stat, const char *file, int line) {
  if (stat != hipSuccess) {
    fprintf(stderr, "HIP Error: %s %s %d\n", hipGetErrorString(stat), file, line);
  }
}

typedef struct {
  double i, c, h;
} checksum;

// Device functions
__forceinline__ __device__ float sigmoidf(float in) {
  return 1.f / (1.f + expf(-in));  
}

// Fused kernel
__global__ 
void elementWise_fp(int hiddenSize, int miniBatch,
    const float *__restrict__ tmp_h, 
    const float *__restrict__ tmp_i, 
    const float *__restrict__ bias,
    float *__restrict__ linearGates,
    float *__restrict__ h_out,
    float *__restrict__ i_out,
    const float *__restrict__ c_in,
    float *__restrict__ c_out)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int numElements = miniBatch * hiddenSize;

  if (index >= numElements) return;

  int batch = index / hiddenSize;
  int gateIndex = (index % hiddenSize) + 4 * batch * hiddenSize;   

  float g[4];

  for (int i = 0; i < 4; i++) {
    g[i] = tmp_i[i * hiddenSize + gateIndex] + tmp_h[i * hiddenSize + gateIndex];
    g[i] += bias[i * hiddenSize + index % hiddenSize] + bias[(i + 4) * hiddenSize + index % hiddenSize];
    linearGates[gateIndex + i * hiddenSize] = g[i];
  }   

  float in_gate     = sigmoidf(g[0]);
  float forget_gate = sigmoidf(g[1]);
  float in_gate2    = tanhf(g[2]);
  float out_gate    = sigmoidf(g[3]);

  float val = (forget_gate * c_in[index]) + (in_gate * in_gate2);

  c_out[index] = val;

  val = out_gate * tanhf(val);                                   

  h_out[index] = val;
  i_out[index] = val;
}

__device__
float LCG_random(unsigned int * seed) {
  const unsigned int m = 2147483648;
  const unsigned int a = 26757677;
  const unsigned int c = 1;
  *seed = (a * (*seed) + c) % m;
  return (float) (*seed) / (float) m;
}

__global__
void init (float* data, int size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= size) return;
  unsigned int seed = index ^ size;
  data[index] = LCG_random(&seed);
}

void test(int hiddenSize, int miniBatch, int seqLength, int numLayers, checksum &cs) {
  float *h_data;
  float *i_data;
  float *c_data;
  float *bias;
  float *tmp_h;
  float *tmp_i;
  float *linearGates;

  // Input/output data
  int numElements = hiddenSize * miniBatch;

  int hc_size = (seqLength + 1) * (numLayers) * numElements;
  int i_size = (seqLength) * (numLayers + 1) * numElements;
  int bias_size = numLayers * hiddenSize * 8;
  int tmp_h_size = 4 * numLayers * numElements;
  int tmp_i_size = 4 * seqLength * numElements;

  hipErrCheck(hipMalloc((void**)&h_data, hc_size * sizeof(float)));
  hipErrCheck(hipMalloc((void**)&i_data, i_size * sizeof(float)));
  hipErrCheck(hipMalloc((void**)&c_data, hc_size * sizeof(float)));
  hipErrCheck(hipMalloc((void**)&bias, bias_size * sizeof(float)));

  // Workspace
  hipErrCheck(hipMalloc((void**)&tmp_h, tmp_h_size * sizeof(float)));
  hipErrCheck(hipMalloc((void**)&tmp_i, tmp_i_size * sizeof(float)));

  // Activations
  hipErrCheck(hipMalloc((void**)&linearGates, 4 * seqLength * numLayers * numElements * sizeof(float)));  

  // Initialise with random values on a device
  dim3 blocks (256);
  dim3 grids_hc ((hc_size + 255)/256);
  dim3 grids_b ((bias_size + 255)/256);
  dim3 grids_tmp_h ((tmp_h_size + 255)/256);
  dim3 grids_tmp_i ((tmp_i_size + 255)/256);
          
  hipLaunchKernelGGL(init, grids_tmp_h, blocks, 0, 0, tmp_h, tmp_h_size);
  hipLaunchKernelGGL(init, grids_tmp_i, blocks, 0, 0, tmp_i, tmp_i_size);
  hipLaunchKernelGGL(init, grids_hc, blocks, 0, 0, c_data, hc_size);
  hipLaunchKernelGGL(init, grids_b, blocks, 0, 0, bias, bias_size);

  hipDeviceSynchronize();

  int lStart = 0;
  int lEnd = 0;
  int rStart = 0;
  int rEnd = 0;
  int recurBatchSize = 2;

  dim3 grids_p ((numElements + 255)/256);
  
  while (true) {
    // Many layer "scheduling".
    if (lEnd == 0) {
      lStart = 0;
      lEnd = 1;
      rStart = 0;
    }
    else {
      // Move "up" and "left"
      lStart++;
      lEnd++;

      rStart -= recurBatchSize;

      // Over the top or off the left, reset to layer 0
      if (lEnd > numLayers || rStart < 0) {
        rStart += (lStart + 1) * recurBatchSize;

        lStart = 0;
        lEnd = 1;
      }

      // Off the right, step up
      while (rStart >= seqLength && lEnd <= numLayers) {
        lStart++;
        lEnd++;
        rStart -= recurBatchSize;
      }

      // Over the top or off the left, done!
      if (lEnd > numLayers || rStart < 0) {
        break;
      }
    }

    rEnd = rStart + recurBatchSize;
    if (rEnd > seqLength) rEnd = seqLength;

    for (int layer = lStart; layer < lEnd; layer++) {         
      for (int i = rStart; i < rEnd; i++)
        hipLaunchKernelGGL(elementWise_fp, grids_p, blocks, 0, 0, hiddenSize, miniBatch,
         tmp_h + 4 * layer * numElements, 
         tmp_i + 4 * i * numElements, 
         bias + 8 * layer * hiddenSize,
         linearGates + 4 * (i * numElements + layer * seqLength * numElements),
         h_data + (i + 1) * numElements + layer * (seqLength + 1) * numElements,
         i_data + i * numElements + (layer + 1) * seqLength * numElements,
         c_data + i * numElements + layer * (seqLength + 1) * numElements,
         c_data + (i + 1) * numElements + layer * (seqLength + 1) * numElements);
      hipErrCheck(hipGetLastError());
    }
  }

  float *testOutputi = (float*)malloc(numElements * seqLength * sizeof(float));
  float *testOutputh = (float*)malloc(numElements * numLayers * sizeof(float));
  float *testOutputc = (float*)malloc(numElements * numLayers * sizeof(float));

  hipDeviceSynchronize();
  
  hipErrCheck(hipMemcpy(testOutputi, i_data + numLayers * seqLength * numElements, 
    seqLength * numElements * sizeof(float), hipMemcpyDeviceToHost));
  for (int layer = 0; layer < numLayers; layer++) {
    hipErrCheck(hipMemcpy(testOutputh + layer * numElements, 
      h_data + seqLength * numElements + layer * (seqLength + 1) * numElements, 
      numElements * sizeof(float), hipMemcpyDeviceToHost));
    hipErrCheck(hipMemcpy(testOutputc + layer * numElements, 
      c_data + seqLength * numElements + layer * (seqLength + 1) * numElements, 
      numElements * sizeof(float), hipMemcpyDeviceToHost));
  }

  double checksumi = 0.;
  double checksumh = 0.;
  double checksumc = 0.;

  for (int m = 0; m < miniBatch; m++) {
    for (int j = 0; j < seqLength; j++) {
      for (int i = 0; i < hiddenSize; i++) {
        checksumi += testOutputi[j * numElements + m * hiddenSize + i];
        //if (hiddenSize <= 8) printf("i: (%d,%d): %E\n", j, i, testOutputi[j * numElements + m * hiddenSize + i]);
      }
    }
    for (int j = 0; j < numLayers; j++) {
      for (int i = 0; i < hiddenSize; i++) {         
        checksumh += testOutputh[j * numElements + m * hiddenSize + i];
        checksumc += testOutputc[j * numElements + m * hiddenSize + i];
      }
    }
  }

  free(testOutputi);
  free(testOutputc);
  free(testOutputh);

  hipErrCheck(hipFree(h_data));
  hipErrCheck(hipFree(i_data));  
  hipErrCheck(hipFree(c_data));  

  hipErrCheck(hipFree(bias));
  hipErrCheck(hipFree(tmp_h));
  hipErrCheck(hipFree(tmp_i));
  hipErrCheck(hipFree(linearGates));

  cs.i = checksumi;
  cs.c = checksumc;
  cs.h = checksumh;
}


int main(int argc, char* argv[]) {
  int seqLength;
  int numLayers;
  int hiddenSize;
  int miniBatch; 

  if (argc == 5) {
    seqLength = atoi(argv[1]);
    numLayers = atoi(argv[2]);
    hiddenSize = atoi(argv[3]);
    miniBatch = atoi(argv[4]);   
  }
  else if (argc == 1) {
    printf("Running with default settings\n");
    seqLength = 100;
    numLayers = 4;
    hiddenSize = 512;
    miniBatch = 64;
  }
  else {
    printf("Usage: ./%s <seqLength> <numLayers> <hiddenSize> <miniBatch>\n", argv[1]);
    return 1;      
  }

  printf("seqLength %d, numLayers %d, hiddenSize %d, miniBatch %d\n",
         seqLength, numLayers, hiddenSize, miniBatch);  

  int numRuns = 100;
  checksum cs;
  
  for (int run = 0; run < numRuns; run++) {
    test(hiddenSize, miniBatch, seqLength, numLayers, cs);
  }

  printf("i checksum %E     ", cs.i);
  printf("c checksum %E     ", cs.c);
  printf("h checksum %E\n", cs.h);


  return 0;
}

