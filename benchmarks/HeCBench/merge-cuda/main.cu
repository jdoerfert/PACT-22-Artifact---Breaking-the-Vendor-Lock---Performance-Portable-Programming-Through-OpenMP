#include <stdio.h>
#include <stdint.h>
#include <limits.h>
#include <stdlib.h>
#include <float.h>
#include <iostream>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include "kernels.h"

#define CSV 0
#if(CSV)
#define PS(X, S) std::cout << X << ", " << S << ", "; fflush(stdout);
#define PV(X) std::cout << X << ", "; fflush(stdout);
#else
#define PS(X, S) std::cout << X << " " << S <<" :\n"; fflush(stdout);
#define PV(X) std::cout << "\t" << #X << " \t: " << X << "\n"; fflush(stdout);
#endif

/*
 * Produce 64-bits of pseudo-randomness
 * Note: not very "good" or "random" 
 */
template<typename vec_t>
vec_t rand64() {
  vec_t rtn;
  do {
    uint32_t * rtn32 = (uint32_t *)&rtn;
    rtn32[0] = rand();
    if(sizeof(vec_t) > 4) rtn32[1] = rand();
  } while(!(rtn < getPositiveInfinity<vec_t>() &&
        rtn > getNegativeInfinity<vec_t>()));
  return rtn;
}

/*
 * Perform <runs> merges of two sorted pseudorandom <vec_t> arrays of length <size> 
 * Checks the output of each merge for correctness
 */
#define PADDING 1024
template<typename vec_t, uint32_t blocks, uint32_t threads, uint32_t runs>
void mergeType(uint64_t size) {
  // Prepare host and device vectors
  std::vector<vec_t> hA (size + PADDING);
  std::vector<vec_t> hB (size + PADDING);
  std::vector<vec_t> hC (2*size + PADDING);

  vec_t *dA;
  vec_t *dB;
  vec_t *dC;

  hipMalloc((void**)&dA, (size + PADDING) * sizeof(vec_t));
  hipMalloc((void**)&dB, (size + PADDING) * sizeof(vec_t));
  hipMalloc((void**)&dC, (2*size + PADDING) * sizeof(vec_t));

  uint32_t *dpi; // diagonal_path_intersections;
  hipMalloc((void**)&dpi, (2 * (blocks + 1)) * sizeof(uint32_t));

  uint32_t errors = 0;

  for(uint32_t i = 0; i < runs; i++) {

    // Generate two sorted psuedorandom arrays
    for (uint64_t n = 0; n < size; n++) {
       hA[n] = rand64<vec_t>();
       hB[n] = rand64<vec_t>();
    }

    for (uint64_t n = size; n < size + PADDING; n++) {
      hA[n] = getPositiveInfinity<vec_t>();
      hB[n] = getPositiveInfinity<vec_t>();
    }

    std::sort(hA.begin(), hA.end());
    std::sort(hB.begin(), hB.end());

    hipMemcpy(dA, hA.data(), (size + PADDING) * sizeof(vec_t), hipMemcpyHostToDevice);
    hipMemcpy(dB, hB.data(), (size + PADDING) * sizeof(vec_t), hipMemcpyHostToDevice);

    // Perform the global diagonal intersection serach to divide work among SMs
    workloadDiagonals<vec_t><<<blocks, 32>>> (dA, size, dB, size, dpi);

    // Merge between global diagonals independently on each block
    mergeSinglePath<vec_t,false,false> <<<blocks, threads>>>
        (dA, size, dB, size, dpi, dC, size * 2);

    // Test for errors
    hipMemcpy(hC.data(), dC, size * sizeof(vec_t), hipMemcpyDeviceToHost);
    for(uint32_t i = 1; i < size; i++) {
      errors += hC[i] < hC[i-1];
    }
  }

  // Print error info
  PV(errors);

  hipFree(dA);
  hipFree(dB);
  hipFree(dC);
  hipFree(dpi);
}

/* 
 * Performs <runs> merge tests for each type at a given size
 */
template<uint32_t blocks, uint32_t threads, uint32_t runs>
void mergeAllTypes(uint64_t size) {
  PS("uint32_t", size)  mergeType<uint32_t, blocks, threads, runs>(size); printf("\n");
  PS("float",    size)  mergeType<float,    blocks, threads, runs>(size); printf("\n");
  PS("uint64_t", size)  mergeType<uint64_t, blocks, threads, runs>(size); printf("\n");
  PS("double", size)    mergeType<double,   blocks, threads, runs>(size); printf("\n");
}

int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("Usage: %s <length of the arrays>\n", argv[0]);
    return 1;
  }
  // length is sufficiently large; 
  // otherwise there are invalid global reads in the kernel mergeSinglePath
  const uint64_t length = atol(argv[1]);

  const int blocks = 112;
  const int threads = 128;  // do not change
  const int runs = 100;
  mergeAllTypes<blocks, threads, runs>(length);
  return 0;
}

