#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "reference.cpp"

__global__ void smoothingFilter(
    int Lx, int Ly, 
    int Threshold, int MaxRad, 
    const float*__restrict Img,
            int*__restrict Box,
          float*__restrict Norm)
{
  int tid = threadIdx.x;
  int tjd = threadIdx.y;
  int i = blockIdx.x * blockDim.x + tid;
  int j = blockIdx.y * blockDim.y + tjd;
  int stid = tjd * blockDim.x + tid;
  int gtid = j * Lx + i;  

  // part of shared memory may be unused
  __shared__ float s_Img[1024];

  if ( i < Lx && j < Ly )
    s_Img[stid] = Img[gtid];

  __syncthreads();

  if ( i < Lx && j < Ly )
  {
    // Smoothing parameters
    float sum = 0.f;
    int q = 1;
    int s = q;
    int ksum = 0;

    // Continue until parameters are met
    while (sum < Threshold && q < MaxRad)
    {
      s = q;
      sum = 0.f;
      ksum = 0;

      // Normal adaptive smoothing
      for (int ii = -s; ii < s+1; ii++)
        for (int jj = -s; jj < s+1; jj++)
          if ( (i-s >= 0) && (i+s < Ly) && (j-s >= 0) && (j+s < Lx) )
          {
            ksum++;
            // Compute within bounds of block dimensions
            if( tid-s >= 0 && tid+s < blockDim.x && tjd-s >= 0 && tjd+s < blockDim.y )
              sum += s_Img[stid + ii*blockDim.x + jj];
            // Compute block borders with global memory
            else
              sum += Img[gtid + ii*Lx + jj];
          }
      q++;
    }
    Box[gtid] = s;

    // Normalization for each box
    for (int ii = -s; ii < s+1; ii++)
      for (int jj = -s; jj < s+1; jj++)
        if (ksum != 0) 
          atomicAdd(&Norm[gtid + ii*Lx + jj], __fdividef(1.f, (float)ksum));
  }
}

__global__ void normalizeFilter(int Lx, int Ly, float*__restrict Img, const float*__restrict Norm)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if ( i < Lx && j < Ly ) {
    int gtid = j * Lx + i;  
    const float norm = Norm[gtid];
    if (norm != 0) Img[gtid] = __fdividef(Img[gtid], norm);
  }
}

__global__ void outFilter( 
    int Lx, int Ly,
    const float*__restrict Img,
    const   int*__restrict Box,
          float*__restrict Out )
{
  int tid = threadIdx.x;
  int tjd = threadIdx.y;
  int i = blockIdx.x * blockDim.x + tid;
  int j = blockIdx.y * blockDim.y + tjd;
  int stid = tjd * blockDim.x + tid;
  int gtid = j * Lx + i;  

  // part of shared memory may be unused
  __shared__ float s_Img[1024];

  if ( i < Lx && j < Ly )
    s_Img[stid] = Img[gtid];

  __syncthreads();

  if ( i < Lx && j < Ly )
  {
    const int s = Box[gtid];
    float sum = 0.f;
    int ksum  = 0;

    for (int ii = -s; ii < s+1; ii++)
      for (int jj = -s; jj < s+1; jj++)
        if ( (i-s >= 0) && (i+s < Lx) && (j-s >= 0) && (j+s < Ly) )
        {
          ksum++;
          if( tid-s >= 0 && tid+s < blockDim.x && tjd-s >= 0 && tjd+s < blockDim.y )
            sum += s_Img[stid + ii*blockDim.y + jj];
          else
            sum += Img[gtid + ii*Ly + jj];
        }
    if ( ksum != 0 ) Out[gtid] = __fdividef(sum , (float)ksum);
  }
}

int main(int argc, char* argv[]) {
  if (argc != 5) {
     printf("./%s <image dimension> <threshold> <max box size> <iterations>\n", argv[0]);
     exit(1);
  }

  // only a square image is supported
  const int Lx = atoi(argv[1]);
  const int Ly = Lx;
  const int size = Lx * Ly;

  const int Threshold = atoi(argv[2]);
  const int MaxRad = atoi(argv[3]);
  const int repeat = atoi(argv[4]);
 
  // input image
  float *img = (float*) malloc (sizeof(float) * size);

  // host and device results
  float *norm = (float*) malloc (sizeof(float) * size);
  float *h_norm = (float*) malloc (sizeof(float) * size);

  int *box = (int*) malloc (sizeof(int) * size);
  int *h_box = (int*) malloc (sizeof(int) * size);

  float *out = (float*) malloc (sizeof(float) * size);
  float *h_out = (float*) malloc (sizeof(float) * size);

  srand(123);
  for (int i = 0; i < size; i++) {
    img[i] = rand() % 256;
    norm[i] = box[i] = out[i] = 0;
  }

  float *d_img;
  hipMalloc((void**)&d_img, sizeof(float) * size);

  float *d_norm;
  hipMalloc((void**)&d_norm, sizeof(float) * size);

  int *d_box;
  hipMalloc((void**)&d_box, sizeof(int) * size);

  float *d_out;
  hipMalloc((void**)&d_out, sizeof(float) * size);

  dim3 grids ((Lx+15)/16, (Ly+15)/16);
  dim3 blocks (16, 16);

  // reset output
  hipMemcpy(d_out, out, sizeof(float) * size, hipMemcpyHostToDevice);

  for (int i = 0; i < repeat; i++) {
    // restore input image
    hipMemcpy(d_img, img, sizeof(float) * size, hipMemcpyHostToDevice);
    // reset norm
    hipMemcpy(d_norm, norm, sizeof(float) * size, hipMemcpyHostToDevice);
    // launch three kernels
    hipLaunchKernelGGL(smoothingFilter, grids, blocks, 0, 0, Lx, Ly, Threshold, MaxRad, d_img, d_box, d_norm);
    hipLaunchKernelGGL(normalizeFilter, grids, blocks, 0, 0, Lx, Ly, d_img, d_norm);
    hipLaunchKernelGGL(outFilter, grids, blocks, 0, 0, Lx, Ly, d_img, d_box, d_out);
  }

  hipMemcpy(out, d_out, sizeof(float) * size, hipMemcpyDeviceToHost);
  hipMemcpy(box, d_box, sizeof(int) * size, hipMemcpyDeviceToHost);
  hipMemcpy(norm, d_norm, sizeof(float) * size, hipMemcpyDeviceToHost);

  // verify
  reference (Lx, Ly, Threshold, MaxRad, img, h_box, h_norm, h_out);
  verify(size, MaxRad, norm, h_norm, out, h_out, box, h_box);

  hipFree(d_img);
  hipFree(d_norm);
  hipFree(d_box);
  hipFree(d_out);
  free(img);
  free(norm);
  free(h_norm);
  free(box);
  free(h_box);
  free(out);
  free(h_out);
  return 0;
}
