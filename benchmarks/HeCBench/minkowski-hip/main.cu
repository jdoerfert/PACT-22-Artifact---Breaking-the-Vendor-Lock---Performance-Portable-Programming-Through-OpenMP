#include "hip/hip_runtime.h"
#include <iostream>
#include <limits>
#include <cmath>
#include <hip/hip_runtime.h>

using namespace std;

#define BLOCK_SIZE 16

/**
 * Each element of the product matrix c[i][j] is computed from a unique row and
 * column of the factor matrices, a[i][k] and b[k][j]
 */

// Matrix size constants.
constexpr int m_size = 512 * 8;  // Must be a multiple of 8.
constexpr int M = m_size / 8;
constexpr int N = m_size / 4;
constexpr int P = m_size / 2;

#include "verify.cpp"

__global__ 
void minkowski(
  const float *__restrict a, 
  const float *__restrict b, 
        float *__restrict c, 
  const float p,
  const float one_over_p,
  const int m, const int n, const int k)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if( col < k && row < m)
    {
        float sum = 0;
        for(int i = 0; i < n; i++)
        {
            sum += powf(fabsf(a[row * n + i] - b[i * k + col]), p);
        }
        c[row * k + col] = powf(sum, one_over_p);
    }
}

int main() {
  int i, j;

  // 2D arrays on host side.
  float(*a_host)[N] = new float[M][N];
  float(*b_host)[P] = new float[N][P];
  // host-side cpu result
  float(*c_host)[P] = new float[M][P];
  // host-side gpu result
  float(*c_back)[P] = new float[M][P];

  for (i = 0; i < M; i++)
    for (j = 0; j < N; j++)
      a_host[i][j] = 1.f / N;

  srand(123);
  for (i = 0; i < N; i++)
    for (j = 0; j < P; j++)
      b_host[i][j] = rand() % 256;

  for (j = 0; j < P; j++) { 
    float sum = 0;
    for (i = 0; i < N; i++)
      sum += b_host[i][j];
    for (i = 0; i < N; i++)
      b_host[i][j] /= sum;
  }

  float *a_device, *b_device, *c_device;

  hipMalloc((void **) &a_device, sizeof(float)*M*N);
  hipMalloc((void **) &b_device, sizeof(float)*N*P);
  hipMalloc((void **) &c_device, sizeof(float)*M*P);

  hipMemcpy(a_device, a_host, sizeof(float)*M*N, hipMemcpyHostToDevice);
  hipMemcpy(b_device, b_host, sizeof(float)*N*P, hipMemcpyHostToDevice);

  unsigned int grid_cols = (P + BLOCK_SIZE - 1) / BLOCK_SIZE;
  unsigned int grid_rows = (M + BLOCK_SIZE - 1) / BLOCK_SIZE;
  dim3 dimGrid(grid_cols, grid_rows);
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

  cout << "Problem size: c(" << M << "," << P << ") = a(" << M << "," << N
       << ") * b(" << N << "," << P << ")\n";

  for (int k = 1; k <= 4; k++) {
    printf("Minkowski distance with p = %d\n", k);
    const float p = (float)k;
    const float one_over_p = 1.f / p;

    for (int i = 0; i < 100; i++)
      hipLaunchKernelGGL(minkowski, dimGrid, dimBlock, 0, 0, a_device, b_device, c_device, p, one_over_p, M, N, P);
    hipMemcpy(c_back, c_device, sizeof(int)*M*P, hipMemcpyDeviceToHost);

    #ifdef VERIFY
    VerifyResult(a_host, b_host, c_host, c_back, p, one_over_p);
    #endif
  }

  delete[] a_host;
  delete[] b_host;
  delete[] c_host;
  delete[] c_back;
  hipFree(a_device);
  hipFree(b_device);
  hipFree(c_device);
  return 0;
}

