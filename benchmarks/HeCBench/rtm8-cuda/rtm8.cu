#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <hip/hip_runtime.h>

#define nt 30
#define nx 680
#define ny 134
#define nz 450

#include "mysecond.c"

inline __host__ __device__ int indexTo1D(int x, int y, int z){
  return x + y*nx + z*nx*ny;
}

void rtm8_cpu(float* vsq, float* current_s, float* current_r, float* next_s, float* next_r, float* image, float* a, size_t N)
{
#ifdef _OPENMP
  #pragma omp parallel for collapse(3)
#endif
  for (int z = 4; z < nz - 4; z++) {
    for (int y = 4; y < ny - 4; y++) {
      for (int x = 4; x < nx - 4; x++) {
        float div =
          a[0] * current_s[indexTo1D(x,y,z)] +
          a[1] * (current_s[indexTo1D(x+1,y,z)] + current_s[indexTo1D(x-1,y,z)] +
              current_s[indexTo1D(x,y+1,z)] + current_s[indexTo1D(x,y-1,z)] +
              current_s[indexTo1D(x,y,z+1)] + current_s[indexTo1D(x,y,z-1)]) +
          a[2] * (current_s[indexTo1D(x+2,y,z)] + current_s[indexTo1D(x-2,y,z)] +
              current_s[indexTo1D(x,y+2,z)] + current_s[indexTo1D(x,y-2,z)] +
              current_s[indexTo1D(x,y,z+2)] + current_s[indexTo1D(x,y,z-2)]) +
          a[3] * (current_s[indexTo1D(x+3,y,z)] + current_s[indexTo1D(x-3,y,z)] +
              current_s[indexTo1D(x,y+3,z)] + current_s[indexTo1D(x,y-3,z)] +
              current_s[indexTo1D(x,y,z+3)] + current_s[indexTo1D(x,y,z-3)]) +
          a[4] * (current_s[indexTo1D(x+4,y,z)] + current_s[indexTo1D(x-4,y,z)] +
              current_s[indexTo1D(x,y+4,z)] + current_s[indexTo1D(x,y-4,z)] +
              current_s[indexTo1D(x,y,z+4)] + current_s[indexTo1D(x,y,z-4)]);

        next_s[indexTo1D(x,y,z)] = 2*current_s[indexTo1D(x,y,z)] - next_s[indexTo1D(x,y,z)]
          + vsq[indexTo1D(x,y,z)]*div;
        div =
          a[0] * current_r[indexTo1D(x,y,z)] +
          a[1] * (current_r[indexTo1D(x+1,y,z)] + current_r[indexTo1D(x-1,y,z)] +
              current_r[indexTo1D(x,y+1,z)] + current_r[indexTo1D(x,y-1,z)] +
              current_r[indexTo1D(x,y,z+1)] + current_r[indexTo1D(x,y,z-1)]) +
          a[2] * (current_r[indexTo1D(x+2,y,z)] + current_r[indexTo1D(x-2,y,z)] +
              current_r[indexTo1D(x,y+2,z)] + current_r[indexTo1D(x,y-2,z)] +
              current_r[indexTo1D(x,y,z+2)] + current_r[indexTo1D(x,y,z-2)]) +
          a[3] * (current_r[indexTo1D(x+3,y,z)] + current_r[indexTo1D(x-3,y,z)] +
              current_r[indexTo1D(x,y+3,z)] + current_r[indexTo1D(x,y-3,z)] +
              current_r[indexTo1D(x,y,z+3)] + current_r[indexTo1D(x,y,z-3)]) +
          a[4] * (current_r[indexTo1D(x+4,y,z)] + current_r[indexTo1D(x-4,y,z)] +
              current_r[indexTo1D(x,y+4,z)] + current_r[indexTo1D(x,y-4,z)] +
              current_r[indexTo1D(x,y,z+4)] + current_r[indexTo1D(x,y,z-4)]);

        next_r[indexTo1D(x,y,z)] = 2 * current_r[indexTo1D(x,y,z)]
          - next_r[indexTo1D(x,y,z)] + vsq[indexTo1D(x,y,z)] * div;

        image[indexTo1D(x,y,z)] = next_s[indexTo1D(x,y,z)] * next_r[indexTo1D(x,y,z)];
      }
    }
  }
}
  

 __global__ void
rtm8(float* vsq, float* current_s, float* current_r, float* next_s, float* next_r, float* image, float* a, size_t N)
{
  unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned y = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned z = blockIdx.z * blockDim.z + threadIdx.z;
  float div;
  if ((4 <= x && x < (nx - 4) ) && (4 <= y && y < (ny - 4)) && (4 <= z && z < (nz - 4))){
    div =
      a[0] * current_s[indexTo1D(x,y,z)] +
      a[1] * (current_s[indexTo1D(x+1,y,z)] + current_s[indexTo1D(x-1,y,z)] +
          current_s[indexTo1D(x,y+1,z)] + current_s[indexTo1D(x,y-1,z)] +
          current_s[indexTo1D(x,y,z+1)] + current_s[indexTo1D(x,y,z-1)]) +
      a[2] * (current_s[indexTo1D(x+2,y,z)] + current_s[indexTo1D(x-2,y,z)] +
          current_s[indexTo1D(x,y+2,z)] + current_s[indexTo1D(x,y-2,z)] +
          current_s[indexTo1D(x,y,z+2)] + current_s[indexTo1D(x,y,z-2)]) +
      a[3] * (current_s[indexTo1D(x+3,y,z)] + current_s[indexTo1D(x-3,y,z)] +
          current_s[indexTo1D(x,y+3,z)] + current_s[indexTo1D(x,y-3,z)] +
          current_s[indexTo1D(x,y,z+3)] + current_s[indexTo1D(x,y,z-3)]) +
      a[4] * (current_s[indexTo1D(x+4,y,z)] + current_s[indexTo1D(x-4,y,z)] +
          current_s[indexTo1D(x,y+4,z)] + current_s[indexTo1D(x,y-4,z)] +
          current_s[indexTo1D(x,y,z+4)] + current_s[indexTo1D(x,y,z-4)]);

    next_s[indexTo1D(x,y,z)] = 2*current_s[indexTo1D(x,y,z)] - next_s[indexTo1D(x,y,z)]
      + vsq[indexTo1D(x,y,z)]*div;
    div =
      a[0] * current_r[indexTo1D(x,y,z)] +
      a[1] * (current_r[indexTo1D(x+1,y,z)] + current_r[indexTo1D(x-1,y,z)] +
          current_r[indexTo1D(x,y+1,z)] + current_r[indexTo1D(x,y-1,z)] +
          current_r[indexTo1D(x,y,z+1)] + current_r[indexTo1D(x,y,z-1)]) +
      a[2] * (current_r[indexTo1D(x+2,y,z)] + current_r[indexTo1D(x-2,y,z)] +
          current_r[indexTo1D(x,y+2,z)] + current_r[indexTo1D(x,y-2,z)] +
          current_r[indexTo1D(x,y,z+2)] + current_r[indexTo1D(x,y,z-2)]) +
      a[3] * (current_r[indexTo1D(x+3,y,z)] + current_r[indexTo1D(x-3,y,z)] +
          current_r[indexTo1D(x,y+3,z)] + current_r[indexTo1D(x,y-3,z)] +
          current_r[indexTo1D(x,y,z+3)] + current_r[indexTo1D(x,y,z-3)]) +
      a[4] * (current_r[indexTo1D(x+4,y,z)] + current_r[indexTo1D(x-4,y,z)] +
          current_r[indexTo1D(x,y+4,z)] + current_r[indexTo1D(x,y-4,z)] +
          current_r[indexTo1D(x,y,z+4)] + current_r[indexTo1D(x,y,z-4)]);

    next_r[indexTo1D(x,y,z)] = 2 * current_r[indexTo1D(x,y,z)]
      - next_r[indexTo1D(x,y,z)] + vsq[indexTo1D(x,y,z)] * div;

    image[indexTo1D(x,y,z)] = next_s[indexTo1D(x,y,z)] * next_r[indexTo1D(x,y,z)];
  }
}


int main() {
  const int ArraySize = nx * ny * nz;
  float* next_s = (float*)malloc(ArraySize * sizeof(float));
  float* current_s = (float*)malloc(ArraySize * sizeof(float));
  float* next_r = (float*)malloc(ArraySize * sizeof(float));
  float* current_r = (float*)malloc(ArraySize * sizeof(float));
  float* vsq = (float*)malloc(ArraySize * sizeof(float));
  float* image_gpu = (float*)malloc(ArraySize * sizeof(float));
  float* image_cpu = (float*)malloc(ArraySize * sizeof(float));

  float a[5];
  double pts, t0, t1, dt, flops, pt_rate, flop_rate, speedup, memory;

  memory = ArraySize*sizeof(float)*6;
  pts = nt;
  pts = pts*(nx-8)*(ny-8)*(nz-8);
  flops = 67*pts;
  printf("memory (MB) = %f\n", memory/1e6);
  printf("pts (billions) = %f\n", pts/1e9);
  printf("Tflops = %f\n", flops/1e12);

  // Initialization of matrix
  a[0] = -1./560.;
  a[1] = 8./315;
  a[2] = -0.2;
  a[3] = 1.6;
  a[4] = -1435./504.;

  for (int z = 0; z < nz; z++) {
    for (int y = 0; y < ny; y++) {
      for (int x = 0; x < nx; x++) {
        vsq[indexTo1D(x,y,z)] = 1.0;
        next_s[indexTo1D(x,y,z)] = 0;
        current_s[indexTo1D(x,y,z)] = 1.0;
        next_r[indexTo1D(x,y,z)] = 0;
        current_r[indexTo1D(x,y,z)] = 1.0;
        image_gpu[indexTo1D(x,y,z)] = image_cpu[indexTo1D(x,y,z)] = 0.5;
      }
    }
  }

  t0 = mysecond();
  //allocate and copy matrix to device
  float* vsq_d;
  float* next_s_d;
  float* current_s_d;
  float* next_r_d;
  float* current_r_d;
  float* image_d;
  float* a_d;

  hipMalloc(&vsq_d, ArraySize * sizeof(float));
  hipMalloc(&next_s_d, ArraySize * sizeof(float));
  hipMalloc(&current_s_d, ArraySize * sizeof(float));
  hipMalloc(&next_r_d, ArraySize * sizeof(float));
  hipMalloc(&current_r_d, ArraySize * sizeof(float));
  hipMalloc(&image_d, ArraySize * sizeof(float));
  hipMalloc(&a_d, 5 * sizeof(float));

  hipMemcpy(vsq_d, vsq, ArraySize * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(next_s_d, next_s, ArraySize * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(current_s_d, current_s, ArraySize * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(next_r_d, next_r, ArraySize * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(current_r_d, current_r, ArraySize * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(image_d, image_gpu, ArraySize * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(a_d, a, 5 * sizeof(float), hipMemcpyHostToDevice);

  int groupSize = 16;
  int nx_pad = (nx + groupSize - 1) / groupSize ;
  int ny_pad = (ny + groupSize - 1) / groupSize ;
  int nz_pad = nz;

  // Launch the kernel nt times
  for (int t = 0; t < nt; t++) {
    rtm8<<<dim3(nx_pad, ny_pad, nz_pad), dim3(groupSize, groupSize, 1)>>>(
        vsq_d, current_s_d, next_s_d, current_r_d, next_r_d, image_d, a_d, ArraySize);
  }

  //copy back image value
  hipMemcpy(image_gpu, image_d, ArraySize * sizeof(float), hipMemcpyDeviceToHost);
  t1 = mysecond();
  dt = t1 - t0;

  t0 = mysecond();
  for (int t = 0; t < nt; t++) {
    rtm8_cpu(vsq, current_s, next_s, current_r, next_r, image_cpu, a, ArraySize);
  }
  t1 = mysecond();

  // verification
  for (int i = 0; i < ArraySize; i++) 
    if (fabsf(image_cpu[i] - image_gpu[i]) > 0.1) {
      printf("@index %d cpu: %f gpu %f\n", i, image_cpu[i], image_gpu[i]);
      break;
    }

  pt_rate = pts/dt;
  flop_rate = flops/dt;
  printf("dt = %f\n", dt);
  printf("pt_rate (millions/sec) = %f\n", pt_rate/1e6);
  printf("flop_rate (Gflops) = %f\n", flop_rate/1e9);
  printf("speedup over cpu = %f\n", (t1 - t0) / dt);

  //release arrays
  free(vsq);
  free(next_s);
  free(current_s);
  free(next_r);
  free(current_r);
  free(image_cpu);
  free(image_gpu);
  hipFree(vsq_d);
  hipFree(next_s_d);
  hipFree(current_s_d);
  hipFree(next_r_d);
  hipFree(current_r_d);
  hipFree(image_d);
  hipFree(a_d);

  return 0;
}

