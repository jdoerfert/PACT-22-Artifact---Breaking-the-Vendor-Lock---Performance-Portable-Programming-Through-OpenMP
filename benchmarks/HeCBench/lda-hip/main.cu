#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <numeric>
#include <hip/hip_runtime.h>
#include "kernel.h"

int main(int argc, char* argv[]) {

  if (argc != 2) {
    printf("Usage: %s <repeat>\n", argv[0]);
    return 1;
  }

  // repeat kernel execution
  const int repeat = atoi(argv[1]);

  int i;
  srand(123);

  const int num_topics = 1000;
  const int num_words  = 10266;
  const int block_cnt  = 500;
  const int num_indptr = block_cnt; // max: num_words
  const int block_dim  = 256;
  const int num_iters  = 64;
 
  std::vector<float> alpha(num_topics);
  for (i = 0; i < num_topics; i++)  alpha[i] = (float) rand() / (float) RAND_MAX;

  std::vector<float> beta(num_topics * num_words);
  for (i = 0; i < num_topics * num_words; i++)  beta[i] = (float) rand() / (float) RAND_MAX;

  std::vector<float> grad_alpha(num_topics * block_cnt, 0.0f);
  std::vector<float> new_beta(num_topics * num_words, 0.0f);
  std::vector<int> h_locks(num_words, 0);
  std::vector<float> gamma (num_indptr * num_topics);

  std::vector<int> indptr (num_indptr+1, 0);
  indptr[num_indptr] = num_words-1;
  for (i = num_indptr; i >= 1; i--) {
    int t = indptr[i] - 1 - (rand() % (num_words/num_indptr));
    if (t < 0) break;
    indptr[i-1] = t;
  }
  const int num_cols = num_words;

  std::vector<int> cols (num_cols);
  std::vector<float> counts (num_cols);

  for (i = 0; i < num_cols; i++) {
    cols[i] = i;
    counts[i] = 0.5f; // arbitrary
  }

  float *d_alpha, *d_beta, *d_grad_alpha, *d_new_beta;
  float *d_counts, *d_gamma, *d_vali_losses, *d_train_losses;
  int *d_locks, *d_cols, *d_indptr;
  bool *d_vali;

  hipMalloc((void**)&d_alpha, sizeof(float) * num_topics);
  hipMemcpy(d_alpha, alpha.data(), sizeof(float) * num_topics, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_beta, sizeof(float) * num_topics * num_words);
  hipMemcpy(d_beta, beta.data(), sizeof(float) * num_topics * num_words, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_grad_alpha, sizeof(float) * num_topics * block_cnt);
  hipMemcpy(d_grad_alpha, grad_alpha.data(), sizeof(float) * block_cnt * num_topics, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_new_beta, sizeof(float) * num_topics * num_words);
  hipMemcpy(d_new_beta, new_beta.data(), sizeof(float) * num_topics * num_words, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_locks, sizeof(int) * num_words);
  hipMemcpy(d_locks, h_locks.data(), sizeof(int) * num_words, hipMemcpyHostToDevice);
  
  hipMalloc((void**)&d_cols, sizeof(int) * num_cols);
  hipMemcpy(d_cols, cols.data(), sizeof(int) * num_cols, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_indptr, sizeof(int) * (num_indptr + 1));
  hipMemcpy(d_indptr, indptr.data(), sizeof(int) * (num_indptr + 1), hipMemcpyHostToDevice);

  hipMalloc((void**)&d_vali, sizeof(bool) * num_cols);

  hipMalloc((void**)&d_counts, sizeof(float) * num_cols);
  hipMemcpy(d_counts, counts.data(), sizeof(float) * num_cols, hipMemcpyHostToDevice);

  // gamma will be initialized in the kernel
  hipMalloc((void**)&d_gamma, sizeof(float) * num_indptr * num_topics);

  // reset losses
  hipMalloc((void**)&d_train_losses, sizeof(float) * block_cnt);
  hipMemset(d_train_losses, 0, sizeof(float) * block_cnt);

  hipMalloc((void**)&d_vali_losses, sizeof(float) * block_cnt);
  hipMemset(d_vali_losses, 0, sizeof(float) * block_cnt);

  // store device results
  std::vector<float> train_losses(block_cnt), vali_losses(block_cnt);

  // training
  hipMemset(d_vali, 0, sizeof(bool) * num_cols); 
  bool init_gamma = false;
  for (i = 0; i < repeat; i++) {
    if (i == 0) init_gamma = true;
    hipLaunchKernelGGL(EstepKernel, dim3(block_cnt), dim3(block_dim), 4 * num_topics * sizeof(float), 0, 
      d_cols,
      d_indptr,
      d_vali,
      d_counts,
      init_gamma, num_cols, num_indptr, num_topics, num_iters,
      d_alpha,
      d_beta,
      d_gamma,
      d_grad_alpha,
      d_new_beta,
      d_train_losses,
      d_vali_losses,
      d_locks);
  }

  // validation
  hipMemset(d_vali, 0xFFFFFFFF, sizeof(bool) * num_cols); 
  for (i = 0; i < repeat; i++) {
    hipLaunchKernelGGL(EstepKernel, dim3(block_cnt), dim3(block_dim), 4 * num_topics * sizeof(float), 0, 
      d_cols,
      d_indptr,
      d_vali,
      d_counts,
      init_gamma, num_cols, num_indptr, num_topics, num_iters,
      d_alpha,
      d_beta,
      d_gamma,
      d_grad_alpha,
      d_new_beta,
      d_train_losses,
      d_vali_losses,
      d_locks);
  }

  hipMemcpy(vali_losses.data(), d_vali_losses, sizeof(float) * block_cnt, hipMemcpyDeviceToHost);
  hipMemcpy(train_losses.data(), d_train_losses, sizeof(float) * block_cnt, hipMemcpyDeviceToHost);

  float total_train_loss = std::accumulate(train_losses.begin(), train_losses.end(), 0.0f);
  float total_vali_loss = std::accumulate(vali_losses.begin(), vali_losses.end(), 0.0f);
  printf("Total train and validate loss: %f %f\n", total_train_loss, total_vali_loss);

  hipFree(d_cols);
  hipFree(d_indptr);
  hipFree(d_vali);
  hipFree(d_counts);
  hipFree(d_alpha);
  hipFree(d_beta);
  hipFree(d_gamma);
  hipFree(d_grad_alpha);
  hipFree(d_new_beta);
  hipFree(d_train_losses);
  hipFree(d_vali_losses);
  hipFree(d_locks);

  return 0;
}
