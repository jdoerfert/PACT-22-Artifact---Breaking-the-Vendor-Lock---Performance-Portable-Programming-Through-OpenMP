#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define m1  0x5555555555555555
#define m2  0x3333333333333333 
#define m4  0x0f0f0f0f0f0f0f0f 
#define h01 0x0101010101010101

#define BLOCK_SIZE 256

// reference implementation
int popcount_ref(unsigned long x)
{
  int count;
  for (count=0; x; count++)
    x &= x - 1;
  return count;
}

// HIP kernels
__global__ void pc1 (const unsigned long* __restrict data, int* __restrict r, const int length)
{
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  if (i >= length) return;
  unsigned long x = data[i];
  x -= (x >> 1) & m1;             //put count of each 2 bits into those 2 bits
  x = (x & m2) + ((x >> 2) & m2); //put count of each 4 bits into those 4 bits 
  x = (x + (x >> 4)) & m4;        //put count of each 8 bits into those 8 bits 
  x += x >>  8;  //put count of each 16 bits into their lowest 8 bits
  x += x >> 16;  //put count of each 32 bits into their lowest 8 bits
  x += x >> 32;  //put count of each 64 bits into their lowest 8 bits
  r[i] = x & 0x7f;
}

__global__ void pc2 (const unsigned long* __restrict data, int* __restrict r, const int length)
{
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  if (i >= length) return;
  unsigned long x = data[i];
  x -= (x >> 1) & m1;             //put count of each 2 bits into those 2 bits
  x = (x & m2) + ((x >> 2) & m2); //put count of each 4 bits into those 4 bits 
  x = (x + (x >> 4)) & m4;        //put count of each 8 bits into those 8 bits 
  r[i] = (x * h01) >> 56;  //returns left 8 bits of x + (x<<8) + (x<<16) + (x<<24) + ... 
}

__global__ void pc3 (const unsigned long* __restrict data, int* __restrict r, const int length)
{
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  if (i >= length) return;
  char count;
  unsigned long x = data[i];
  for (count=0; x; count++) x &= x - 1;
  r[i] = count;
}

__global__ void pc4 (const unsigned long* __restrict data, int* __restrict r, const int length)
{
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  if (i >= length) return;
  unsigned long x = data[i];
  char cnt = 0;
  for (char i = 0; i < 64; i++)
  {
    cnt = cnt + (x & 0x1);
    x = x >> 1;
  }
  r[i] = cnt;
}

__global__ void pc5 (const unsigned long* __restrict data, int* __restrict r, const int length)
{
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  if (i >= length) return;
  unsigned long x = data[i];
  const unsigned char a[256] = { 0,1,1,2,1,2,2,3,1,2,2,3,2,3,3,4,1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,4,5,5,6,5,6,6,7,5,6,6,7,6,7,7,8};
  const unsigned char b[256] = { 0,1,1,2,1,2,2,3,1,2,2,3,2,3,3,4,1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,4,5,5,6,5,6,6,7,5,6,6,7,6,7,7,8};
  const unsigned char c[256] = { 0,1,1,2,1,2,2,3,1,2,2,3,2,3,3,4,1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,4,5,5,6,5,6,6,7,5,6,6,7,6,7,7,8};
  const unsigned char d[256] = { 0,1,1,2,1,2,2,3,1,2,2,3,2,3,3,4,1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,4,5,5,6,5,6,6,7,5,6,6,7,6,7,7,8};

  unsigned char i1 = a[(x & 0xFF)];
  unsigned char i2 = a[(x >> 8) & 0xFF];
  unsigned char i3 = b[(x >> 16) & 0xFF];
  unsigned char i4 = b[(x >> 24) & 0xFF];
  unsigned char i5 = c[(x >> 32) & 0xFF];
  unsigned char i6 = c[(x >> 40) & 0xFF];
  unsigned char i7 = d[(x >> 48) & 0xFF];
  unsigned char i8 = d[(x >> 56) & 0xFF];
  r[i] = (i1+i2)+(i3+i4)+(i5+i6)+(i7+i8);
}

__global__ void pc6 (const unsigned long* __restrict data, int* __restrict r, const int length)
{
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  if (i >= length) return;
  r[i] = __popcll(data[i]);
}

void checkResults(const unsigned long *d, const int *r, const int length)
{
  int error = 0;
  for (int i=0;i<length;i++)
    if (popcount_ref(d[i]) != r[i]) {
      error = 1;
      break;
    }

  if (error)
    printf("Fail\n");
  else
    printf("Success\n");
}

int main(int argc, char* argv[])
{
  const int length = atoi(argv[1]);
  unsigned long *data = NULL;
  int* __restrict result = NULL;
  posix_memalign((void**)&data, 1024, length*sizeof(unsigned long));
  posix_memalign((void**)&result, 1024, length*sizeof(int));

  // initialize input
  srand(2);
  for (int i = 0; i < length; i++) {
    unsigned long t = (unsigned long)rand() << 32;
    data[i] = t | rand();
  }

  // run each popcount implementation 100 times

  unsigned long* d_data;
  hipMalloc((void**)&d_data, sizeof(unsigned long)*length);
  hipMemcpy(d_data, data, sizeof(unsigned long)*length, hipMemcpyHostToDevice);

  int* d_result;
  hipMalloc((void**)&d_result, sizeof(int)*length);

  dim3 grids ((length+BLOCK_SIZE-1)/BLOCK_SIZE);
  dim3 threads (BLOCK_SIZE);

  for (int n = 0; n < 100; n++) {
    hipLaunchKernelGGL(pc1, dim3(grids), dim3(threads), 0, 0, d_data, d_result, length);
  }
  hipMemcpy(result, d_result, sizeof(int)*length, hipMemcpyDeviceToHost);
  checkResults(data, result, length);
  //========================================================================================

  for (int n = 0; n < 100; n++) {
    hipLaunchKernelGGL(pc2, dim3(grids), dim3(threads), 0, 0, d_data, d_result, length);
  }
  hipMemcpy(result, d_result, sizeof(int)*length, hipMemcpyDeviceToHost);
  checkResults(data, result, length);
  //========================================================================================

  for (int n = 0; n < 100; n++) {
    hipLaunchKernelGGL(pc3, dim3(grids), dim3(threads), 0, 0, d_data, d_result, length);
  }
  hipMemcpy(result, d_result, sizeof(int)*length, hipMemcpyDeviceToHost);
  checkResults(data, result, length);
  //========================================================================================

  for (int n = 0; n < 100; n++) {
    hipLaunchKernelGGL(pc4, dim3(grids), dim3(threads), 0, 0, d_data, d_result, length);
  }
  hipMemcpy(result, d_result, sizeof(int)*length, hipMemcpyDeviceToHost);
  checkResults(data, result, length);
  //========================================================================================

  for (int n = 0; n < 100; n++) {
    hipLaunchKernelGGL(pc5, dim3(grids), dim3(threads), 0, 0, d_data, d_result, length);
  }
  hipMemcpy(result, d_result, sizeof(int)*length, hipMemcpyDeviceToHost);
  checkResults(data, result, length);
  //========================================================================================

  for (int n = 0; n < 100; n++) {
    hipLaunchKernelGGL(pc6, dim3(grids), dim3(threads), 0, 0, d_data, d_result, length);
  }
  hipMemcpy(result, d_result, sizeof(int)*length, hipMemcpyDeviceToHost);
  checkResults(data, result, length);
  //========================================================================================

  hipFree(d_data);
  hipFree(d_result);
  free(data);
  free(result);
  return 0;
}
