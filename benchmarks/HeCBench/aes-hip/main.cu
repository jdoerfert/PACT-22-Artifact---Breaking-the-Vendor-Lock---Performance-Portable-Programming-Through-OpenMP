#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "SDKBitMap.h"
#include "aes.h"
#include "kernels.cu"
#include "reference.cu"
#include "utils.cu"

int main(int argc, char * argv[])
{
  const unsigned int keySizeBits = 128;
  const unsigned int rounds = 10;
  const unsigned int seed = 123;

  const int iterations = atoi(argv[1]);
  const bool decrypt = atoi(argv[2]);
  const char* filePath = argv[3];

  SDKBitMap image;
  image.load(filePath);
  const int width  = image.getWidth();
  const int height = image.getHeight();

  /* check condition for the bitmap to be initialized */
  if (width <= 0 || height <= 0) return 1;

  std::cout << "Image width and height: " 
            << width << " " << height << std::endl;

  uchar4 *pixels = image.getPixels();

  unsigned int sizeBytes = width*height*sizeof(uchar);
  uchar *input = (uchar*)malloc(sizeBytes); 

  /* initialize the input array, do NOTHING but assignment when decrypt*/
  if (decrypt)
    convertGrayToGray(pixels, input, height, width);
  else
    convertColorToGray(pixels, input, height, width);

  unsigned int keySize = keySizeBits/8; // 1 Byte = 8 bits

  unsigned int keySizeBytes = keySize*sizeof(uchar);

  uchar *key = (uchar*)malloc(keySizeBytes);

  fillRandom<uchar>(key, keySize, 1, 0, 255, seed); 

  // expand the key
  unsigned int explandedKeySize = (rounds+1)*keySize;
  uchar *expandedKey = (uchar*)malloc(explandedKeySize*sizeof(uchar));
  uchar *roundKey    = (uchar*)malloc(explandedKeySize*sizeof(uchar));

  keyExpansion(key, expandedKey, keySize, explandedKeySize);
  for(unsigned int i = 0; i < rounds+1; ++i)
  {
    createRoundKey(expandedKey + keySize*i, roundKey + keySize*i);
  }

  // save device result
  uchar* output = (uchar*)malloc(sizeBytes);

  uchar *inputBuffer;
  hipMalloc((void**)&inputBuffer, width * height);
  hipMemcpy(inputBuffer, input, width * height, hipMemcpyHostToDevice);

  uchar *outputBuffer;
  hipMalloc((void**)&outputBuffer, width * height);

  uchar *rKeyBuffer;
  hipMalloc((void**)&rKeyBuffer, explandedKeySize);
  hipMemcpy(rKeyBuffer, roundKey, explandedKeySize, hipMemcpyHostToDevice);

  uchar *sBoxBuffer;
  hipMalloc((void**)&sBoxBuffer, 256);
  hipMemcpy(sBoxBuffer, sbox, 256, hipMemcpyHostToDevice);

  uchar *rsBoxBuffer;
  hipMalloc((void**)&rsBoxBuffer, 256);
  hipMemcpy(rsBoxBuffer, rsbox, 256, hipMemcpyHostToDevice);

  std::cout << "Executing kernel for " << iterations 
            << " iterations" << std::endl;
  std::cout << "-------------------------------------------" << std::endl;

  dim3 grid (width/4, height/4);
  dim3 block (1, 4);

  for(int i = 0; i < iterations; i++)
  {
    if (decrypt) 
      hipLaunchKernelGGL(AESDecrypt, dim3(grid), dim3(block ), 0, 0, 
        (uchar4*)outputBuffer,
        (uchar4*)inputBuffer,
        (uchar4*)rKeyBuffer,
        rsBoxBuffer,
        width, rounds);
    else
      hipLaunchKernelGGL(AESEncrypt, dim3(grid), dim3(block ), 0, 0, 
        (uchar4*)outputBuffer,
        (uchar4*)inputBuffer,
        (uchar4*)rKeyBuffer,
        sBoxBuffer,
        width, rounds);

    hipMemcpy(output, outputBuffer, width * height, hipMemcpyDeviceToHost);
  }

  // Verify
  uchar *verificationOutput = (uchar *) malloc(width*height*sizeof(uchar));

  reference(verificationOutput, input, roundKey, explandedKeySize, 
      width, height, decrypt, rounds, keySize);

  /* compare the results and see if they match */
  if(memcmp(output, verificationOutput, height*width*sizeof(uchar)) == 0)
    std::cout<<"Passed!\n";
  else
    std::cout<<"Failed\n";

  /* release program resources (input memory etc.) */
  hipFree(inputBuffer);
  hipFree(outputBuffer);
  hipFree(rKeyBuffer);
  hipFree(sBoxBuffer);
  hipFree(rsBoxBuffer);

  if(input) free(input);

  if(key) free(key);

  if(expandedKey) free(expandedKey);

  if(roundKey) free(roundKey);

  if(output) free(output);

  if(verificationOutput) free(verificationOutput);

  return 0;
}

