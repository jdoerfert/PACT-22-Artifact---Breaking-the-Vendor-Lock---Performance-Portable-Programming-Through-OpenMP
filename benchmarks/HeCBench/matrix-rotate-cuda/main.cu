#include <cmath>
#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>

__global__ void rotate_matrix_parallel (float *matrix, const int n) {
  int layer = blockIdx.x * blockDim.x + threadIdx.x;
  if (layer < n/2) {
    int first = layer;
    int last = n - 1 - layer;
    for(int i = first; i < last; ++i) {
      int offset = i - first;

      float top = matrix[first*n+i]; // save top
      // left -> top
      matrix[first*n+i] = matrix[(last-offset)*n+first];

      // bottom -> left
      matrix[(last-offset)*n+first] = matrix[last*n+(last-offset)];

      // right -> bottom
      matrix[last*n+(last-offset)] = matrix[i*n+last];

      // top -> right
      matrix[i*n+last] = top; // right <- saved top
    }
  }
}

void rotate_matrix_serial(float *matrix, int n) {

  for (int layer = 0; layer < n / 2; ++layer) {
    int first = layer;
    int last = n - 1 - layer;
    for(int i = first; i < last; ++i) {
      int offset = i - first;
        float top = matrix[first*n+i]; // save top
        // left -> top
        matrix[first*n+i] = matrix[(last-offset)*n+first];

        // bottom -> left
        matrix[(last-offset)*n+first] = matrix[last*n+(last-offset)];

        // right -> bottom
        matrix[last*n+(last-offset)] = matrix[i*n+last];

        // top -> right
        matrix[i*n+last] = top; // right <- saved top
    }
  }
}

int main(int argc, char** argv) {

  const int n = atoi(argv[1]);
  float *serial_res = (float*) aligned_alloc(1024, n*n*sizeof(float));
  float *parallel_res = (float*) aligned_alloc(1024, n*n*sizeof(float));

  for (int i = 0; i < n; i++)
    for (int j = 0; j < n; j++)
      serial_res[i*n+j] = parallel_res[i*n+j] = i*n+j;

  float *d_parallel_res;
  hipMalloc((void**)&d_parallel_res, n*n*sizeof(float));
  hipMemcpy(d_parallel_res, parallel_res, n*n*sizeof(float), hipMemcpyHostToDevice);

  for (int i = 0; i < 100; i++) {
    rotate_matrix_serial(serial_res, n);
    rotate_matrix_parallel<<<(n/2+255)/256, 256>>>(d_parallel_res, n);
  }
  hipMemcpy(parallel_res, d_parallel_res, n*n*sizeof(float), hipMemcpyDeviceToHost);

  int errors = 0;
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      if (serial_res[i*n+j] != parallel_res[i*n+j]) {
        errors++; 
        break;
      }
    }
  }
  if (errors) 
    printf("fail\n");
  else 
    printf("success\n");

  free(serial_res);
  free(parallel_res);
  hipFree(d_parallel_res);
  return 0;
}

