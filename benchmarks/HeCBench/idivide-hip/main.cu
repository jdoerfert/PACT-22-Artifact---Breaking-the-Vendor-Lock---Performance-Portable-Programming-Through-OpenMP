#include <iostream>
#include <cstdio>
#include <chrono>
#include <hip/hip_runtime.h>

#define NOW std::chrono::high_resolution_clock::now()

#include "fastdiv.h"
#include "kernels.h"

// Functional test returns 1 when it fails; otherwise it returns 0
int test()
{
  const int blocks = 256;
  const int divisor_count = 100000;
  const int divident_count = 1000000;

  int grids = (divident_count + blocks - 1) / blocks;

  int buf[4];
  int * buf_d;
  hipMalloc(&buf_d, sizeof(int) * 4);

  std::cout << "Running functional test on " << divisor_count << " divisors, with " 
            << grids * blocks << " dividents for each divisor" << std::endl;

  for(int d = 1; d < divisor_count; ++d)
  {
    for(int sign = 1; sign >= -1; sign -= 2)
    {
      int divisor = d * sign;
      hipMemset(buf_d, 0, sizeof(int) * 4);
      hipLaunchKernelGGL(check, grids, blocks, 0, 0, divisor, buf_d);
      hipMemcpy(buf, buf_d, sizeof(int) * 4, hipMemcpyDeviceToHost);

      if (buf[0] > 0)
      {
        std::cout << buf[0] << " wrong results, one of them is for divident " 
                  << buf[1] << ", correct quotient = " << buf[2] 
                  << ", fast computed quotient = " << buf[3] << std::endl;
        hipFree(buf_d);
        return 1;
      }
    }
  }

  hipFree(buf_d);
  return 0;
}

int main(int argc, char* argv[])
{
  // performance evaluation after functional test is done
  if (test()) return 1;

  const int grids = 32 * 1024;
  const int blocks = 256;

  // warmup may be needed for accurate performance measurement with chrono
  for (int i = 0; i < 100; i++) {
    hipLaunchKernelGGL(HIP_KERNEL_NAME(throughput_test<int>), grids, blocks, 0, 0, 3, 5, 7, 0, 0);
    hipLaunchKernelGGL(HIP_KERNEL_NAME(throughput_test<int_fastdiv>), grids, blocks, 0, 0, 3, 5, 7, 0, 0);
  }
  hipDeviceSynchronize();

  std::cout << "THROUGHPUT TEST" << std::endl;

  std::cout << "Benchmarking plain division by constant... ";
  auto start = NOW;

  for (int i = 0; i < 100; i++)
    hipLaunchKernelGGL(HIP_KERNEL_NAME(throughput_test<int>), grids, blocks, 0, 0, 3, 5, 7, 0, 0);
  hipDeviceSynchronize();

  auto end = NOW;
  std::chrono::duration<double> elapsed_time_slow = end-start;
  std::cout << elapsed_time_slow.count() << " seconds" << std::endl;

  std::cout << "Benchmarking fast division by constant... ";
  start = NOW;

  for (int i = 0; i < 100; i++)
    hipLaunchKernelGGL(HIP_KERNEL_NAME(throughput_test<int_fastdiv>), grids, blocks, 0, 0, 3, 5, 7, 0, 0);
  hipDeviceSynchronize();

  end = NOW;
  std::chrono::duration<double> elapsed_time_fast = end-start;
  std::cout << elapsed_time_fast.count() << " seconds" << std::endl;

  std::cout << "Speedup = " << elapsed_time_slow.count() / elapsed_time_fast.count() << std::endl;

  // warmup
  for (int i = 0; i < 100; i++) {
    hipLaunchKernelGGL(HIP_KERNEL_NAME(latency_test<int>), grids, blocks, 0, 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 0, 0);
    hipLaunchKernelGGL(HIP_KERNEL_NAME(latency_test<int_fastdiv>), grids, blocks, 0, 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 0, 0);
  }
  hipDeviceSynchronize();

  std::cout << "LATENCY TEST" << std::endl;
  std::cout << "Benchmarking plain division by constant... ";
  start = NOW;

  for (int i = 0; i < 100; i++)
    hipLaunchKernelGGL(HIP_KERNEL_NAME(latency_test<int>), grids, blocks, 0, 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 0, 0);
  hipDeviceSynchronize();

  end = NOW;
  elapsed_time_slow = end-start;
  std::cout << elapsed_time_slow.count() << " seconds" << std::endl;

  std::cout << "Benchmarking fast division by constant... ";
  start = NOW;

  for (int i = 0; i < 100; i++)
    hipLaunchKernelGGL(HIP_KERNEL_NAME(latency_test<int_fastdiv>), grids, blocks, 0, 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 0, 0);
  hipDeviceSynchronize();

  end = NOW;
  elapsed_time_fast = end-start;
  std::cout << elapsed_time_fast.count() << " seconds" << std::endl;

  std::cout << "Speedup = " << elapsed_time_slow.count() / elapsed_time_fast.count() << std::endl;
  return 0;
}
