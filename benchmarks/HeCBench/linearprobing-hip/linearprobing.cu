#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <vector>
#include "linearprobing.h"

// 32 bit Murmur3 hash
__device__ uint32_t hash(uint32_t k)
{
  k ^= k >> 16;
  k *= 0x85ebca6b;
  k ^= k >> 13;
  k *= 0xc2b2ae35;
  k ^= k >> 16;
  return k & (kHashTableCapacity-1);
}


// Insert the key/values in kvs into the hashtable
__global__ void
k_hashtable_insert(KeyValue*__restrict hashtable,
                   const KeyValue*__restrict kvs, unsigned int numkvs)
{
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < numkvs)
  {
    uint32_t key = kvs[tid].key;
    uint32_t value = kvs[tid].value;
    uint32_t slot = hash(key);

    while (true)
    {
      uint32_t prev = atomicCAS(&hashtable[slot].key, kEmpty, key);
      if (prev == kEmpty || prev == key)
      {
        hashtable[slot].value = value;
        return;
      }

      slot = (slot + 1) & (kHashTableCapacity-1);
    }
  }
}

void insert_hashtable(KeyValue*__restrict pHashTable, const KeyValue*__restrict kvs, uint32_t num_kvs)
{
  // Copy the keyvalues to device 
  KeyValue* device_kvs;
  hipMalloc(&device_kvs, sizeof(KeyValue) * num_kvs);
  hipMemcpy(device_kvs, kvs, sizeof(KeyValue) * num_kvs, hipMemcpyHostToDevice);

  // Insert all the keys into the hash table
  const int threadblocksize = 256;
  int gridsize = ((uint32_t)num_kvs + threadblocksize - 1) / threadblocksize;
  hipLaunchKernelGGL(k_hashtable_insert, dim3(gridsize), dim3(threadblocksize), 0, 0, pHashTable, device_kvs, (uint32_t)num_kvs);
  hipDeviceSynchronize();

  hipFree(device_kvs);
}

// Delete each key in kvs from the hash table, if the key exists
// A deleted key is left in the hash table, but its value is set to kEmpty
// Deleted keys are not reused; once a key is assigned a slot, it never moves
__global__ void 
k_hashtable_delete(KeyValue*__restrict hashtable, 
                   const KeyValue*__restrict kvs, unsigned int numkvs)
{
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < kHashTableCapacity)
  {
    uint32_t key = kvs[tid].key;
    uint32_t slot = hash(key);

    while (true)
    {
      if (hashtable[slot].key == key)
      {
        hashtable[slot].value = kEmpty;
        return;
      }
      if (hashtable[slot].key == kEmpty)
      {
        return;
      }
      slot = (slot + 1) & (kHashTableCapacity - 1);
    }
  }
}

void delete_hashtable(KeyValue* pHashTable, const KeyValue* kvs, uint32_t num_kvs)
{
  // Copy the keyvalues to device
  KeyValue* device_kvs;
  hipMalloc(&device_kvs, sizeof(KeyValue) * num_kvs);
  hipMemcpy(device_kvs, kvs, sizeof(KeyValue) * num_kvs, hipMemcpyHostToDevice);

  // Insert all the keys into the hash table
  const int threadblocksize = 256;
  int gridsize = ((uint32_t)num_kvs + threadblocksize - 1) / threadblocksize;
  hipLaunchKernelGGL(k_hashtable_delete, dim3(gridsize), dim3(threadblocksize), 0, 0, pHashTable, device_kvs, (uint32_t)num_kvs);
  hipDeviceSynchronize();

  hipFree(device_kvs);
}

// Iterate over every item in the hashtable; return non-empty key/values
__global__ void
k_iterate_hashtable(KeyValue*__restrict pHashTable,
                    KeyValue*__restrict kvs, uint32_t* kvs_size)
{
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < kHashTableCapacity) 
  {
    if (pHashTable[tid].key != kEmpty) 
    {
      uint32_t value = pHashTable[tid].value;
      if (value != kEmpty)
      {
        uint32_t size = atomicAdd(kvs_size, 1);
        kvs[size] = pHashTable[tid];
      }
    }
  }
}

std::vector<KeyValue> iterate_hashtable(KeyValue* pHashTable)
{
  uint32_t* device_num_kvs;
  hipMalloc(&device_num_kvs, sizeof(uint32_t));
  hipMemset(device_num_kvs, 0, sizeof(uint32_t));

  KeyValue* device_kvs;
  hipMalloc(&device_kvs, sizeof(KeyValue) * kNumKeyValues);

  const int threadblocksize = 256;
  int gridsize = (kHashTableCapacity + threadblocksize - 1) / threadblocksize;
  hipLaunchKernelGGL(k_iterate_hashtable, dim3(gridsize), dim3(threadblocksize), 0, 0, pHashTable, device_kvs, device_num_kvs);

  uint32_t num_kvs;
  hipMemcpy(&num_kvs, device_num_kvs, sizeof(uint32_t), hipMemcpyDeviceToHost);

  std::vector<KeyValue> kvs;
  kvs.resize(num_kvs);

  hipMemcpy(kvs.data(), device_kvs, sizeof(KeyValue) * num_kvs, hipMemcpyDeviceToHost);

  hipFree(device_kvs);
  hipFree(device_num_kvs);

  return kvs;
}

