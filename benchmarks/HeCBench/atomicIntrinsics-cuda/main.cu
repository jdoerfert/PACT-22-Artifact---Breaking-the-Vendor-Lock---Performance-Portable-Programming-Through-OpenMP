#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* A simple program demonstrating trivial use of global memory atomic
 * device functions (atomic*() functions).
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// Includes CUDA
#include <hip/hip_runtime.h>

// Includes, kernels
#include "simpleAtomicIntrinsics_kernel.cuh"


#define min(a,b) (a) < (b) ? (a) : (b)
#define max(a,b) (a) > (b) ? (a) : (b)


////////////////////////////////////////////////////////////////////////////////
//! Compute reference data set
//! Each element is multiplied with the number of threads / array length
//! @param reference  reference data, computed but preallocated
//! @param idata      input data as provided to device
//! @param len        number of elements in reference / idata
////////////////////////////////////////////////////////////////////////////////
void
computeGold(int *gpuData, const int len)
{
    int val = 0;

    for (int i = 0; i < len; ++i)
    {
        val += 10;
    }

    if (val != gpuData[0])
    {
        printf("Add failed %d %d\n", val, gpuData[0]);
    }

    val = 0;

    for (int i = 0; i < len; ++i)
    {
        val -= 10;
    }

    if (val != gpuData[1])
    {
        printf("Sub failed: %d %d\n", val, gpuData[1]);
    }

    val = -(1<<8);

    for (int i = 0; i < len; ++i)
    {
        val = max(val, i);
    }

    if (val != gpuData[2])
    {
        printf("Max failed: %d %d\n", val, gpuData[2]);
    }

    val = 1 << 8;

    for (int i = 0; i < len; ++i)
    {
        val = min(val, i);
    }

    if (val != gpuData[3])
    {
        printf("Min failed: %d %d\n", val, gpuData[3]);
    }

    val = 0xff;

    for (int i = 0; i < len; ++i)
    {
        val &= (2 * i + 7);
    }

    if (val != gpuData[4])
    {
        printf("And failed: %d %d\n", val, gpuData[4]);
    }

    val = 0;

    for (int i = 0; i < len; ++i)
    {
        val |= (1 << i);
    }

    if (val != gpuData[5])
    {
        printf("Or failed: %d %d\n", val, gpuData[5]);
    }

    val = 0xff;

    for (int i = 0; i < len; ++i)
    {
        val ^= i;
    }

    if (val != gpuData[6])
    {
        printf("Xor failed %d %d\n", val, gpuData[6]);
    }

    printf("PASS\n");
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    unsigned int len = 1 << 27;
    unsigned int numThreads = 256;
    unsigned int numBlocks = (len + numThreads - 1) / numThreads;
    unsigned int numData = 7;
    unsigned int memSize = sizeof(int) * numData;
    int gpuData[] = {0, 0, -(1<<8), 1<<8, 0xff, 0, 0xff};

    // allocate device memory for result
    int *dOData;
    hipMalloc((void **) &dOData, memSize);

    for (int i = 0; i < 1; i++) {
      // copy host memory to device to initialize to zero
      hipMemcpy(dOData, gpuData, memSize, hipMemcpyHostToDevice);

      // execute the kernel
      testKernel<<<numBlocks, numThreads>>>(dOData);
    }

    //Copy result from device to host
    hipMemcpy(gpuData, dOData, memSize, hipMemcpyDeviceToHost);

    computeGold(gpuData, numThreads * numBlocks);

    hipFree(dOData);
    return 0;
}
