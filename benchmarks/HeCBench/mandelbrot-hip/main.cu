//==============================================================
// Copyright © 2019 Intel Corporation
//
// SPDX-License-Identifier: MIT
//
// A HIP port: Zheming Jin
// =============================================================

#include <chrono>
#include "common.hpp"
#include "mandel.hpp"

using namespace std;

void ShowDevice() {
  // Output platform and device information.
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, 0);

    std::cout << "Device name " << devProp.name << std::endl;
}

void Execute() {
  // Demonstrate the Mandelbrot calculation serial and parallel
  MandelParallel m_par(row_size, col_size, max_iterations);
  MandelSerial m_ser(row_size, col_size, max_iterations);

  // Run the code once to trigger JIT
  m_par.Evaluate();

  // Run the parallel version
  common::MyTimer t_par;
  // time the parallel computation
  for (int i = 0; i < repetitions; ++i) 
    m_par.Evaluate();
  common::Duration parallel_time = t_par.elapsed();

  // Print the results
  m_par.Print();

  // Run the serial version
  common::MyTimer t_ser;
  m_ser.Evaluate();
  common::Duration serial_time = t_ser.elapsed();

  // Report the results
  cout << std::setw(20) << "serial time: " << serial_time.count() << "s\n";
  cout << std::setw(20) << "parallel time: " << (parallel_time / repetitions).count() << "s\n";

  // Validating
  m_par.Verify(m_ser);
}

void Usage(string program_name) {
  // Utility function to display argument usage
  cout << " Incorrect parameters\n";
  cout << " Usage: ";
  cout << program_name << "\n\n";
  exit(-1);
}

int main(int argc, char *argv[]) {
  if (argc != 1) {
    Usage(argv[0]);
  }

  try {
    ShowDevice();
    Execute();
  } catch (...) {
    cout << "Failure\n";
    terminate();
  }
  cout << "Success\n";
  return 0;
}
