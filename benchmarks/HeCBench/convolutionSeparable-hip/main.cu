#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "conv.h"

int main(int argc, char **argv)
{
  const unsigned int imageW = 3072;
  const unsigned int imageH = 3072;

  float* h_Kernel    = (float*)malloc(KERNEL_LENGTH * sizeof(float));
  float* h_Input     = (float*)malloc(imageW * imageH * sizeof(float));
  float* h_Buffer    = (float*)malloc(imageW * imageH * sizeof(float));
  float* h_OutputCPU = (float*)malloc(imageW * imageH * sizeof(float));
  float* h_OutputGPU = (float*)malloc(imageW * imageH * sizeof(float));

  srand(2009);
  for(unsigned int i = 0; i < KERNEL_LENGTH; i++)
    h_Kernel[i] = (float)(rand() % 16);

  for(unsigned int i = 0; i < imageW * imageH; i++)
    h_Input[i] = (float)(rand() % 16);


  float* d_Kernel;
  hipMalloc((void**)&d_Kernel, sizeof(float)*KERNEL_LENGTH);
  hipMemcpy(d_Kernel, h_Kernel, sizeof(float)*KERNEL_LENGTH, hipMemcpyHostToDevice);

  float* d_Input;
  hipMalloc((void**)&d_Input, sizeof(float)*imageW*imageH);
  hipMemcpy(d_Input, h_Input, sizeof(float)*imageW*imageH, hipMemcpyHostToDevice);

  float* d_Buffer;
  hipMalloc((void**)&d_Buffer, sizeof(float)*imageW*imageH);

  float* d_Output;
  hipMalloc((void**)&d_Output, sizeof(float)*imageW*imageH);


  //Just a single run or a warmup iteration
  convolutionRows(
      d_Buffer,
      d_Input,
      d_Kernel,
      imageW,
      imageH,
      imageW);

  hipMemcpy(h_Buffer, d_Buffer, sizeof(float)*imageW * imageH, hipMemcpyDeviceToHost);

  convolutionColumns(
      d_Output,
      d_Buffer,
      d_Kernel,
      imageW,
      imageH,
      imageW);

  const int numIterations = 100;

  for(int iter = 0; iter < numIterations; iter++){
    convolutionRows(
        d_Buffer,
        d_Input,
        d_Kernel,
        imageW,
        imageH,
        imageW);

    convolutionColumns(
        d_Output,
        d_Buffer,
        d_Kernel,
        imageW,
        imageH,
        imageW);
  }

  hipMemcpy(h_OutputGPU, d_Output, sizeof(float)*imageW * imageH, hipMemcpyDeviceToHost);

  printf("Comparing against Host/C++ computation...\n"); 
  convolutionRowHost(h_Buffer, h_Input, h_Kernel, imageW, imageH, KERNEL_RADIUS);
  convolutionColumnHost(h_OutputCPU, h_Buffer, h_Kernel, imageW, imageH, KERNEL_RADIUS);
  double sum = 0, delta = 0;
  double L2norm;
  for(unsigned int i = 0; i < imageW * imageH; i++){
    delta += (h_OutputCPU[i] - h_OutputGPU[i]) * (h_OutputCPU[i] - h_OutputGPU[i]);
    sum += h_OutputCPU[i] * h_OutputCPU[i];
  }
  L2norm = sqrt(delta / sum);
  printf("Relative L2 norm: %.3e\n\n", L2norm);

  free(h_OutputGPU);
  free(h_OutputCPU);
  free(h_Buffer);
  free(h_Input);
  free(h_Kernel);
  hipFree(d_Kernel);
  hipFree(d_Input);
  hipFree(d_Buffer);
  hipFree(d_Output);

  printf("%s\n", L2norm < 1e-6 ? "PASS" : "FAIL");

  return 0;
}
