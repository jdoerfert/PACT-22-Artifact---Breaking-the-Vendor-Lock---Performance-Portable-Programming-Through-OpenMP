#include "hip/hip_runtime.h"
#include "morphology.h"

enum class MorphOpType {
    ERODE,
    DILATE,
};


template <MorphOpType opType>
inline __device__ unsigned char elementOp(unsigned char lhs, unsigned char rhs)
{
}

template <>
inline __device__ unsigned char elementOp<MorphOpType::ERODE>(unsigned char lhs, unsigned char rhs)
{
    return min(lhs, rhs);
}

template <>
inline __device__ unsigned char elementOp<MorphOpType::DILATE>(unsigned char lhs, unsigned char rhs)
{
    return max(lhs, rhs);
}


template <MorphOpType opType>
inline __device__ unsigned char borderValue()
{
}

template <>
inline __device__ unsigned char borderValue<MorphOpType::ERODE>()
{
    return BLACK;
}

template <>
inline __device__ unsigned char borderValue<MorphOpType::DILATE>()
{
    return WHITE;
}


// NOTE: step-efficient parallel scan
template <MorphOpType opType>
__device__ void reversedScan(unsigned char* __restrict__ buffer,
        unsigned char* __restrict__ opArray,
        const int selSize,
        const int tid)
{
    opArray[tid] = buffer[tid];
    __syncthreads();

    for (int offset = 1; offset < selSize; offset *= 2) {
        if (tid <= selSize - 1 - offset) {
            opArray[tid] = elementOp<opType>(opArray[tid], opArray[tid + offset]);
        }
        __syncthreads();
    }
}

// NOTE: step-efficient parallel scan
template <MorphOpType opType>
__device__ void scan(unsigned char* __restrict__ buffer,
        unsigned char* __restrict__ opArray,
        const int selSize,
        const int tid)
{
    opArray[tid] = buffer[tid];
    __syncthreads();

    for (int offset = 1; offset < selSize; offset *= 2) {
        if (tid >= offset) {
            opArray[tid] = elementOp<opType>(opArray[tid], opArray[tid - offset]);
        }
        __syncthreads();
    }
}

// NOTE: step-efficient parallel scan
template <MorphOpType opType>
__device__ void twoWayScan(unsigned char* __restrict__ buffer,
        unsigned char* __restrict__ opArray,
        const int selSize,
        const int tid)
{
    opArray[tid] = buffer[tid];
    opArray[tid + selSize] = buffer[tid + selSize];
    __syncthreads();

    for (int offset = 1; offset < selSize; offset *= 2) {
        if (tid >= offset) {
            opArray[tid + selSize - 1] = 
                elementOp<opType>(opArray[tid + selSize - 1], opArray[tid + selSize - 1 - offset]);
        }
        if (tid <= selSize - 1 - offset) {
            opArray[tid] = elementOp<opType>(opArray[tid], opArray[tid + offset]);
        }
        __syncthreads();
    }
}


template <MorphOpType opType>
__global__ void vhgw_horiz(unsigned char* __restrict__ dst,
        unsigned char* __restrict__ src,
        const int width,
        const int height,
        const int selSize
        )
{
    HIP_DYNAMIC_SHARED(unsigned char, sMem);
    unsigned char* buffer = sMem;
    unsigned char* opArray = buffer + 2 * selSize;

    const int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    const int tidy = threadIdx.y + blockIdx.y * blockDim.y;
    if (tidx >= width || tidy >= height) {
        return;
    }

    buffer[threadIdx.x] = src[tidy * width + tidx];
    if (tidx + selSize < width) {
        buffer[threadIdx.x + selSize] = src[tidy * width + tidx + selSize];
    }
    __syncthreads();

    twoWayScan<opType>(buffer, opArray, selSize, threadIdx.x);

    if (tidx + selSize/2 < width - selSize/2) {
        dst[tidy * width + tidx + selSize/2] = 
            elementOp<opType>(opArray[threadIdx.x], opArray[threadIdx.x + selSize - 1]);
    }
}

template <MorphOpType opType>
__global__ void vhgw_vert(unsigned char* __restrict__ dst,
        unsigned char* __restrict__ src,
        const int width,
        const int height,
        const int selSize
        )
{
    HIP_DYNAMIC_SHARED(unsigned char, sMem);
    unsigned char* buffer = sMem;
    unsigned char* opArray = buffer + 2 * selSize;

    const int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    const int tidy = threadIdx.y + blockIdx.y * blockDim.y;
    if (tidy >= height || tidx >= width) {
        return;
    }

    buffer[threadIdx.y] = src[tidy * width + tidx];
    if (tidy + selSize < height) {
        buffer[threadIdx.y + selSize] = src[(tidy + selSize) * width + tidx];
    }
    __syncthreads();

    twoWayScan<opType>(buffer, opArray, selSize, threadIdx.y);

    if (tidy + selSize/2 < height - selSize/2) {
        dst[(tidy + selSize/2) * width + tidx] = 
            elementOp<opType>(opArray[threadIdx.y], opArray[threadIdx.y + selSize - 1]);
    }

    if (tidy < selSize/2 || tidy >= height - selSize/2) {
        dst[tidy * width + tidx] = borderValue<opType>();
    }
}


template <MorphOpType opType>
void morphology(unsigned char* img_d,
        unsigned char* tmp_d,
        const int width,
        const int height,
        const int hsize,
        const int vsize)
{
    unsigned int memSize = width * height * sizeof(unsigned char);
    hipMemset(tmp_d, 0, memSize);

    dim3 blockSize;
    blockSize.x = hsize;
    blockSize.y = 1;
    dim3 gridSize;
    gridSize.x = roundUp(width, blockSize.x);
    gridSize.y = roundUp(height, blockSize.y);
    size_t sMemSize = 4 * hsize * sizeof(unsigned char);
    hipLaunchKernelGGL(HIP_KERNEL_NAME(vhgw_horiz<opType>), dim3(gridSize), dim3(blockSize), sMemSize, 0, tmp_d, img_d, width, height, hsize);

    blockSize.x = 1;
    blockSize.y = vsize;
    gridSize.x = roundUp(width, blockSize.x);
    gridSize.y = roundUp(height, blockSize.y);
    sMemSize = 4 * vsize * sizeof(unsigned char);
    hipLaunchKernelGGL(HIP_KERNEL_NAME(vhgw_vert<opType>), dim3(gridSize), dim3(blockSize), sMemSize, 0, img_d, tmp_d, width, height, vsize);
}


extern "C"
void erode(unsigned char* img_d,
        unsigned char* tmp_d,
        const int width,
        const int height,
        const int hsize,
        const int vsize)
{
    morphology<MorphOpType::ERODE>(img_d, tmp_d, width, height, hsize, vsize);
}

extern "C"
void dilate(unsigned char* img_d,
        unsigned char* tmp_d,
        const int width,
        const int height,
        const int hsize,
        const int vsize)
{
    morphology<MorphOpType::DILATE>(img_d, tmp_d, width, height, hsize, vsize);
}
