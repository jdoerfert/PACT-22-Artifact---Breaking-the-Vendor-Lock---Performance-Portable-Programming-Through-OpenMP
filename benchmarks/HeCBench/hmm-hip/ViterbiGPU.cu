#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>

__global__ void
viterbi (const float* maxProbOld, 
         const float* mtState, 
         const float* mtEmit, 
         int *obs, 
         float* maxProbNew, 
         int* path, 
         const int nState,
         const int t)
{
  // find the most probable previous state leading to iState
  int iState = blockDim.x * blockIdx.x + threadIdx.x;
  if (iState < nState) {
    float maxProb = 0.0;
    int maxState = -1;
    for (int preState = 0; preState < nState; preState++) 
    {
      float p = maxProbOld[preState] + mtState[iState*nState + preState];
      if (p > maxProb) 
      {
        maxProb = p;
        maxState = preState;
      }
    }
    maxProbNew[iState] = maxProb + mtEmit[obs[t]*nState+iState];
    path[(t-1)*nState+iState] = maxState;
  }
}

///////////////////////////////////////////////////////////////////////////////
// Using Viterbi algorithm to search for a Hidden Markov Model for the most
// probable state path given the observation sequence.
///////////////////////////////////////////////////////////////////////////////
int ViterbiGPU(float &viterbiProb,
    int   *__restrict__ viterbiPath,
    int   *__restrict__ obs, 
    const int nObs, 
    float *__restrict__ initProb,
    float *__restrict__ mtState, 
    const int nState,
    const int nEmit,
    float *__restrict__ mtEmit)
{

  float maxProbNew[nState];
  int path[(nObs-1)*nState];

  float *d_mtState, *d_mtEmit, *d_maxProbOld, *d_maxProbNew;
  int *d_obs, *d_path;

  hipMalloc((void**)&d_mtState, sizeof(float)*nState*nState);
  hipMalloc((void**)&d_mtEmit, sizeof(float)*nEmit*nState);
  hipMalloc((void**)&d_obs, sizeof(int)*nObs);
  hipMalloc((void**)&d_maxProbOld, sizeof(float)*nState);
  hipMalloc((void**)&d_maxProbNew, sizeof(float)*nState);
  hipMalloc((void**)&d_path, sizeof(int)*(nObs-1)*nState);

  hipMemcpy(d_mtState, mtState, sizeof(float)*nState*nState, hipMemcpyHostToDevice);
  hipMemcpy(d_mtEmit, mtEmit, sizeof(float)*nEmit*nState, hipMemcpyHostToDevice);
  hipMemcpy(d_obs, obs, sizeof(int)*nObs, hipMemcpyHostToDevice);

  dim3 gridDim ((nState + 255)/256);
  dim3 blockDim (256);

  // initial probability
  hipMemcpy(d_maxProbOld, initProb, sizeof(float)*nState, hipMemcpyHostToDevice);

  // main iteration of Viterbi algorithm
  for (int t = 1; t < nObs; t++) // for every input observation
  { 
    hipLaunchKernelGGL(viterbi, gridDim, blockDim, 0, 0, 
              d_maxProbOld, d_mtState, d_mtEmit, d_obs, d_maxProbNew, d_path, nState, t);
    hipMemcpy(d_maxProbOld, d_maxProbNew, sizeof(float)*nState, hipMemcpyDeviceToDevice);
  }
  hipMemcpy(maxProbNew, d_maxProbNew, sizeof(float)*nState, hipMemcpyDeviceToHost);
  hipMemcpy(path, d_path, sizeof(int)*(nObs-1)*nState, hipMemcpyDeviceToHost);

  // find the final most probable state
  float maxProb = 0.0;
  int maxState = -1;
  for (int i = 0; i < nState; i++) 
  {
    if (maxProbNew[i] > maxProb) 
    {
      maxProb = maxProbNew[i];
      maxState = i;
    }
  }
  viterbiProb = maxProb;

  // backtrace to find the Viterbi path
  viterbiPath[nObs-1] = maxState;
  for (int t = nObs-2; t >= 0; t--) 
  {
    viterbiPath[t] = path[t*nState+viterbiPath[t+1]];
  }

  return 1;
}
