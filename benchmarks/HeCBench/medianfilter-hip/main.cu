#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <hip/hip_runtime.h>
#include "shrUtils.h"
#include "MedianFilter.cu"

#ifndef min
#define min(a,b) (a < b ? a : b)
#endif

// Import host computation function 
extern "C" void MedianFilterHost(unsigned int* uiInputImage, unsigned int* uiOutputImage, 
                                 unsigned int uiWidth, unsigned int uiHeight);

void MedianFilterGPU(
    unsigned int* uiInputImage, 
    unsigned int* uiOutputImage, 
    uchar4* cmDevBufIn,
    unsigned int* cmDevBufOut,
    const int uiImageWidth,
    const int uiImageHeight);

int main(int argc, char** argv)
{
  // Image data file
  const char* cPathAndName = argv[1]; 
  unsigned int uiImageWidth = 1920;   // Image width
  unsigned int uiImageHeight = 1080;  // Image height

  size_t szBuffBytes;                 // Size of main image buffers
  size_t szBuffWords;                 

  //char* cPathAndName = NULL;          // var for full paths to data, src, etc.
  unsigned int* uiInput;              // Host input buffer 
  unsigned int* uiOutput;             // Host output buffer

  // One device processes the whole image
  szBuffWords = uiImageHeight * uiImageWidth;
  szBuffBytes = szBuffWords * sizeof (unsigned int);

  uiInput = (unsigned int*) malloc (szBuffBytes);
  uiOutput = (unsigned int*) malloc (szBuffBytes);

  shrLoadPPM4ub(cPathAndName, (unsigned char **)&uiInput, &uiImageWidth, &uiImageHeight);

  printf("Image File\t = %s\nImage Dimensions = %u w x %u h x %lu bpp\n\n", 
         cPathAndName, uiImageWidth, uiImageHeight, sizeof(unsigned int)<<3);

  uchar4* cmDevBufIn;
  hipMalloc((void**)&cmDevBufIn, szBuffBytes);

  unsigned int* cmDevBufOut;
  hipMalloc((void**)&cmDevBufOut, szBuffBytes);

  // Warmup call 
  MedianFilterGPU (uiInput, uiOutput, cmDevBufIn, 
                   cmDevBufOut, uiImageWidth, uiImageHeight);

  // Process n loops on the GPU
  const int iCycles = 150;
  printf("\nRunning MedianFilterGPU for %d cycles...\n\n", iCycles);
  for (int i = 0; i < iCycles; i++)
  {
    MedianFilterGPU (uiInput, uiOutput, cmDevBufIn, 
                     cmDevBufOut, uiImageWidth, uiImageHeight);
  }

  // Compute on host 
  unsigned int* uiGolden = (unsigned int*)malloc(szBuffBytes);
  MedianFilterHost(uiInput, uiGolden, uiImageWidth, uiImageHeight);

  // Compare GPU and Host results:  Allow variance of 1 GV in up to 0.01% of pixels 
  printf("Comparing GPU Result to CPU Result...\n"); 
  shrBOOL bMatch = shrCompareuit(uiGolden, uiOutput, (uiImageWidth * uiImageHeight), 1.0f, 0.0001f);
  printf("\nGPU Result %s CPU Result within tolerance...\n", 
         (bMatch == shrTRUE) ? "matches" : "DOESN'T match"); 

  // Cleanup and exit
  free(uiGolden);
  free(uiInput);
  free(uiOutput);
  hipFree(cmDevBufIn);
  hipFree(cmDevBufOut);

  if(bMatch == shrTRUE) 
    printf("PASS\n");
  else
    printf("FAIL\n");

  return EXIT_SUCCESS;
}

// Copies input data from host buf to the device, runs kernel, 
// copies output data back to output host buf
void MedianFilterGPU(
    unsigned int* uiInputImage, 
    unsigned int* uiOutputImage, 
    uchar4* cmDevBufIn,
    unsigned int* cmDevBufOut,
    const int uiImageWidth,
    const int uiImageHeight)
{
  size_t szGlobalWorkSize[2];         // 2D global work items (ND range) for Median kernel
  size_t szLocalWorkSize[2];          // 2D local work items (work group) for Median kernel
  const int iBlockDimX = 16;
  const int iBlockDimY = 4;
  const int iLocalPixPitch = iBlockDimX + 2;

  hipMemcpy(cmDevBufIn, (uchar4*)uiInputImage, 
    uiImageWidth * uiImageHeight * sizeof(uchar4), hipMemcpyHostToDevice);

  szLocalWorkSize[0] = iBlockDimX;
  szLocalWorkSize[1] = iBlockDimY;
  szGlobalWorkSize[0] = shrRoundUp((int)szLocalWorkSize[0], uiImageWidth); 
  szGlobalWorkSize[1] = shrRoundUp((int)szLocalWorkSize[1], uiImageHeight);

  dim3 lws(szLocalWorkSize[0], szLocalWorkSize[1]);
  dim3 gws(szGlobalWorkSize[0] / szLocalWorkSize[0], 
           szGlobalWorkSize[1] / szLocalWorkSize[1]);

  hipLaunchKernelGGL(ckMedian, dim3(gws), dim3(lws), sizeof(uchar4)*iLocalPixPitch*(iBlockDimY+2), 0, 
       cmDevBufIn, cmDevBufOut, iLocalPixPitch, uiImageWidth, uiImageHeight);

  hipMemcpy((uchar4*)uiOutputImage, cmDevBufOut, 
    uiImageWidth * uiImageHeight * sizeof(uchar4), hipMemcpyDeviceToHost);
}
