#include "hip/hip_runtime.h"
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>

#include "OptionParser.h"
#include "Timer.h"
#include "Utility.h"

// ****************************************************************************
// Function: addBenchmarkSpecOptions
//
// Purpose:
//   Add benchmark specific options parsing
//
// Arguments:
//   op: the options parser / parameter database
//
// Returns:  nothing
//
// Programmer: Kyle Spafford
// Creation: December 15, 2009
//
// Modifications:
//
// ****************************************************************************
void addBenchmarkSpecOptions(OptionParser &op)
{
  ;
}

// ****************************************************************************
// Function: triad
//
// Purpose:
//   A simple vector addition kernel
//   C = A + s*B
//
// Arguments:
//   A,B - input vectors
//   C - output vectors
//   s - scalar
//
// Returns:  nothing
//
// Programmer: Kyle Spafford
// Creation: December 15, 2009
//
// Modifications:
//
// ****************************************************************************
__global__ void triad(float* A, float* B, float* C, float s)
{
  int gid = threadIdx.x + (blockIdx.x * blockDim.x);
  C[gid] = A[gid] + s*B[gid];
}

// ****************************************************************************
// Function: RunBenchmark
//
// Purpose:
//   Implements the Stream Triad benchmark in CUDA.  This benchmark
//   is designed to test CUDA's overall data transfer speed. It executes
//   a vector addition operation with no temporal reuse. Data is read
//   directly from the global memory. This implementation tiles the input
//   array and pipelines the vector addition computation with
//   the data download for the next tile. However, since data transfer from
//   host to device is much more expensive than the simple vector computation,
//   data transfer operations should completely dominate the execution time.
//
// Arguments:
//   resultDB: results from the benchmark are stored in this db
//   op: the options parser (contains input parameters)
//
// Returns:  nothing
//
// Programmer: Kyle Spafford
// Creation: December 15, 2009
//
// Modifications:
//
// ****************************************************************************
void RunBenchmark(OptionParser &op)
{
  auto start = std::chrono::high_resolution_clock::now();

  const bool verbose = op.getOptionBool("verbose");
  const int n_passes = op.getOptionInt("passes");

  // 256k through 8M bytes
  const int nSizes = 9;
  const size_t blockSizes[] = { 64, 128, 256, 512, 1024, 2048, 4096, 8192,
    16384 };
  const size_t memSize = 16384;
  const size_t numMaxFloats = 1024 * memSize / 4;
  const size_t halfNumFloats = numMaxFloats / 2;

  // Create some host memory pattern
  srand48(8650341L);
  float *h_mem;
  hipHostMalloc((void**)&h_mem, sizeof(float) * numMaxFloats);

  // Allocate some device memory
  float* d_memA0, *d_memB0, *d_memC0;
  hipMalloc((void**) &d_memA0, blockSizes[nSizes - 1] * 1024);
  hipMalloc((void**) &d_memB0, blockSizes[nSizes - 1] * 1024);
  hipMalloc((void**) &d_memC0, blockSizes[nSizes - 1] * 1024);

  float* d_memA1, *d_memB1, *d_memC1;
  hipMalloc((void**) &d_memA1, blockSizes[nSizes - 1] * 1024);
  hipMalloc((void**) &d_memB1, blockSizes[nSizes - 1] * 1024);
  hipMalloc((void**) &d_memC1, blockSizes[nSizes - 1] * 1024);

  float scalar = 1.75f;

  const size_t blockSize = 128;

  // Number of passes. Use a large number for stress testing.
  // A small value is sufficient for computing sustained performance.
  for (int pass = 0; pass < n_passes; ++pass)
  {
    // Step through sizes forward
    for (int i = 0; i < nSizes; ++i)
    {
      int elemsInBlock = blockSizes[i] * 1024 / sizeof(float);
      for (int j = 0; j < halfNumFloats; ++j)
        h_mem[j] = h_mem[halfNumFloats + j]
          = (float) (drand48() * 10.0);

      // Copy input memory to the device
      if (verbose) {
        cout << ">> Executing Triad with vectors of length "
          << numMaxFloats << " and block size of "
          << elemsInBlock << " elements." << "\n";
        printf("Block:%05ldKB\n", blockSizes[i]);
      }

      // start submitting blocks of data of size elemsInBlock
      // overlap the computation of one block with the data
      // download for the next block and the results upload for
      // the previous block
      int crtIdx = 0;
      size_t globalWorkSize = elemsInBlock / blockSize;

      hipStream_t streams[2];
      hipStreamCreate(&streams[0]);
      hipStreamCreate(&streams[1]);

      int TH = Timer::Start();

      hipMemcpyAsync(d_memA0, h_mem, blockSizes[i] * 1024,
          hipMemcpyHostToDevice, streams[0]);
      hipMemcpyAsync(d_memB0, h_mem, blockSizes[i] * 1024,
          hipMemcpyHostToDevice, streams[0]);

      hipLaunchKernelGGL(triad, dim3(globalWorkSize), dim3(blockSize), 0, streams[0], d_memA0, d_memB0, d_memC0, scalar);

      if (elemsInBlock < numMaxFloats)
      {
        // start downloading data for next block
        hipMemcpyAsync(d_memA1, h_mem + elemsInBlock, blockSizes[i]
            * 1024, hipMemcpyHostToDevice, streams[1]);
        hipMemcpyAsync(d_memB1, h_mem + elemsInBlock, blockSizes[i]
            * 1024, hipMemcpyHostToDevice, streams[1]);
      }

      int blockIdx = 1;
      unsigned int currStream = 1;
      while (crtIdx < numMaxFloats)
      {
        currStream = blockIdx & 1;
        // Start copying back the answer from the last kernel
        if (currStream)
        {
          hipMemcpyAsync(h_mem + crtIdx, d_memC0, elemsInBlock
              * sizeof(float), hipMemcpyDeviceToHost, streams[0]);
        }
        else
        {
          hipMemcpyAsync(h_mem + crtIdx, d_memC1, elemsInBlock
              * sizeof(float), hipMemcpyDeviceToHost, streams[1]);
        }

        crtIdx += elemsInBlock;

        if (crtIdx < numMaxFloats)
        {
          // Execute the kernel
          if (currStream)
          {
            hipLaunchKernelGGL(triad, dim3(globalWorkSize), dim3(blockSize), 0, streams[1], d_memA1, d_memB1, d_memC1, scalar);
          }
          else
          {
            hipLaunchKernelGGL(triad, dim3(globalWorkSize), dim3(blockSize), 0, streams[0], d_memA0, d_memB0, d_memC0, scalar);
          }
        }

        if (crtIdx+elemsInBlock < numMaxFloats)
        {
          // Download data for next block
          if (currStream)
          {
            hipMemcpyAsync(d_memA0, h_mem+crtIdx+elemsInBlock,
                blockSizes[i]*1024, hipMemcpyHostToDevice,
                streams[0]);
            hipMemcpyAsync(d_memB0, h_mem+crtIdx+elemsInBlock,
                blockSizes[i]*1024, hipMemcpyHostToDevice,
                streams[0]);
          }
          else
          {
            hipMemcpyAsync(d_memA1, h_mem+crtIdx+elemsInBlock,
                blockSizes[i]*1024, hipMemcpyHostToDevice,
                streams[1]);
            hipMemcpyAsync(d_memB1, h_mem+crtIdx+elemsInBlock,
                blockSizes[i]*1024, hipMemcpyHostToDevice,
                streams[1]);
          }
        }
        blockIdx += 1;
        currStream = !currStream;
      }

      hipDeviceSynchronize();
      double time = Timer::Stop(TH, "thread synchronize");

      double triad = ((double)numMaxFloats * 2.0) / (time*1e9);
      if (verbose)
        std::cout << "TriadFlops " << triad << " GFLOPS/s\n";

      double bdwth = ((double)numMaxFloats*sizeof(float)*3.0)
        / (time*1000.*1000.*1000.);
      if (verbose)
        std::cout << "TriadBdwth " << bdwth << " GB/s\n";


      // Checking memory for correctness. The two halves of the array
      // should have the same results.
      if (verbose) cout << ">> checking memory\n";
      for (int j=0; j<halfNumFloats; ++j)
      {
        if (h_mem[j] != h_mem[j+halfNumFloats])
        {
          cout << "Error; hostMem[" << j << "]=" << h_mem[j]
            << " is different from its twin element hostMem["
            << (j+halfNumFloats) << "]: "
            << h_mem[j+halfNumFloats] << "stopping check\n";
          break;
        }
      }
      if (verbose) cout << ">> finish!" << endl;

      // Zero out the test host memory
      for (int j=0; j<numMaxFloats; ++j)
        h_mem[j] = 0.0f;
    }
  }

  // Cleanup
  hipFree(d_memA0);
  hipFree(d_memB0);
  hipFree(d_memC0);
  hipFree(d_memA1);
  hipFree(d_memB1);
  hipFree(d_memC1);
  hipHostFree(h_mem);

  auto end = std::chrono::high_resolution_clock::now();
  auto elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
  printf("Total execution time (function 'RunBechmark') (in ms): %ld \n", elapsed);
}
