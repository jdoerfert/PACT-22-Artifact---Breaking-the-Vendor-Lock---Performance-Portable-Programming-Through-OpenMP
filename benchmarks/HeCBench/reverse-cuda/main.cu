#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <random>
#include <hip/hip_runtime.h>

__global__ void reverse (int *d, const int len)
{
  __shared__ int s[256];
  int t = threadIdx.x;
  s[t] = d[t];
  __syncthreads();
  d[t] = s[len-t-1];
}

int main(int argc, char* argv[]) {

  if (argc != 2) {
    printf("Usage: ./%s <iterations>\n", argv[0]);
    return 1;
  }

  // specify the number of test cases
  const int iteration = atoi(argv[1]);

  // number of elements to reverse
  const int len = 256;
  const int elem_size = len * sizeof(int);

  // save device result
  int test[len];

  // save expected results after performing preverse operations even/odd times
  int error = 0;
  int gold_odd[len];
  int gold_even[len];

  for (int i = 0; i < len; i++) {
    gold_odd[i] = len-i-1;
    gold_even[i] = i;
  }

  int *d_test;
  hipMalloc((void**)&d_test, elem_size);

  std::default_random_engine generator (123);
  // bound the number of reverse operations
  std::uniform_int_distribution<int> distribution(100, 9999);

  for (int i = 0; i < iteration; i++) {
    const int count = distribution(generator);

    hipMemcpy(d_test, gold_even, elem_size, hipMemcpyHostToDevice);

    for (int j = 0; j < count; j++)
      reverse<<<1, len>>> (d_test, len);

    hipMemcpy(test, d_test, elem_size, hipMemcpyDeviceToHost);

    if (count % 2 == 0)
      error = memcmp(test, gold_even, elem_size);
    else
      error = memcmp(test, gold_odd, elem_size);
    
    if (error) break;
  }
  
  printf("%s\n", error ? "FAIL" : "PASS");

  hipFree(d_test);
  return 0;
}
