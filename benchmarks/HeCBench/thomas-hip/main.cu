#include <iostream>
#include <hip/hip_runtime.h>
#include "ThomasMatrix.hpp"
#include "utils.hpp"
#include "cuThomasBatch.h"

// CPU kernel
void solve_seq(const double* l, const double* d, double* u, double* rhs, const int n, const int N) 
{
  int first,last;
  for (int j = 0; j < N; ++j)
  {
    first = j*n;
    last = first + n - 1;

    u[first] /= d[first];
    rhs[first] /= d[first];

    for (int i = first+1; i < last; i++) {
      u[i] /= d[i] - l[i]*u[i-1];
      rhs[i] = (rhs[i] - l[i]*rhs[i-1]) / (d[i] - l[i]*u[i-1]);
    }

    rhs[last] = (rhs[last] - l[last]*rhs[last-1]) / (d[last] - l[last]*u[last-1]);

    for (int i = last-1; i >= first; i--) {
      rhs[i] -= u[i]*rhs[i+1];
    }
  }
}

int main(int argc, char const *argv[])
{

  if(argc < 4 or argc > 4){
    std::cout << "Usage: ./run [system size] [#systems] [thread block size]" << std::endl;
    return -1;
  }

  const int M = std::stoi(argv[1]); // c++11
  const int N = std::stoi(argv[2]);
  const int BlockSize  = std::stoi(argv[3]);  // GPU thread block size

  const int matrix_byte_size = M * N * sizeof(double);

  //Loading a synthetic tridiagonal matrix into our structure
  ThomasMatrix params = loadThomasMatrixSyn(M);

  double* u_seq = (double*) malloc(matrix_byte_size);
  double* u_Thomas_host =  (double*) malloc(matrix_byte_size);
  double* u_input = (double*) malloc(matrix_byte_size);

  double* d_seq = (double*) malloc(matrix_byte_size);
  double* d_Thomas_host =  (double*) malloc(matrix_byte_size);
  double* d_input = (double*) malloc(matrix_byte_size);

  double* l_seq = (double*) malloc(matrix_byte_size);
  double* l_Thomas_host =  (double*) malloc(matrix_byte_size);
  double* l_input = (double*) malloc(matrix_byte_size);

  double* rhs_seq = (double*) malloc(matrix_byte_size);
  double* rhs_Thomas_host = (double*) malloc(matrix_byte_size);
  double* rhs_input = (double*) malloc(matrix_byte_size);

  double* rhs_seq_output = (double*) malloc(matrix_byte_size);
  double* rhs_seq_interleave = (double*) malloc(matrix_byte_size);

  for (int i = 0; i < N; ++i)
  {
    for (int j = 0; j < M; ++j)
    {
      u_seq[(i * M) + j] = params.a[j];
      u_input[(i * M) + j] = params.a[j];

      d_seq[(i * M) + j] = params.d[j];
      d_input[(i * M) + j] = params.d[j];

      l_seq[(i * M) + j] = params.b[j];
      l_input[(i * M) + j] = params.b[j];

      rhs_seq[(i * M) + j] = params.rhs[j];
      rhs_input[(i * M) + j] = params.rhs[j];

    }
  }

  // Sequantial CPU Execution for correct error check
  for (int n = 0; n < 100; n++) {
    solve_seq( l_seq, d_seq, u_seq, rhs_seq, M, N );
  }

  for (int i = 0; i < M*N; ++i) {
    rhs_seq_output[i] = rhs_seq[i];
  }

  // initialize again because u_seq and rhs_seq are modified by solve_seq
  for (int i = 0; i < N; ++i)
  {
    for (int j = 0; j < M; ++j)
    {
      u_seq[(i * M) + j] = params.a[j];
      u_input[(i * M) + j] = params.a[j];

      d_seq[(i * M) + j] = params.d[j];
      d_input[(i * M) + j] = params.d[j];

      l_seq[(i * M) + j] = params.b[j];
      l_input[(i * M) + j] = params.b[j];

      rhs_seq[(i * M) + j] = params.rhs[j];
      rhs_input[(i * M) + j] = params.rhs[j];

    }
  }


  // transpose the inputs for sequential accesses on a GPU 
  for (int i = 0; i < M; ++i)
  {
    for (int j = 0; j < N; ++j)
    {
      u_Thomas_host[i*N+j] = u_input[j*M+i];
      l_Thomas_host[i*N+j] = l_input[j*M+i];
      d_Thomas_host[i*N+j] = d_input[j*M+i];
      rhs_Thomas_host[i*N+j] = rhs_input[j*M+i];
      rhs_seq_interleave[i*N+j] = rhs_seq_output[j*M+i];

    }
  }

 
  // Run GPU kernel

  double *u_device;
  double *d_device;
  double *l_device;
  double *rhs_device;

  hipMalloc((void**)&u_device, matrix_byte_size);
  hipMalloc((void**)&l_device, matrix_byte_size);
  hipMalloc((void**)&d_device, matrix_byte_size);
  hipMalloc((void**)&rhs_device, matrix_byte_size);

  hipMemcpyAsync(u_device, u_Thomas_host, matrix_byte_size, hipMemcpyHostToDevice, 0);
  hipMemcpyAsync(l_device, l_Thomas_host, matrix_byte_size, hipMemcpyHostToDevice, 0);
  hipMemcpyAsync(d_device, d_Thomas_host, matrix_byte_size, hipMemcpyHostToDevice, 0);
  hipMemcpyAsync(rhs_device, rhs_Thomas_host, matrix_byte_size, hipMemcpyHostToDevice,  0);
  for (int n = 0; n < 100; n++) {
    hipLaunchKernelGGL(cuThomasBatch, dim3((N/BlockSize)+1), dim3(BlockSize), 0, 0, l_device, d_device, u_device, rhs_device, M, N);
  }
  hipMemcpyAsync(rhs_Thomas_host, rhs_device, matrix_byte_size, hipMemcpyDeviceToHost, 0);
  hipDeviceSynchronize();

  // verify
  calcError(rhs_seq_interleave,rhs_Thomas_host,N*M);


  free(u_seq);  
  free(u_Thomas_host);
  free(u_input);

  free(d_seq);  
  free(d_Thomas_host);
  free(d_input);

  free(l_seq);  
  free(l_Thomas_host);
  free(l_input);

  free(rhs_seq);  
  free(rhs_Thomas_host);
  free(rhs_input);

  free(rhs_seq_output);
  free(rhs_seq_interleave);

  hipFree(l_device);
  hipFree(d_device);
  hipFree(u_device);
  hipFree(rhs_device);

  return 0;

}


