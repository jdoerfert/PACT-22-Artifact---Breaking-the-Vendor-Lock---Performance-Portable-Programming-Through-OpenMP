#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <hip/hip_runtime.h>

#define COL_P_X 0
#define COL_P_Y 1
#define COL_P_Z 2
#define COL_N_X 3
#define COL_N_Y 4
#define COL_N_Z 5
#define COL_RSq 6
#define COL_DIM 7

// compute the xyz images using the inverse focal length invF
template<typename T>
__global__ void surfel_render(
  const T *__restrict__ s,
  int N,
  T f,
  int w,
  int h,
  T *__restrict__ d)
{
  const int idx = threadIdx.x + blockIdx.x*blockDim.x;
  const int idy = threadIdx.y + blockIdx.y*blockDim.y;

  if(idx < w && idy < h)
  {
    T ray[3];
    ray[0] = T(idx)-(w-1)*(T)0.5;
    ray[1] = T(idy)-(h-1)*(T)0.5;
    ray[2] = f;
    T pt[3];
    T n[3];
    T p[3];
    T dMin = 1e20;
    
    for (int i=0; i<N; ++i) {
      p[0] = s[i*COL_DIM+COL_P_X];
      p[1] = s[i*COL_DIM+COL_P_Y];
      p[2] = s[i*COL_DIM+COL_P_Z];
      n[0] = s[i*COL_DIM+COL_N_X];
      n[1] = s[i*COL_DIM+COL_N_Y];
      n[2] = s[i*COL_DIM+COL_N_Z];
      T rSqMax = s[i*COL_DIM+COL_RSq];
      T pDotn = p[0]*n[0]+p[1]*n[1]+p[2]*n[2];
      T dsDotRay = ray[0]*n[0] + ray[1]*n[1] + ray[2]*n[2];
      T alpha = pDotn / dsDotRay;
      pt[0] = ray[0]*alpha - p[0];
      pt[1] = ray[1]*alpha - p[1];
      pt[2] = ray[2]*alpha - p[2];
      T t = ray[2]*alpha;
      T rSq = pt[0] * pt[0] + pt[1] * pt[1] + pt[2] * pt[2];
      if (rSq < rSqMax && dMin > t) {
        dMin = t; // ray hit the surfel 
      }
    }
    d[idy*w+idx] = dMin > (T)100 ? (T)0 : dMin;
  }
}

template <typename T>
void surfelRenderTest(int n, int w, int h)
{
  const int src_size = n*7;
  const int dst_size = w*h;

  T *d_src, *d_dst;
  hipMalloc((void**)&d_dst, dst_size * sizeof(T));
  hipMalloc((void**)&d_src, src_size * sizeof(T));

  T *h_dst = (T*) malloc (dst_size * sizeof(T));
  T *h_src = (T*) malloc (src_size * sizeof(T));

  srand(123);
  for (int i = 0; i < src_size; i++)
    h_src[i] = rand() % 256;

  T inverseFocalLength[3] = {0.005, 0.02, 0.036};

  hipMemcpy(d_src, h_src, src_size * sizeof(T), hipMemcpyHostToDevice); 

  dim3 threads(16, 16);
  dim3 blocks((w+15)/16, (h+15)/16);
  for (int f = 0; f < 3; f++) {
    for (int i = 0; i < 100; i++)
      hipLaunchKernelGGL(HIP_KERNEL_NAME(surfel_render<T>), blocks, threads, 
                         0, 0, d_src, n, inverseFocalLength[f], w, h, d_dst);

    hipMemcpy(h_dst, d_dst, dst_size * sizeof(T), hipMemcpyDeviceToHost); 
    T *min = std::min_element( h_dst, h_dst + w*h );
    T *max = std::max_element( h_dst, h_dst + w*h );
    printf("value range [%e, %e]\n", *min, *max);
  }

  free(h_dst);
  free(h_src);
  hipFree(d_dst);
  hipFree(d_src);
}

int main(int argc, char *argv[]) {
  int n = atoi(argv[1]);
  int w = atoi(argv[2]);
  int h = atoi(argv[3]);
  surfelRenderTest<float>(n, w, h);
  surfelRenderTest<double>(n, w, h);
  return 0;
}
