#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "SDKBitMap.h"
#include "aes.h"
#include "kernels.cu"
#include "reference.cu"
#include "utils.cu"

int main(int argc, char * argv[])
{
  const unsigned int keySizeBits = 128;
  const unsigned int rounds = 10;
  const unsigned int seed = 123;

  const int iterations = atoi(argv[1]);
  const bool decrypt = atoi(argv[2]);
  const char* filePath = argv[3];

  SDKBitMap image;
  image.load(filePath);
  const int width  = image.getWidth();
  const int height = image.getHeight();

  /* check condition for the bitmap to be initialized */
  if (width <= 0 || height <= 0) return 1;

  std::cout << "Image width and height: " 
            << width << " " << height << std::endl;

  uchar4 *pixels = image.getPixels();

  unsigned int sizeBytes = width*height*sizeof(uchar);
  uchar *input = (uchar*)malloc(sizeBytes); 

  /* initialize the input array, do NOTHING but assignment when decrypt*/
  if (decrypt)
    convertGrayToGray(pixels, input, height, width);
  else
    convertColorToGray(pixels, input, height, width);

  unsigned int keySize = keySizeBits/8; // 1 Byte = 8 bits

  unsigned int keySizeBytes = keySize*sizeof(uchar);

  uchar *key = (uchar*)malloc(keySizeBytes);

  fillRandom<uchar>(key, keySize, 1, 0, 255, seed); 

  // expand the key
  unsigned int explandedKeySize = (rounds+1)*keySize;
  uchar *expandedKey = (uchar*)malloc(explandedKeySize*sizeof(uchar));
  uchar *roundKey    = (uchar*)malloc(explandedKeySize*sizeof(uchar));

  keyExpansion(key, expandedKey, keySize, explandedKeySize);
  for(unsigned int i = 0; i < rounds+1; ++i)
  {
    createRoundKey(expandedKey + keySize*i, roundKey + keySize*i);
  }

  // save device result
  uchar* output = (uchar*)malloc(sizeBytes);

  uchar *inputBuffer;
  hipMalloc((void**)&inputBuffer, width * height);
  hipMemcpy(inputBuffer, input, width * height, hipMemcpyHostToDevice);

  uchar *outputBuffer;
  hipMalloc((void**)&outputBuffer, width * height);

  uchar *rKeyBuffer;
  hipMalloc((void**)&rKeyBuffer, explandedKeySize);
  hipMemcpy(rKeyBuffer, roundKey, explandedKeySize, hipMemcpyHostToDevice);

  uchar *sBoxBuffer;
  hipMalloc((void**)&sBoxBuffer, 256);
  hipMemcpy(sBoxBuffer, sbox, 256, hipMemcpyHostToDevice);

  uchar *rsBoxBuffer;
  hipMalloc((void**)&rsBoxBuffer, 256);
  hipMemcpy(rsBoxBuffer, rsbox, 256, hipMemcpyHostToDevice);

  std::cout << "Executing kernel for " << iterations 
            << " iterations" << std::endl;
  std::cout << "-------------------------------------------" << std::endl;

  dim3 grid (width/4, height/4);
  dim3 block (1, 4);

  for(int i = 0; i < iterations; i++)
  {
    if (decrypt) 
      AESDecrypt<<< grid, block >>>(
        (uchar4*)outputBuffer,
        (uchar4*)inputBuffer,
        (uchar4*)rKeyBuffer,
        rsBoxBuffer,
        width, rounds);
    else
      AESEncrypt<<< grid, block >>>(
        (uchar4*)outputBuffer,
        (uchar4*)inputBuffer,
        (uchar4*)rKeyBuffer,
        sBoxBuffer,
        width, rounds);

    hipMemcpy(output, outputBuffer, width * height, hipMemcpyDeviceToHost);
  }

  // Verify
  uchar *verificationOutput = (uchar *) malloc(width*height*sizeof(uchar));

  reference(verificationOutput, input, roundKey, explandedKeySize, 
      width, height, decrypt, rounds, keySize);

  /* compare the results and see if they match */
  if(memcmp(output, verificationOutput, height*width*sizeof(uchar)) == 0)
    std::cout<<"Passed!\n";
  else
    std::cout<<"Failed\n";

  /* release program resources (input memory etc.) */
  hipFree(inputBuffer);
  hipFree(outputBuffer);
  hipFree(rKeyBuffer);
  hipFree(sBoxBuffer);
  hipFree(rsBoxBuffer);

  if(input) free(input);

  if(key) free(key);

  if(expandedKey) free(expandedKey);

  if(roundKey) free(roundKey);

  if(output) free(output);

  if(verificationOutput) free(verificationOutput);

  return 0;
}

