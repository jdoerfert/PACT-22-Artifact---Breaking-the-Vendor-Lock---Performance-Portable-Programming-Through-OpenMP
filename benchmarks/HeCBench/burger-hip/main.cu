#include "hip/hip_runtime.h"
/*
   MIT License

   Copyright (c) 2020 Soumya Sen

   Permission is hereby granted, free of charge, to any person obtaining a copy
   of this software and associated documentation files (the "Software"), to deal
   in the Software without restriction, including without limitation the rights
   to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
   copies of the Software, and to permit persons to whom the Software is
   furnished to do so, subject to the following conditions:

   The above copyright notice and this permission notice shall be included in all
   copies or substantial portions of the Software.

   THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
   IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
   FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
   AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
   LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
   OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
   SOFTWARE.
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define idx(i,j)   (i)*y_points+(j)

__global__ 
void core (
    double *__restrict__ u_new,
    double *__restrict__ v_new,
    const double *__restrict__ u,
    const double *__restrict__ v,
    const int x_points,
    const int y_points,
    const double nu,
    const double del_t,
    const double del_x,
    const double del_y)
{
  int i = blockIdx.y * blockDim.y + threadIdx.y + 1;
  int j = blockIdx.x * blockDim.x + threadIdx.x + 1;
  if (j < x_points - 1 && i < y_points - 1) {
    u_new[idx(i,j)] = u[idx(i,j)] + 
      (nu*del_t/(del_x*del_x)) * (u[idx(i,j+1)] + u[idx(i,j-1)] - 2 * u[idx(i,j)]) + 
      (nu*del_t/(del_y*del_y)) * (u[idx(i+1,j)] + u[idx(i-1,j)] - 2 * u[idx(i,j)]) - 
      (del_t/del_x)*u[idx(i,j)] * (u[idx(i,j)] - u[idx(i,j-1)]) - 
      (del_t/del_y)*v[idx(i,j)] * (u[idx(i,j)] - u[idx(i-1,j)]);

    v_new[idx(i,j)] = v[idx(i,j)] +
      (nu*del_t/(del_x*del_x)) * (v[idx(i,j+1)] + v[idx(i,j-1)] - 2 * v[idx(i,j)]) + 
      (nu*del_t/(del_y*del_y)) * (v[idx(i+1,j)] + v[idx(i-1,j)] - 2 * v[idx(i,j)]) -
      (del_t/del_x)*u[idx(i,j)] * (v[idx(i,j)] - v[idx(i,j-1)]) - 
      (del_t/del_y)*v[idx(i,j)] * (v[idx(i,j)] - v[idx(i-1,j)]);
  }
}

__global__ 
void bound_h (
    double *__restrict__ u_new,
    double *__restrict__ v_new,
    const int x_points,
    const int y_points)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < x_points) {
    u_new[idx(0,i)] = 1.0;
    v_new[idx(0,i)] = 1.0;
    u_new[idx(y_points-1,i)] = 1.0;
    v_new[idx(y_points-1,i)] = 1.0;
  }
}

__global__ 
void bound_v (
    double *__restrict__ u_new,
    double *__restrict__ v_new,
    const int x_points,
    const int y_points)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  if (j < y_points) {
    u_new[idx(j,0)] = 1.0;
    v_new[idx(j,0)] = 1.0;
    u_new[idx(j,x_points-1)] = 1.0;
    v_new[idx(j,x_points-1)] = 1.0;
  }
}

__global__ 
void update (
    double *__restrict__ u,
    double *__restrict__ v,
    const double *__restrict__ u_new,
    const double *__restrict__ v_new,
    const int n)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    u[i] = u_new[i];
    v[i] = v_new[i];
  }
}

int main(int argc, char* argv[])
{
  if (argc != 3) {
    printf("Usage: %s dimx dimy\n", argv[0]);
    exit(-1);
  }

  // Define the domain
  const int x_points = atoi(argv[1]);
  const int y_points = atoi(argv[2]);
  const double x_len = 2.0;
  const double y_len = 2.0;
  const double del_x = x_len/(x_points-1);
  const double del_y = y_len/(y_points-1);

  const int grid_elems = x_points * y_points;
  const int grid_size = sizeof(double) * grid_elems;

  double *x = (double*) malloc (sizeof(double) * x_points);
  double *y = (double*) malloc (sizeof(double) * y_points);
  double *u = (double*) malloc (grid_size);
  double *v = (double*) malloc (grid_size);
  double *u_new = (double*) malloc (grid_size);
  double *v_new = (double*) malloc (grid_size);

  // store device results
  double *du = (double*) malloc (grid_size);
  double *dv = (double*) malloc (grid_size);

  // Define the parameters
  const int num_itrs = 100;     // Number of time iterations
  const double nu = 0.01;
  const double sigma = 0.0009;
  const double del_t = sigma * del_x * del_y / nu;      // CFL criteria

  printf("2D Burger's equation\n");
  printf("Grid dimension: x = %d y = %d\n", x_points, y_points);

  for(int i = 0; i < x_points; i++) x[i] = i * del_x;
  for(int i = 0; i < y_points; i++) y[i] = i * del_y;

  for(int i = 0; i < y_points; i++){
    for(int j = 0; j < x_points; j++){
      u[idx(i,j)] = 1.0;
      v[idx(i,j)] = 1.0;
      u_new[idx(i,j)] = 1.0;
      v_new[idx(i,j)] = 1.0;

      if(x[j] > 0.5 && x[j] < 1.0 && y[i] > 0.5 && y[i] < 1.0){
        u[idx(i,j)] = 2.0;
        v[idx(i,j)] = 2.0;
        u_new[idx(i,j)] = 2.0;
        v_new[idx(i,j)] = 2.0;
      }
    }
  }

  double *d_u_new;
  hipMalloc((void**)&d_u_new, grid_size);

  double *d_v_new;
  hipMalloc((void**)&d_v_new, grid_size);

  double *d_u;
  hipMalloc((void**)&d_u, grid_size);

  double *d_v;
  hipMalloc((void**)&d_v, grid_size);

  hipMemcpy(d_u_new, u_new, grid_size, hipMemcpyHostToDevice);
  hipMemcpy(d_v_new, v_new, grid_size, hipMemcpyHostToDevice);
  hipMemcpy(d_u, u, grid_size, hipMemcpyHostToDevice);
  hipMemcpy(d_v, v, grid_size, hipMemcpyHostToDevice);

  // ranges of the four kernels
  dim3 grid ((x_points-2+15)/16, (y_points-2+15)/16);
  dim3 block (16, 16);
  dim3 grid2 ((x_points+255)/256);
  dim3 block2 (256);
  dim3 grid3 ((y_points+255)/256);
  dim3 block3 (256);
  dim3 grid4 ((grid_elems+255)/256);
  dim3 block4 (256);

  for(int itr = 0; itr < num_itrs; itr++){

    hipLaunchKernelGGL(core, dim3(grid), dim3(block), 0, 0, d_u_new, d_v_new, d_u, d_v, x_points, y_points, nu, del_t, del_x, del_y);

    // Boundary conditions
    hipLaunchKernelGGL(bound_h, dim3(grid2), dim3(block2), 0, 0, d_u_new, d_v_new, x_points, y_points);

    hipLaunchKernelGGL(bound_v, dim3(grid3), dim3(block3), 0, 0, d_u_new, d_v_new, x_points, y_points);

    // Updating older values to newer ones
    hipLaunchKernelGGL(update, dim3(grid4), dim3(block4), 0, 0, d_u, d_v, d_u_new, d_v_new, grid_elems);
  }

  hipMemcpy(du, d_u, grid_size, hipMemcpyDeviceToHost);
  hipMemcpy(dv, d_v, grid_size, hipMemcpyDeviceToHost);

  printf("Serial computing for verification...\n");

  // Reset velocities
  for(int i = 0; i < y_points; i++){
    for(int j = 0; j < x_points; j++){
      u[idx(i,j)] = 1.0;
      v[idx(i,j)] = 1.0;
      u_new[idx(i,j)] = 1.0;
      v_new[idx(i,j)] = 1.0;

      if(x[j] > 0.5 && x[j] < 1.0 && y[i] > 0.5 && y[i] < 1.0){
        u[idx(i,j)] = 2.0;
        v[idx(i,j)] = 2.0;
        u_new[idx(i,j)] = 2.0;
        v_new[idx(i,j)] = 2.0;
      }
    }
  }

  for(int itr = 0; itr < num_itrs; itr++){

    for(int i = 1; i < y_points-1; i++){
      for(int j = 1; j < x_points-1; j++){
        u_new[idx(i,j)] = u[idx(i,j)] + (nu*del_t/(del_x*del_x)) * (u[idx(i,j+1)] + u[idx(i,j-1)] - 2 * u[idx(i,j)]) + 
          (nu*del_t/(del_y*del_y)) * (u[idx(i+1,j)] + u[idx(i-1,j)] - 2 * u[idx(i,j)]) - 
          (del_t/del_x)*u[idx(i,j)] * (u[idx(i,j)] - u[idx(i,j-1)]) - 
          (del_t/del_y)*v[idx(i,j)] * (u[idx(i,j)] - u[idx(i-1,j)]);

        v_new[idx(i,j)] = v[idx(i,j)] + (nu*del_t/(del_x*del_x)) * (v[idx(i,j+1)] + v[idx(i,j-1)] - 2 * v[idx(i,j)]) + 
          (nu*del_t/(del_y*del_y)) * (v[idx(i+1,j)] + v[idx(i-1,j)] - 2 * v[idx(i,j)]) -
          (del_t/del_x)*u[idx(i,j)] * (v[idx(i,j)] - v[idx(i,j-1)]) - 
          (del_t/del_y)*v[idx(i,j)] * (v[idx(i,j)] - v[idx(i-1,j)]);
      }
    }

    // Boundary conditions
    for(int i = 0; i < x_points; i++){
      u_new[idx(0,i)] = 1.0;
      v_new[idx(0,i)] = 1.0;
      u_new[idx(y_points-1,i)] = 1.0;
      v_new[idx(y_points-1,i)] = 1.0;
    }

    for(int j = 0; j < y_points; j++){
      u_new[idx(j,0)] = 1.0;
      v_new[idx(j,0)] = 1.0;
      u_new[idx(j,x_points-1)] = 1.0;
      v_new[idx(j,x_points-1)] = 1.0;
    }

    // Updating older values to newer ones
    for(int i = 0; i < y_points; i++){
      for(int j = 0; j < x_points; j++){
        u[idx(i,j)] = u_new[idx(i,j)];
        v[idx(i,j)] = v_new[idx(i,j)];
      }
    }
  }

  bool ok = true;
  for(int i = 0; i < y_points; i++){
    for(int j = 0; j < x_points; j++){
      if (fabs(du[idx(i,j)] - u[idx(i,j)]) > 1e-6 || 
          fabs(dv[idx(i,j)] - v[idx(i,j)]) > 1e-6) ok = false;
    }
  }
  printf("%s\n", ok ? "PASS" : "FAIL");

  free(x);
  free(y);
  free(u);
  free(v);
  free(du);
  free(dv);
  free(u_new);
  free(v_new);
  hipFree(d_u);
  hipFree(d_v);
  hipFree(d_u_new);
  hipFree(d_v_new);

  return 0;
}
