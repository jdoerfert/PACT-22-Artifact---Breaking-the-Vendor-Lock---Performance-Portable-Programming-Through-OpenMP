#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "reference.cpp"

__global__ void smoothingFilter(
    int Lx, int Ly, 
    int Threshold, int MaxRad, 
    const float*__restrict Img,
            int*__restrict Box,
          float*__restrict Norm)
{
  int tid = threadIdx.x;
  int tjd = threadIdx.y;
  int i = blockIdx.x * blockDim.x + tid;
  int j = blockIdx.y * blockDim.y + tjd;
  int stid = tjd * blockDim.x + tid;
  int gtid = j * Lx + i;  

  // part of shared memory may be unused
  __shared__ float s_Img[1024];

  if ( i < Lx && j < Ly )
    s_Img[stid] = Img[gtid];

  __syncthreads();

  if ( i < Lx && j < Ly )
  {
    // Smoothing parameters
    float sum = 0.f;
    int q = 1;
    int s = q;
    int ksum = 0;

    // Continue until parameters are met
    while (sum < Threshold && q < MaxRad)
    {
      s = q;
      sum = 0.f;
      ksum = 0;

      // Normal adaptive smoothing
      for (int ii = -s; ii < s+1; ii++)
        for (int jj = -s; jj < s+1; jj++)
          if ( (i-s >= 0) && (i+s < Ly) && (j-s >= 0) && (j+s < Lx) )
          {
            ksum++;
            // Compute within bounds of block dimensions
            if( tid-s >= 0 && tid+s < blockDim.x && tjd-s >= 0 && tjd+s < blockDim.y )
              sum += s_Img[stid + ii*blockDim.x + jj];
            // Compute block borders with global memory
            else
              sum += Img[gtid + ii*Lx + jj];
          }
      q++;
    }
    Box[gtid] = s;

    // Normalization for each box
    for (int ii = -s; ii < s+1; ii++)
      for (int jj = -s; jj < s+1; jj++)
        if (ksum != 0) 
          atomicAdd(&Norm[gtid + ii*Lx + jj], __fdividef(1.f, (float)ksum));
  }
}

__global__ void normalizeFilter(int Lx, int Ly, float*__restrict Img, const float*__restrict Norm)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if ( i < Lx && j < Ly ) {
    int gtid = j * Lx + i;  
    const float norm = Norm[gtid];
    if (norm != 0) Img[gtid] = __fdividef(Img[gtid], norm);
  }
}

__global__ void outFilter( 
    int Lx, int Ly,
    const float*__restrict Img,
    const   int*__restrict Box,
          float*__restrict Out )
{
  int tid = threadIdx.x;
  int tjd = threadIdx.y;
  int i = blockIdx.x * blockDim.x + tid;
  int j = blockIdx.y * blockDim.y + tjd;
  int stid = tjd * blockDim.x + tid;
  int gtid = j * Lx + i;  

  // part of shared memory may be unused
  __shared__ float s_Img[1024];

  if ( i < Lx && j < Ly )
    s_Img[stid] = Img[gtid];

  __syncthreads();

  if ( i < Lx && j < Ly )
  {
    const int s = Box[gtid];
    float sum = 0.f;
    int ksum  = 0;

    for (int ii = -s; ii < s+1; ii++)
      for (int jj = -s; jj < s+1; jj++)
        if ( (i-s >= 0) && (i+s < Lx) && (j-s >= 0) && (j+s < Ly) )
        {
          ksum++;
          if( tid-s >= 0 && tid+s < blockDim.x && tjd-s >= 0 && tjd+s < blockDim.y )
            sum += s_Img[stid + ii*blockDim.y + jj];
          else
            sum += Img[gtid + ii*Ly + jj];
        }
    if ( ksum != 0 ) Out[gtid] = __fdividef(sum , (float)ksum);
  }
}

int main(int argc, char* argv[]) {
  if (argc != 5) {
     printf("./%s <image dimension> <threshold> <max box size> <iterations>\n", argv[0]);
     exit(1);
  }

  // only a square image is supported
  const int Lx = atoi(argv[1]);
  const int Ly = Lx;
  const int size = Lx * Ly;

  const int Threshold = atoi(argv[2]);
  const int MaxRad = atoi(argv[3]);
  const int repeat = atoi(argv[4]);
 
  // input image
  float *img = (float*) malloc (sizeof(float) * size);

  // host and device results
  float *norm = (float*) malloc (sizeof(float) * size);
  float *h_norm = (float*) malloc (sizeof(float) * size);

  int *box = (int*) malloc (sizeof(int) * size);
  int *h_box = (int*) malloc (sizeof(int) * size);

  float *out = (float*) malloc (sizeof(float) * size);
  float *h_out = (float*) malloc (sizeof(float) * size);

  srand(123);
  for (int i = 0; i < size; i++) {
    img[i] = rand() % 256;
    norm[i] = box[i] = out[i] = 0;
  }

  float *d_img;
  hipMalloc((void**)&d_img, sizeof(float) * size);

  float *d_norm;
  hipMalloc((void**)&d_norm, sizeof(float) * size);

  int *d_box;
  hipMalloc((void**)&d_box, sizeof(int) * size);

  float *d_out;
  hipMalloc((void**)&d_out, sizeof(float) * size);

  dim3 grids ((Lx+15)/16, (Ly+15)/16);
  dim3 blocks (16, 16);

  // reset output
  hipMemcpy(d_out, out, sizeof(float) * size, hipMemcpyHostToDevice);

  for (int i = 0; i < repeat; i++) {
    // restore input image
    hipMemcpy(d_img, img, sizeof(float) * size, hipMemcpyHostToDevice);
    // reset norm
    hipMemcpy(d_norm, norm, sizeof(float) * size, hipMemcpyHostToDevice);
    // launch three kernels
    smoothingFilter<<<grids, blocks>>>(Lx, Ly, Threshold, MaxRad, d_img, d_box, d_norm);
    normalizeFilter<<<grids, blocks>>>(Lx, Ly, d_img, d_norm);
    outFilter<<<grids, blocks>>>(Lx, Ly, d_img, d_box, d_out);
  }

  hipMemcpy(out, d_out, sizeof(float) * size, hipMemcpyDeviceToHost);
  hipMemcpy(box, d_box, sizeof(int) * size, hipMemcpyDeviceToHost);
  hipMemcpy(norm, d_norm, sizeof(float) * size, hipMemcpyDeviceToHost);

  // verify
  reference (Lx, Ly, Threshold, MaxRad, img, h_box, h_norm, h_out);
  verify(size, MaxRad, norm, h_norm, out, h_out, box, h_box);

  hipFree(d_img);
  hipFree(d_norm);
  hipFree(d_box);
  hipFree(d_out);
  free(img);
  free(norm);
  free(h_norm);
  free(box);
  free(h_box);
  free(out);
  free(h_out);
  return 0;
}
