#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <hip/hip_runtime.h>
#include "utils.h"

// kernel execution times
#define REPEAT 100

__global__
void wyllie ( long *list , const int size )
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index < size )
  {
    long node, next;
    while ( ((node = list[index]) >> 32) != NIL && 
            ((next = list[node >> 32]) >> 32) != NIL )
    {
      long temp = (node & MASK) ;
      temp += (next & MASK) ;
      temp += (next >> 32) << 32;
      __syncthreads();
      list [ index ] = temp ;
    }
  }
}

int main(int argc, char* argv[]) {
  if (argc != 3) {
    printf("Usage: ./%s <list size> <0:an ordered list | otherwise: a random list>\n", argv[0]);
    exit(-1);
  }

  int elems = atoi(argv[1]);
  int setRandomList = atoi(argv[2]);
  int i;

  std::vector<int> next (elems);
  std::vector<int> rank (elems);
  std::vector<long> list (elems);
  std::vector<long> d_res (elems);
  std::vector<long> h_res (elems);

  // generate an array in which each element contains the index of the next element
  if (setRandomList)
    random_list(next);
  else
    ordered_list(next);

  // initialize the rank list
  for (i = 0; i < elems; i++) {
    rank[i] = next[i] == NIL ? 0 : 1;
  }

  // pack next and rank as a 64-bit number
  for (i = 0; i < elems; i++) list[i] = ((long)next[i] << 32) | rank[i];

  // run list ranking on a device
  long *d_list;
  hipMalloc((void**)&d_list, sizeof(long) * elems); 

  dim3 grid ((elems + 255)/256);
  dim3 block (256);

  for (i = 0; i < REPEAT; i++) {
    hipMemcpy(d_list, list.data(), sizeof(long) * elems, hipMemcpyHostToDevice);
    hipLaunchKernelGGL(wyllie, dim3(grid), dim3(block), 0, 0, d_list, elems);
  }

  hipMemcpy(d_res.data(), d_list, sizeof(long) * elems, hipMemcpyDeviceToHost);
  hipFree(d_list); 

  for (i = 0; i < elems; i++) d_res[i] &= MASK;

  // verify
  // compute distance from the *end* of the list (note the first element is the head node)
  h_res[0] = elems-1;
  i = 0;
  for (int r = 1; r < elems; r++) {
    h_res[next[i]] = elems-1-r;
    i = next[i];
  }

 
#ifdef DEBUG
  printf("Ranks:\n");
  for (i = 0; i < elems; i++) {
    printf("%d: %ld %ld\n", i, h_res[i], d_res[i]);
  }
#endif

  printf("%s\n", (h_res == d_res) ? "PASS" : "FAIL");
   
  return 0;
}
