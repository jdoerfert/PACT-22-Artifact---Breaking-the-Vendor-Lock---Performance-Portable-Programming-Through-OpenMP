#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "DCT8x8.h"

void DCT8x8(
    float* d_Dst,
    const float* d_Src,
    unsigned int stride,
    unsigned int imageH,
    unsigned int imageW,
    int dir
);

void Verify(const float* h_OutputGPU, 
                  float* h_OutputCPU, 
            const float* h_Input, 
            const unsigned int stride,
            const unsigned int imageH,
            const unsigned int imageW,
            const int dir )
{
  printf("Comparing against Host/C++ computation...\n"); 
  DCT8x8CPU(h_OutputCPU, h_Input, stride, imageH, imageW, dir);
  double sum = 0, delta = 0;
  double L2norm;
  for(unsigned int i = 0; i < imageH; i++)
    for(unsigned int j = 0; j < imageW; j++){
      sum += h_OutputCPU[i * stride + j] * h_OutputCPU[i * stride + j];
      delta += (h_OutputGPU[i * stride + j] - h_OutputCPU[i * stride + j]) * 
               (h_OutputGPU[i * stride + j] - h_OutputCPU[i * stride + j]);
    }
  L2norm = sqrt(delta / sum);
  printf("Relative L2 norm: %.3e\n\n", L2norm);
  if (L2norm < 1E-6) 
    printf("PASS\n"); 
  else
    printf("FAIL\n");
}

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
  float *h_Input, *h_OutputCPU, *h_OutputGPU;

  const unsigned int imageW = 2048, imageH = 2048, stride = 2048;


  printf("Allocating and initializing host memory...\n");
  h_Input     = (float *)malloc(imageH * stride * sizeof(float));
  h_OutputCPU = (float *)malloc(imageH * stride * sizeof(float));
  h_OutputGPU = (float *)malloc(imageH * stride * sizeof(float));
  srand(2009);
  for(unsigned int i = 0; i < imageH; i++)
    for(unsigned int j = 0; j < imageW; j++)
      h_Input[i * stride + j] = (float)rand() / (float)RAND_MAX;

  const int numIterations = 150;


  float* d_Input;
  hipMalloc((void**)&d_Input, sizeof(float) * imageH * stride);
  hipMemcpy(d_Input, h_Input, sizeof(float) * imageH * stride, hipMemcpyHostToDevice);

  float* d_Output;
  hipMalloc((void**)&d_Output, sizeof(float) * imageH * stride);

  int dir = DCT_FORWARD;
  printf("Performing Forward DCT8x8 of %u x %u image on the device\n\n", imageH, imageW);

  for(int iter = 0; iter < numIterations; iter++)
    DCT8x8(
        d_Output,
        d_Input,
        stride,
        imageH,
        imageW,
        dir );

  hipMemcpy(h_OutputGPU, d_Output, sizeof(float) * imageH * stride, hipMemcpyDeviceToHost);

  Verify(h_OutputGPU, h_OutputCPU, h_Input, stride, imageH, imageW, dir);

  dir = DCT_INVERSE;
  printf("Performing Inverse DCT8x8 of %u x %u image on the device\n\n", imageH, imageW);

  for(int iter = 0; iter < numIterations; iter++)
    DCT8x8(
        d_Output,
        d_Input,
        stride,
        imageH,
        imageW,
        dir );

  hipMemcpy(h_OutputGPU, d_Output, sizeof(float) * imageH * stride, hipMemcpyDeviceToHost);

  Verify(h_OutputGPU, h_OutputCPU, h_Input, stride, imageH, imageW, dir);

  hipFree(d_Input);
  hipFree(d_Output);
  free(h_OutputGPU);
  free(h_OutputCPU);
  free(h_Input);

  return 0;
}
