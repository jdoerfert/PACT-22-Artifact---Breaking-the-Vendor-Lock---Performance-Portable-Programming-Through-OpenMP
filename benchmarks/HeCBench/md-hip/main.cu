#include "hip/hip_runtime.h"
#include <cassert>
#include <cfloat>
#include <cmath>
#include <cstdlib>
#include <list>
#include <iostream>
#include <hip/hip_runtime.h>
#include "MD.h"
#include "reference.h"
#include "utils.h"

__global__ void md (
  const POSVECTYPE* __restrict__ position,
        FORCEVECTYPE* __restrict__ force,
  const int* __restrict__ neighborList, 
  const int nAtom,
  const int maxNeighbors, 
  const FPTYPE lj1_t,
  const FPTYPE lj2_t,
  const FPTYPE cutsq_t )
{
  const uint idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx >= nAtom) return;

  POSVECTYPE ipos = position[idx];
  FORCEVECTYPE f = zero;

  int j = 0;
  while (j < maxNeighbors)
  {
    int jidx = neighborList[j*nAtom + idx];

    // Uncoalesced read
    POSVECTYPE jpos = position[jidx];

    // Calculate distance
    FPTYPE delx = ipos.x - jpos.x;
    FPTYPE dely = ipos.y - jpos.y;
    FPTYPE delz = ipos.z - jpos.z;
    FPTYPE r2inv = delx*delx + dely*dely + delz*delz;

    // If distance is less than cutoff, calculate force
    if (r2inv > 0 && r2inv < cutsq_t)
    {
      r2inv = (FPTYPE)1.0 / r2inv;
      FPTYPE r6inv = r2inv * r2inv * r2inv;
      FPTYPE forceC = r2inv * r6inv * (lj1_t * r6inv - lj2_t);

      f.x += delx * forceC;
      f.y += dely * forceC;
      f.z += delz * forceC;
    }
    j++;
  }
  force[idx] = f;
}

int main(int argc, char** argv)
{
  if (argc != 3) {
    std::cout << "Usage: ./" << argv[0] << " <class size> <iteration>\n";
    return 1;
  }

  // Problem Parameters
  int sizeClass = atoi(argv[1]);
  int iteration = atoi(argv[2]);
  const int probSizes[] = { 12288, 24576, 36864, 73728 };
  assert(sizeClass >= 0 && sizeClass < 4);
  assert(iteration >= 0);

  int nAtom = probSizes[sizeClass];

  // Allocate problem data on host
  POSVECTYPE* position = (POSVECTYPE*) malloc(nAtom * sizeof(POSVECTYPE));
  FORCEVECTYPE* h_force = (FORCEVECTYPE*) malloc(nAtom * sizeof(FORCEVECTYPE));
  int *neighborList = (int*) malloc(maxNeighbors * nAtom * sizeof(int));

  std::cout << "Initializing test problem (this can take several minutes for large problems).\n";

  // Seed random number generator
  srand(123);

  // Notes on positions 
  // When the potential energy becomes exceedingly large as the distance 
  // between two atoms is very close, the host and device results may differ significantly
  for (int i = 0; i < nAtom; i++)
  {
    position[i].x = rand() % domainEdge;
    position[i].y = rand() % domainEdge;
    position[i].z = rand() % domainEdge;
  }

  std::cout << "Finished.\n";
  int totalPairs = buildNeighborList<FPTYPE, POSVECTYPE>(nAtom, position, neighborList);
  std::cout << totalPairs << " of " << nAtom*maxNeighbors << " pairs within cutoff distance = " 
       << 100.0 * ((double)totalPairs / (nAtom*maxNeighbors)) << " %\n";

  POSVECTYPE*   d_position;
  FORCEVECTYPE* d_force;
  int* d_neighborList;
  hipMalloc((void**)&d_force, nAtom * sizeof(FORCEVECTYPE));
  hipMalloc((void**)&d_position, nAtom * sizeof(POSVECTYPE));
  hipMalloc((void**)&d_neighborList, nAtom * maxNeighbors * sizeof(int));

  hipMemcpy(d_position, position, nAtom * sizeof(POSVECTYPE), hipMemcpyHostToDevice);
  hipMemcpy(d_neighborList, neighborList, nAtom * maxNeighbors * sizeof(int), hipMemcpyHostToDevice);

  dim3 grids ((nAtom+255) / 256);
  dim3 block (256);

  // warmup and result verification
  hipLaunchKernelGGL(md, grids, block, 0, 0, d_position, d_force, d_neighborList,
      nAtom, maxNeighbors, lj1, lj2, cutsq);

  hipMemcpy(h_force, d_force, nAtom * sizeof(FORCEVECTYPE), hipMemcpyDeviceToHost);

  std::cout << "Performing Correctness Check (may take several minutes)\n";

  checkResults<FPTYPE, FORCEVECTYPE, POSVECTYPE>(h_force, position, neighborList, nAtom);

  for (int j = 0; j < iteration; j++)
  {
    hipLaunchKernelGGL(md, grids, block, 0, 0, d_position, d_force, d_neighborList,
      nAtom, maxNeighbors, lj1, lj2, cutsq);
  }
  hipDeviceSynchronize();

  hipFree(d_position);
  hipFree(d_force);
  hipFree(d_neighborList);

  free(position);
  free(h_force);
  free(neighborList);

  return 0;
}
