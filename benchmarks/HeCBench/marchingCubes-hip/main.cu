#include "hip/hip_runtime.h"
//
// An implementation of Parallel Marching Blocks algorithm
//

#include <cstdio>
#include <random>
#include <hip/hip_runtime.h>
#include "tables.h"

// problem size
constexpr unsigned int N(1024);
constexpr unsigned int Nd2(N / 2);
constexpr unsigned int voxelXLv1(16);
constexpr unsigned int voxelYLv1(16);
constexpr unsigned int voxelZLv1(64);
constexpr unsigned int gridXLv1((N - 1) / (voxelXLv1 - 1));
constexpr unsigned int gridYLv1((N - 1) / (voxelYLv1 - 1));
constexpr unsigned int gridZLv1((N - 1) / (voxelZLv1 - 1));
constexpr unsigned int countingThreadNumLv1(128);
constexpr unsigned int blockNum(gridXLv1* gridYLv1* gridZLv1);
constexpr unsigned int countingBlockNumLv1(blockNum / countingThreadNumLv1);

constexpr unsigned int voxelXLv2(4);
constexpr unsigned int voxelYLv2(4);
constexpr unsigned int voxelZLv2(8);
constexpr unsigned int blockXLv2(5);
constexpr unsigned int blockYLv2(5);
constexpr unsigned int blockZLv2(9);
constexpr unsigned int voxelNumLv2(blockXLv2* blockYLv2* blockZLv2);

constexpr unsigned int countingThreadNumLv2(1024);
constexpr unsigned int gridXLv2(gridXLv1* blockXLv2);
constexpr unsigned int gridYLv2(gridYLv1* blockYLv2);
//constexpr unsigned int gridZLv2(gridZLv1* blockZLv2);

__inline__ __device__ float f(unsigned int x, unsigned int y, unsigned int z)
{
  constexpr float d(2.0f / N);
  float xf((int(x - Nd2)) * d);//[-1, 1)
  float yf((int(z - Nd2)) * d);
  float zf((int(z - Nd2)) * d);
  return 1.f - 16.f * xf * yf * zf - 4.f * (xf * xf + yf * yf + zf * zf);
}

__inline__ __device__ float zeroPoint(unsigned int x, float v0, float v1, float isoValue)
{
  return ((x * (v1 - isoValue) + (x + 1) * (isoValue - v0)) / (v1 - v0) - Nd2) * (2.0f / N);
}

__inline__ __device__ float transformToCoord(unsigned int x)
{
  return (int(x) - int(Nd2)) * (2.0f / N);
}

__global__ void computeMinMaxLv1(float*__restrict minMax)
{
  __shared__ float sminMax[64];
  constexpr unsigned int threadNum(voxelXLv1 * voxelYLv1);
  constexpr unsigned int warpNum(threadNum / 32);
  unsigned int x(blockIdx.x * (voxelXLv1 - 1) + threadIdx.x);
  unsigned int y(blockIdx.y * (voxelYLv1 - 1) + threadIdx.y);
  unsigned int z(blockIdx.z * (voxelZLv1 - 1));
  unsigned int tid(threadIdx.x + voxelXLv1 * threadIdx.y);
  unsigned int laneid = tid % 32;
  unsigned int blockid(blockIdx.x + gridXLv1 * (blockIdx.y + gridYLv1 * blockIdx.z));
  unsigned int warpid(tid >> 5);
  float v(f(x, y, z));
  float minV(v), maxV(v);
  for (int c0(1); c0 < voxelZLv1; ++c0)
  {
    v = f(x, y, z + c0);
    if (v < minV)minV = v;
    if (v > maxV)maxV = v;
  }
#pragma unroll
  for (int c0(16); c0 > 0; c0 /= 2)
  {
    float t0, t1;
    t0 = __shfl_down(minV, c0);
    t1 = __shfl_down(maxV, c0);
    if (t0 < minV)minV = t0;
    if (t1 > maxV)maxV = t1;
  }
  if (laneid == 0)
  {
    sminMax[warpid] = minV;
    sminMax[warpid + warpNum] = maxV;
  }
  __syncthreads();
  if (warpid == 0)
  {
    minV = sminMax[laneid];
    maxV = sminMax[laneid + warpNum];
#pragma unroll
    for (int c0(warpNum / 2); c0 > 0; c0 /= 2)
    {
      float t0, t1;
      t0 = __shfl_down(minV, c0);
      t1 = __shfl_down(maxV, c0);
      if (t0 < minV)minV = t0;
      if (t1 > maxV)maxV = t1;
    }
    if (laneid == 0)
    {
      minMax[blockid * 2] = minV;
      minMax[blockid * 2 + 1] = maxV;
    }
  }
}

__global__ void compactLv1(
  float isoValue, 
  const float*__restrict minMax,
  unsigned int*__restrict blockIndices,
  unsigned int*__restrict countedBlockNum)
{
  __shared__ unsigned int sums[32];
  constexpr unsigned int warpNum(countingThreadNumLv1 / 32);
  unsigned int tid(threadIdx.x);
  unsigned int laneid = tid % 32;
  unsigned int bIdx(blockIdx.x * countingThreadNumLv1 + tid);
  unsigned int warpid(tid >> 5);
  unsigned int test;
  if (minMax[2 * bIdx] <= isoValue && minMax[2 * bIdx + 1] >= isoValue)test = 1;
  else test = 0;
  unsigned int testSum(test);
#pragma unroll
  for (int c0(1); c0 < 32; c0 *= 2)
  {
    unsigned int tp(__shfl_up(testSum, c0));
    if (laneid >= c0)testSum += tp;
  }
  if (laneid == 31)sums[warpid] = testSum;
  __syncthreads();
  if (warpid == 0)
  {
    unsigned int warpSum = sums[laneid];
#pragma unroll
    for (int c0(1); c0 < warpNum; c0 *= 2)
    {
      unsigned int tp(__shfl_up(warpSum, c0));
      if (laneid >= c0) warpSum += tp;
    }
    sums[laneid] = warpSum;
  }
  __syncthreads();
  if (warpid != 0)testSum += sums[warpid - 1];
  if (tid == countingThreadNumLv1 - 1 && testSum != 0)
    sums[31] = atomicAdd(countedBlockNum, testSum);
  __syncthreads();
  if (test)blockIndices[testSum + sums[31] - 1] = bIdx;
}

__global__ void computeMinMaxLv2(
  const unsigned int*__restrict blockIndicesLv1,
  float*__restrict minMax)
{
  unsigned int tid(threadIdx.x);
  unsigned int voxelOffset(threadIdx.y);
  unsigned int blockIndex(blockIndicesLv1[blockIdx.x]);
  unsigned int tp(blockIndex);
  unsigned int x((blockIndex % gridXLv1) * (voxelXLv1 - 1) + (voxelOffset % 5) * (voxelXLv2 - 1) + (tid & 3));
  tp /= gridXLv1;
  unsigned int y((tp % gridYLv1) * (voxelYLv1 - 1) + (voxelOffset / 5) * (voxelYLv2 - 1) + (tid >> 2));
  tp /= gridYLv1;
  unsigned int z(tp * (voxelZLv1 - 1));
  float v(f(x, y, z));
  float minV(v), maxV(v);
  unsigned int idx(2 * (voxelOffset + voxelNumLv2 * blockIdx.x));
  for (int c0(0); c0 < blockZLv2; ++c0)
  {
    for (int c1(1); c1 < voxelZLv2; ++c1)
    {
      v = f(x, y, z + c1);
      if (v < minV)minV = v;
      if (v > maxV)maxV = v;
    }
    z += voxelZLv2 - 1;
#pragma unroll
    for (int c1(8); c1 > 0; c1 /= 2)
    {
      float t0, t1;
      t0 = __shfl_down(minV, c1);
      t1 = __shfl_down(maxV, c1);
      if (t0 < minV)minV = t0;
      if (t1 > maxV)maxV = t1;
    }
    if (tid == 0)
    {
      minMax[idx] = minV;
      minMax[idx + 1] = maxV;
      constexpr unsigned int offsetSize(2 * blockXLv2 * blockYLv2);
      idx += offsetSize;
    }
    minV = v;
    maxV = v;
  }
}

__global__ void compactLv2(
  float isoValue,
  const float*__restrict minMax,
  const unsigned int*__restrict blockIndicesLv1,
  unsigned int*__restrict blockIndicesLv2,
  unsigned int counterBlockNumLv1,
  unsigned int*__restrict countedBlockNumLv2)
{
  __shared__ unsigned int sums[32];
  constexpr unsigned int warpNum(countingThreadNumLv2 / 32);
  unsigned int tid(threadIdx.x);
  unsigned int laneid = tid % 32;
  unsigned int warpid(tid >> 5);
  unsigned int id0(tid + blockIdx.x * countingThreadNumLv2);
  unsigned int id1(id0 / voxelNumLv2);
  unsigned int test;
  if (id1 < counterBlockNumLv1)
  {
    if (minMax[2 * id0] <= isoValue && minMax[2 * id0 + 1] >= isoValue)
      test = 1;
    else
      test = 0;
  }
  else test = 0;
  unsigned int testSum(test);
#pragma unroll
  for (int c0(1); c0 < 32; c0 *= 2)
  {
    unsigned int tp(__shfl_up(testSum, c0));
    if (laneid >= c0)testSum += tp;
  }
  if (laneid == 31)sums[warpid] = testSum;
  __syncthreads();
  if (warpid == 0)
  {
    unsigned warpSum = sums[laneid];
#pragma unroll
    for (int c0(1); c0 < warpNum; c0 *= 2)
    {
      unsigned int tp(__shfl_up(warpSum, c0));
      if (laneid >= c0)warpSum += tp;
    }
    sums[laneid] = warpSum;
  }
  __syncthreads();
  if (warpid != 0)testSum += sums[warpid - 1];
  if (tid == countingThreadNumLv2 - 1)
    sums[31] = atomicAdd(countedBlockNumLv2, testSum);
  __syncthreads();

  if (test)
  {
    unsigned int bIdx1(blockIndicesLv1[id1]);
    unsigned int bIdx2;
    unsigned int x1, y1, z1;
    unsigned int x2, y2, z2;
    unsigned int tp1(bIdx1);
    unsigned int tp2((tid + blockIdx.x * countingThreadNumLv2) % voxelNumLv2);
    x1 = tp1 % gridXLv1;
    x2 = tp2 % blockXLv2;
    tp1 /= gridXLv1;
    tp2 /= blockXLv2;
    y1 = tp1 % gridYLv1;
    y2 = tp2 % blockYLv2;
    z1 = tp1 / gridYLv1;
    z2 = tp2 / blockYLv2;
    bIdx2 = x2 + blockXLv2 * (x1 + gridXLv1 * (y2 + blockYLv2 * (y1 + gridYLv1 * (z1 * blockZLv2 + z2))));
    blockIndicesLv2[testSum + sums[31] - 1] = bIdx2;
  }
}

__global__ void generatingTriangles(
  float isoValue, 
  const unsigned int*__restrict blockIndicesLv2,
  const unsigned short *__restrict distinctEdgesTable,
  const int *__restrict triTable,
  const uchar4 *__restrict edgeIDTable,
  unsigned int*__restrict countedVerticesNum,
  unsigned int*__restrict countedTrianglesNum,
  unsigned long long*__restrict triangles,
  float*__restrict coordX,
  float*__restrict coordY,
  float*__restrict coordZ,
  float*__restrict coordZP)
{
  __shared__ unsigned short vertexIndices[voxelZLv2][voxelYLv2][voxelXLv2];
  __shared__ float value[voxelZLv2 + 1][voxelYLv2 + 1][voxelXLv2 + 1];
  __shared__ unsigned int sumsVertices[32];
  __shared__ unsigned int sumsTriangles[32];

  unsigned int blockId(blockIndicesLv2[blockIdx.x]);
  unsigned int tp(blockId);
  unsigned int x((tp % gridXLv2) * (voxelXLv2 - 1) + threadIdx.x);
  tp /= gridXLv2;
  unsigned int y((tp % gridYLv2) * (voxelYLv2 - 1) + threadIdx.y);
  unsigned int z((tp / gridYLv2) * (voxelZLv2 - 1) + threadIdx.z);
  unsigned int eds(7);
  float v(value[threadIdx.z][threadIdx.y][threadIdx.x] = f(x, y, z));
  if (threadIdx.x == voxelXLv2 - 1)
  {
    eds &= 6;
    value[threadIdx.z][threadIdx.y][voxelXLv2] = f(x + 1, y, z);
    if (threadIdx.y == voxelYLv2 - 1)
      value[threadIdx.z][voxelYLv2][voxelXLv2] = f(x + 1, y + 1, z);
  }
  if (threadIdx.y == voxelYLv2 - 1)
  {
    eds &= 5;
    value[threadIdx.z][voxelYLv2][threadIdx.x] = f(x, y + 1, z);
    if (threadIdx.z == voxelZLv2 - 1)
      value[voxelZLv2][voxelYLv2][threadIdx.x] = f(x, y + 1, z + 1);
  }
  if (threadIdx.z == voxelZLv2 - 1)
  {
    eds &= 3;
    value[voxelZLv2][threadIdx.y][threadIdx.x] = f(x, y, z + 1);
    if (threadIdx.x == voxelXLv2 - 1)
      value[voxelZLv2][threadIdx.y][voxelXLv2] = f(x + 1, y, z + 1);
  }
  eds <<= 13;
  __syncthreads();
  unsigned int cubeCase(0);
  if (value[threadIdx.z][threadIdx.y][threadIdx.x] < isoValue) cubeCase |= 1;
  if (value[threadIdx.z][threadIdx.y][threadIdx.x + 1] < isoValue) cubeCase |= 2;
  if (value[threadIdx.z][threadIdx.y + 1][threadIdx.x + 1] < isoValue) cubeCase |= 4;
  if (value[threadIdx.z][threadIdx.y + 1][threadIdx.x] < isoValue) cubeCase |= 8;
  if (value[threadIdx.z + 1][threadIdx.y][threadIdx.x] < isoValue) cubeCase |= 16;
  if (value[threadIdx.z + 1][threadIdx.y][threadIdx.x + 1] < isoValue) cubeCase |= 32;
  if (value[threadIdx.z + 1][threadIdx.y + 1][threadIdx.x + 1] < isoValue) cubeCase |= 64;
  if (value[threadIdx.z + 1][threadIdx.y + 1][threadIdx.x] < isoValue) cubeCase |= 128;

  unsigned int distinctEdges(eds ? distinctEdgesTable[cubeCase] : 0);
  unsigned int numTriangles(eds != 0xe000 ? 0 : distinctEdges & 7);
  unsigned int numVertices(__popc(distinctEdges &= eds));
  unsigned int laneid = (threadIdx.x + voxelXLv2 * (threadIdx.y + voxelYLv2 * threadIdx.z)) % 32;
  unsigned warpid((threadIdx.x + voxelXLv2 * (threadIdx.y + voxelYLv2 * threadIdx.z)) >> 5);
  constexpr unsigned int threadNum(voxelXLv2 * voxelYLv2 * voxelZLv2);
  constexpr unsigned int warpNum(threadNum / 32);
  unsigned int sumVertices(numVertices);
  unsigned int sumTriangles(numTriangles);

#pragma unroll
  for (int c0(1); c0 < 32; c0 *= 2)
  {
    unsigned int tp0(__shfl_up(sumVertices, c0));
    unsigned int tp1(__shfl_up(sumTriangles, c0));
    if (laneid >= c0)
    {
      sumVertices += tp0;
      sumTriangles += tp1;
    }
  }
  if (laneid == 31)
  {
    sumsVertices[warpid] = sumVertices;
    sumsTriangles[warpid] = sumTriangles;
  }
  __syncthreads();
  if (warpid == 0)
  {
    unsigned warpSumVertices = sumsVertices[laneid];
    unsigned warpSumTriangles = sumsTriangles[laneid];
#pragma unroll
    for (int c0(1); c0 < warpNum; c0 *= 2)
    {
      unsigned int tp0(__shfl_up(warpSumVertices, c0));
      unsigned int tp1(__shfl_up(warpSumTriangles, c0));
      if (laneid >= c0)
      {
        warpSumVertices += tp0;
        warpSumTriangles += tp1;
      }
    }
    sumsVertices[laneid] = warpSumVertices;
    sumsTriangles[laneid] = warpSumTriangles;
  }
  __syncthreads();
  if (warpid != 0)
  {
    sumVertices += sumsVertices[warpid - 1];
    sumTriangles += sumsTriangles[warpid - 1];
  }
  if (eds == 0)
  {
    sumsVertices[31] = atomicAdd(countedVerticesNum, sumVertices);
    sumsTriangles[31] = atomicAdd(countedTrianglesNum, sumTriangles);
  }

  unsigned int interOffsetVertices(sumVertices - numVertices);
  sumVertices = interOffsetVertices + sumsVertices[31];//exclusive offset
  sumTriangles = sumTriangles + sumsTriangles[31] - numTriangles;//exclusive offset
  vertexIndices[threadIdx.z][threadIdx.y][threadIdx.x] = interOffsetVertices | distinctEdges;
  __syncthreads();

  for (unsigned int c0(0); c0 < numTriangles; ++c0)
  {
#pragma unroll
    for (unsigned int c1(0); c1 < 3; ++c1)
    {
      int edgeID(triTable[16 * cubeCase + 3 * c0 + c1]);
      uchar4 edgePos(edgeIDTable[edgeID]);
      unsigned short vertexIndex(vertexIndices[threadIdx.z + edgePos.z][threadIdx.y + edgePos.y][threadIdx.x + edgePos.x]);
      unsigned int tp(__popc(vertexIndex >> (16 - edgePos.w)) + (vertexIndex & 0x1fff));
      atomicAdd(triangles, (unsigned long long)(sumsVertices[31] + tp));
    }
  }

  // sumVertices may be too large for a GPU memory
  float zp = 0.f, cx = 0.f, cy = 0.f, cz = 0.f;

  if (distinctEdges & (1 << 15))
  {
    zp = zeroPoint(x, v, value[threadIdx.z][threadIdx.y][threadIdx.x + 1], isoValue);
    cy = transformToCoord(y);
    cz = transformToCoord(z);
  }
  if (distinctEdges & (1 << 14))
  {
    cx = transformToCoord(x);
    zp += zeroPoint(y, v, value[threadIdx.z][threadIdx.y + 1][threadIdx.x], isoValue);
    cz += transformToCoord(z);
  }
  if (distinctEdges & (1 << 13))
  {
    cx += transformToCoord(x);
    cy += transformToCoord(y);
    zp += zeroPoint(z, v, value[threadIdx.z + 1][threadIdx.y][threadIdx.x], isoValue);
  }
  atomicAdd(coordX, cx);
  atomicAdd(coordY, cy);
  atomicAdd(coordZ, cz);
  atomicAdd(coordZP, zp);
}

int main(int argc, char* argv[])
{
  unsigned int iterations = atoi(argv[1]);

  std::uniform_real_distribution<float>rd(0, 1);
  std::mt19937 mt(123);

  float* minMaxLv1Device;
  float* minMaxLv2Device;
  unsigned int* blockIndicesLv1Device;
  unsigned int* blockIndicesLv2Device;
  unsigned int* countedBlockNumLv1Device;
  unsigned int* countedBlockNumLv2Device;
  unsigned short* distinctEdgesTableDevice;
  int* triTableDevice;
  uchar4* edgeIDTableDevice;
  unsigned int* countedVerticesNumDevice;
  unsigned int* countedTrianglesNumDevice;
  unsigned long long* trianglesDevice;
  float *coordXDevice;
  float *coordYDevice;
  float *coordZDevice;
  float *coordZPDevice;

  hipMalloc(&minMaxLv1Device, blockNum * 2 * sizeof(float));
  hipMalloc(&blockIndicesLv1Device, blockNum * sizeof(unsigned int));
  hipMalloc(&countedBlockNumLv1Device, sizeof(unsigned int));
  hipMalloc(&countedBlockNumLv2Device, sizeof(unsigned int));
  hipMalloc(&distinctEdgesTableDevice, sizeof(distinctEdgesTable));
  hipMalloc(&triTableDevice, sizeof(triTable));
  hipMalloc(&edgeIDTableDevice, sizeof(edgeIDTable));
  hipMalloc(&countedVerticesNumDevice, sizeof(unsigned int));
  hipMalloc(&countedTrianglesNumDevice, sizeof(unsigned int));
  hipMemcpy(distinctEdgesTableDevice, distinctEdgesTable, sizeof(distinctEdgesTable), hipMemcpyHostToDevice);
  hipMemcpy(triTableDevice, triTable, sizeof(triTable), hipMemcpyHostToDevice);
  hipMemcpy(edgeIDTableDevice, edgeIDTable, sizeof(edgeIDTable), hipMemcpyHostToDevice);

  // simulate rendering without memory allocation for vertices and triangles 
  hipMalloc(&trianglesDevice, sizeof(unsigned long long));
  hipMalloc(&coordXDevice, sizeof(float));
  hipMalloc(&coordYDevice, sizeof(float));
  hipMalloc(&coordZDevice, sizeof(float));
  hipMalloc(&coordZPDevice, sizeof(float));

  const dim3 BlockSizeLv1{ voxelXLv1, voxelYLv1, 1 };
  const dim3 GridSizeLv1{ gridXLv1, gridYLv1, gridZLv1 };
  
  const dim3 BlockSizeLv2{ voxelXLv2 * voxelYLv2, blockXLv2 * blockYLv2, 1 };
  const dim3 BlockSizeGenerating{ voxelXLv2, voxelYLv2, voxelZLv2 };

  float isoValue(-0.9f);

  unsigned int countedBlockNumLv1;
  unsigned int countedBlockNumLv2;
  unsigned int countedVerticesNum;
  unsigned int countedTrianglesNum;

  for (unsigned int c0(0); c0 < iterations; ++c0)
  {
    hipDeviceSynchronize();
    hipMemset(countedBlockNumLv1Device, 0, sizeof(unsigned int));
    hipMemset(countedBlockNumLv2Device, 0, sizeof(unsigned int));
    hipMemset(countedVerticesNumDevice, 0, sizeof(unsigned int));
    hipMemset(countedTrianglesNumDevice,0, sizeof(unsigned int));
    hipMemset(trianglesDevice, 0, sizeof(unsigned long long));
    hipMemset(coordXDevice, 0, sizeof(float));
    hipMemset(coordYDevice, 0, sizeof(float));
    hipMemset(coordZDevice, 0, sizeof(float));
    hipMemset(coordZPDevice, 0, sizeof(float));

    hipLaunchKernelGGL(computeMinMaxLv1, GridSizeLv1, BlockSizeLv1, 0, 0, minMaxLv1Device);
    hipLaunchKernelGGL(compactLv1, dim3(countingBlockNumLv1), dim3(countingThreadNumLv1), 0, 0, 
      isoValue, minMaxLv1Device, blockIndicesLv1Device, countedBlockNumLv1Device);

    hipMemcpy(&countedBlockNumLv1, countedBlockNumLv1Device, sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMalloc(&minMaxLv2Device, countedBlockNumLv1 * voxelNumLv2 * 2 * sizeof(float));

    hipLaunchKernelGGL(computeMinMaxLv2, dim3(countedBlockNumLv1), BlockSizeLv2, 0, 0, blockIndicesLv1Device, minMaxLv2Device);

    hipMalloc(&blockIndicesLv2Device, countedBlockNumLv1 * voxelNumLv2 * sizeof(unsigned int));
    unsigned int countingBlockNumLv2((countedBlockNumLv1 * voxelNumLv2 + countingThreadNumLv2 - 1) / countingThreadNumLv2);

    hipLaunchKernelGGL(compactLv2, dim3(countingBlockNumLv2), dim3(countingThreadNumLv2 ), 0, 0, 
      isoValue, minMaxLv2Device, blockIndicesLv1Device, blockIndicesLv2Device, countedBlockNumLv1, countedBlockNumLv2Device);

    hipMemcpy(&countedBlockNumLv2, countedBlockNumLv2Device, sizeof(unsigned int), hipMemcpyDeviceToHost);

    hipLaunchKernelGGL(generatingTriangles, dim3(countedBlockNumLv2), BlockSizeGenerating, 0, 0, 
        isoValue, blockIndicesLv2Device,
        distinctEdgesTableDevice, triTableDevice, edgeIDTableDevice,
        countedVerticesNumDevice, countedTrianglesNumDevice, trianglesDevice,
        coordXDevice, coordYDevice, coordZDevice, coordZPDevice);

    hipMemcpy(&countedVerticesNum, countedVerticesNumDevice, sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(&countedTrianglesNum, countedTrianglesNumDevice, sizeof(unsigned int), hipMemcpyDeviceToHost);

    hipFree(minMaxLv2Device);
    hipFree(blockIndicesLv2Device);
  }

  printf("Block Lv1: %u\nBlock Lv2: %u\n", countedBlockNumLv1, countedBlockNumLv2);
  printf("Vertices Size: %u\n", countedBlockNumLv2 * 304);
  printf("Triangles Size: %u\n", countedBlockNumLv2 * 315 * 3);
  printf("Vertices: %u\nTriangles: %u\n", countedVerticesNum, countedTrianglesNum);

  // specific to the problem size
  bool ok = (countedBlockNumLv1 == 8296 && countedBlockNumLv2 == 240380 &&
             countedVerticesNum == 4856560 && countedTrianglesNum == 6101640);
  printf("%s\n", ok ? "PASS" : "FAIL");

  hipFree(minMaxLv1Device);
  hipFree(blockIndicesLv1Device);
  hipFree(countedBlockNumLv1Device);
  hipFree(countedBlockNumLv2Device);
  hipFree(distinctEdgesTableDevice);
  hipFree(triTableDevice);
  hipFree(edgeIDTableDevice);
  hipFree(countedVerticesNumDevice);
  hipFree(countedTrianglesNumDevice);
  hipFree(trianglesDevice);
  hipFree(coordXDevice);
  hipFree(coordYDevice);
  hipFree(coordZDevice);
  hipFree(coordZPDevice);
  return 0;
}
