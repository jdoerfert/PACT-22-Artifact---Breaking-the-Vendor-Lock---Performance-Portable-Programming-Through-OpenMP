#include "hip/hip_runtime.h"
#include <vector>
#include <cstring>
#include <ctime>
#include <cstdio>
#include <hip/hip_runtime.h>
#include "device_kernel_wrapper.h"
#include "datatypes.h"
#include "kernel_common.h"
#include "memory_scheduler.h"


__global__
void device_chain_tiled(
        return_dt *ret, const anchor_dt *a,
        const control_dt *control, score_dt *max_tracker_g, parent_dt *j_tracker_g,
        const int max_dist_x, const int max_dist_y, const int bw);

__host__
void device_chain_kernel_wrapper(
        std::vector<control_dt> &cont,
        std::vector<anchor_dt> &arg,
        std::vector<return_dt> &ret,
        int max_dist_x, int max_dist_y, int bw)
{
    auto batch_count = cont.size() / PE_NUM;

    control_dt *h_control;
    anchor_dt *h_arg;
    return_dt *h_ret;

    hipHostMalloc(&h_control, cont.size() * sizeof(control_dt));
    hipHostMalloc(&h_arg, arg.size() * sizeof(anchor_dt));
    hipHostMalloc(&h_ret, batch_count * TILE_SIZE * PE_NUM * sizeof(return_dt));
    ret.resize(batch_count * TILE_SIZE * PE_NUM);

    memcpy(h_control, cont.data(), cont.size() * sizeof(control_dt));
    memcpy(h_arg, arg.data(), arg.size() * sizeof(anchor_dt));

    struct timespec start, end;
    clock_gettime(CLOCK_BOOTTIME, &start);

    control_dt *d_control;
    anchor_dt *d_arg;
    return_dt *d_ret;
    score_dt *d_max_tracker;
    parent_dt *d_j_tracker;

    hipMalloc((void**)&d_control, cont.size() * sizeof(control_dt));
    hipMalloc((void**)&d_arg, arg.size() * sizeof(anchor_dt));
    hipMalloc((void**)&d_ret, batch_count * TILE_SIZE * PE_NUM * sizeof(return_dt));
    hipMalloc((void**)&d_max_tracker, PE_NUM * BACK_SEARCH_COUNT_GPU * sizeof(score_dt));
    hipMalloc((void**)&d_j_tracker, PE_NUM * BACK_SEARCH_COUNT_GPU * sizeof(parent_dt));

    hipMemcpy(d_control, h_control, cont.size() * sizeof(control_dt), hipMemcpyHostToDevice);
    hipMemcpy(d_arg, h_arg, arg.size() * sizeof(anchor_dt), hipMemcpyHostToDevice);


    for (auto batch = 0; batch < batch_count; batch++) {
            hipLaunchKernelGGL(device_chain_tiled, dim3(BLOCK_NUM), dim3(BACK_SEARCH_COUNT_GPU), 0, 0, 
                    d_ret + batch * PE_NUM * TILE_SIZE,
                    d_arg + batch * PE_NUM * TILE_SIZE_ACTUAL,
                    d_control + batch * PE_NUM ,
                    d_max_tracker,
                    d_j_tracker,
                    max_dist_x, max_dist_y, bw);
    }

    hipMemcpy(h_ret, d_ret, batch_count * TILE_SIZE * PE_NUM * sizeof(return_dt), hipMemcpyDeviceToHost);

    hipFree(d_control);
    hipFree(d_arg);
    hipFree(d_ret);
    hipFree(d_max_tracker);
    hipFree(d_j_tracker);

    clock_gettime(CLOCK_BOOTTIME, &end);
    printf(" ***** offloading took %f seconds for end-to-end\n",
        ( end.tv_sec - start.tv_sec ) + ( end.tv_nsec - start.tv_nsec ) / 1E9);

    memcpy(ret.data(), h_ret, batch_count * TILE_SIZE * PE_NUM * sizeof(return_dt));
}

